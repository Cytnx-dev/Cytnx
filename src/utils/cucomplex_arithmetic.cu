#include "Type.hpp"
#include "utils/cucomplex_arithmetic.hpp"
namespace cytnx {

#ifdef UNI_GPU

  //-----------------------------
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const hipDoubleComplex &rn) {
    return (rn.x == ln.x) && (rn.y == ln.y);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const hipFloatComplex &rn) {
    return ln == hipComplexFloatToDouble(rn);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_double &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_float &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_uint64 &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_uint32 &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_int64 &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_int32 &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_uint16 &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_int16 &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }
  __host__ __device__ bool operator==(const hipDoubleComplex &ln, const cytnx_bool &rn) {
    return ln == make_hipDoubleComplex(rn, 0);
  }

  //-----------------------------
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const hipFloatComplex &rn) {
    return (ln.x == rn.x) && (ln.y == rn.y);
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const hipDoubleComplex &rn) {
    return hipComplexFloatToDouble(ln) == rn;
  }

  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_double &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_float &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_uint64 &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_uint32 &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_int64 &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_int32 &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_uint16 &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_int16 &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const hipFloatComplex &ln, const cytnx_bool &rn) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  //-----------------------
  //__host__ __device__ bool operator==(const hipDoubleComplex &rn,const hipDoubleComplex &ln);
  //__host__ __device__ bool operator==(const hipFloatComplex &rn,const hipDoubleComplex &ln);
  __host__ __device__ bool operator==(const cytnx_double &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_float &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_uint64 &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_uint32 &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_int64 &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_int32 &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_uint16 &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_int16 &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_bool &rn, const hipDoubleComplex &ln) {
    return make_hipDoubleComplex(rn, 0) == ln;
  }
  //----------------------

  //__host__ __device__ bool operator==(const hipDoubleComplex &rn,const hipFloatComplex &ln);
  //__host__ __device__ bool operator==(const hipFloatComplex &rn,const hipFloatComplex &ln);
  __host__ __device__ bool operator==(const cytnx_double &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_float &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_uint64 &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_uint32 &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_int64 &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_int32 &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_uint16 &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_int16 &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }
  __host__ __device__ bool operator==(const cytnx_bool &rn, const hipFloatComplex &ln) {
    return make_hipFloatComplex(rn, 0) == ln;
  }

  //[MUL]

  //-----------------------------
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln,
                                                const hipDoubleComplex &rn) {
    return hipCmul(ln, rn);
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln,
                                                const hipFloatComplex &rn) {
    return hipCmul(ln, hipComplexFloatToDouble(rn));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_double &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_float &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_uint64 &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_uint32 &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_int64 &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_int32 &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_uint16 &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_int16 &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }
  __host__ __device__ hipDoubleComplex operator*(const hipDoubleComplex &ln, const cytnx_bool &rn) {
    return hipCmul(ln, make_hipDoubleComplex(rn, 0));
  }

  //-----------------------------
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const hipFloatComplex &rn) {
    return hipCmulf(ln, rn);
  }
  __host__ __device__ hipDoubleComplex operator*(const hipFloatComplex &ln,
                                                const hipDoubleComplex &rn) {
    return hipCmul(hipComplexFloatToDouble(ln), rn);
  }

  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_double &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_float &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_uint64 &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_uint32 &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_int64 &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_int32 &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_uint16 &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_int16 &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const hipFloatComplex &ln, const cytnx_bool &rn) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  //-----------------------
  __host__ __device__ hipDoubleComplex operator*(const cytnx_double &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_float &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_uint64 &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_uint32 &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_int64 &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_int32 &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_uint16 &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const cytnx_int16 &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  __host__ __device__ hipDoubleComplex operator*(const bool &rn, const hipDoubleComplex &ln) {
    return hipCmul(make_hipDoubleComplex(rn, 0), ln);
  }
  //----------------------

  __host__ __device__ hipFloatComplex operator*(const cytnx_double &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_float &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_uint64 &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_uint32 &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_int64 &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_int32 &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_uint16 &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_int16 &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }
  __host__ __device__ hipFloatComplex operator*(const cytnx_bool &rn, const hipFloatComplex &ln) {
    return hipCmulf(make_hipFloatComplex(rn, 0), ln);
  }

#endif

}  // namespace cytnx
