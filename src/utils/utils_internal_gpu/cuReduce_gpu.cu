#include "hip/hip_runtime.h"
#include "cuReduce_gpu.hpp"
#include "utils/complex_arithmetic.hpp"

namespace cytnx {
  namespace utils_internal {

#define _TNinB_REDUCE_ 512

    template <class X>
    __device__ void warp_unroll(X* smem, int tid) {
      X v = smem[tid];
      __syncwarp();
      v += __shfl_down_sync(0xFFFFFFFFU, v, 16);
      v += __shfl_down_sync(0xFFFFFFFFU, v, 8);
      v += __shfl_down_sync(0xFFFFFFFFU, v, 4);
      v += __shfl_down_sync(0xFFFFFFFFU, v, 2);
      v += __shfl_down_sync(0xFFFFFFFFU, v, 1);
      smem[tid] = v;
      __syncwarp();
    }

    // require, threads per block to be 32*(2^n), n =0,1,2,3,4,5
    template <class T>
    __global__ void cuReduce_kernel(T* out, T* in, cytnx_uint64 Nelem) {
      __shared__ T sD[_TNinB_REDUCE_];  // allocate share mem for each thread
      sD[threadIdx.x] = 0;

      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        sD[threadIdx.x] = in[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();

      if (blockDim.x >= 1024) {
        if (threadIdx.x < 512) {
          sD[threadIdx.x] += sD[threadIdx.x + 512];
        }
        __syncthreads();
      }
      if (blockDim.x >= 512) {
        if (threadIdx.x < 256) {
          sD[threadIdx.x] += sD[threadIdx.x + 256];
        }
        __syncthreads();
      }
      if (blockDim.x >= 256) {
        if (threadIdx.x < 128) {
          sD[threadIdx.x] += sD[threadIdx.x + 128];
        }
        __syncthreads();
      }
      if (blockDim.x >= 128) {
        if (threadIdx.x < 64) {
          sD[threadIdx.x] += sD[threadIdx.x + 64];
        }
        __syncthreads();
      }
      if (blockDim.x >= 64) {
        if (threadIdx.x < 32) {
          sD[threadIdx.x] += sD[threadIdx.x + 32];
        }
        __syncthreads();
      }

      if (threadIdx.x < 32) warp_unroll(sD, threadIdx.x);
      __syncthreads();

      if (threadIdx.x == 0) out[blockIdx.x] = sD[0];  // write to global for block
    }
    //=======================

    __device__ void warp_unroll(hipDoubleComplex* smem, int tid) {
      hipDoubleComplex v = make_hipDoubleComplex(0, 0);
      v.x += smem[tid + 32].x;
      v.y += smem[tid + 32].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 16].x;
      v.y += smem[tid + 16].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 8].x;
      v.y += smem[tid + 8].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 4].x;
      v.y += smem[tid + 4].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 2].x;
      v.y += smem[tid + 2].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 1].x;
      v.y += smem[tid + 1].y;
      __syncwarp();
      smem[tid] = v;

      /* deprecated after volta, warp_unroll(volatile X*smem, int thidx);
      smem[thidx].x += smem[thidx + 32].x; smem[thidx].y += smem[thidx + 32].y;
      smem[thidx].x += smem[thidx + 16].x; smem[thidx].y += smem[thidx + 16].y;
      smem[thidx].x += smem[thidx + 8].x; smem[thidx].y += smem[thidx + 8].y;
      smem[thidx].x += smem[thidx + 4].x; smem[thidx].y += smem[thidx + 4].y;
      smem[thidx].x += smem[thidx + 2].x; smem[thidx].y += smem[thidx + 2].y;
      smem[thidx].x += smem[thidx + 1].x; smem[thidx].y += smem[thidx + 1].y;
      */
    }
    __global__ void cuReduce_kernel_cd(hipDoubleComplex* out, hipDoubleComplex* in,
                                       cytnx_uint64 Nelem) {
      __shared__ hipDoubleComplex sD[_TNinB_REDUCE_];  // allocate share mem for each thread
      sD[threadIdx.x] = make_hipDoubleComplex(0, 0);

      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        sD[threadIdx.x] = in[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();

      if (blockDim.x >= 1024) {
        if (threadIdx.x < 512) {
          sD[threadIdx.x].x += sD[threadIdx.x + 512].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 512].y;
        }
        __syncthreads();
      }
      if (blockDim.x >= 512) {
        if (threadIdx.x < 256) {
          sD[threadIdx.x].x += sD[threadIdx.x + 256].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 256].y;
        }
        __syncthreads();
      }
      if (blockDim.x >= 256) {
        if (threadIdx.x < 128) {
          sD[threadIdx.x].x += sD[threadIdx.x + 128].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 128].y;
        }
        __syncthreads();
      }
      if (blockDim.x >= 128) {
        if (threadIdx.x < 64) {
          sD[threadIdx.x].x += sD[threadIdx.x + 64].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 64].y;
        }
        __syncthreads();
      }

      if (threadIdx.x < 32) warp_unroll(sD, threadIdx.x);
      __syncthreads();

      if (threadIdx.x == 0) out[blockIdx.x] = sD[0];  // write to global for block
    }

    __device__ void warp_unroll(hipFloatComplex* smem, int tid) {
      hipFloatComplex v = make_hipFloatComplex(0, 0);
      v.x += smem[tid + 32].x;
      v.y += smem[tid + 32].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 16].x;
      v.y += smem[tid + 16].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 8].x;
      v.y += smem[tid + 8].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 4].x;
      v.y += smem[tid + 4].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 2].x;
      v.y += smem[tid + 2].y;
      __syncwarp();
      smem[tid] = v;
      v.x += smem[tid + 1].x;
      v.y += smem[tid + 1].y;
      __syncwarp();
      smem[tid] = v;

      /* deprecated after volta.
      smem[thidx].x += smem[thidx + 32].x; smem[thidx].y += smem[thidx + 32].y;
      smem[thidx].x += smem[thidx + 16].x; smem[thidx].y += smem[thidx + 16].y;
      smem[thidx].x += smem[thidx + 8].x; smem[thidx].y += smem[thidx + 8].y;
      smem[thidx].x += smem[thidx + 4].x; smem[thidx].y += smem[thidx + 4].y;
      smem[thidx].x += smem[thidx + 2].x; smem[thidx].y += smem[thidx + 2].y;
      smem[thidx].x += smem[thidx + 1].x; smem[thidx].y += smem[thidx + 1].y;
      */
    }
    __global__ void cuReduce_kernel_cf(hipFloatComplex* out, hipFloatComplex* in,
                                       cytnx_uint64 Nelem) {
      __shared__ hipFloatComplex sD[_TNinB_REDUCE_];  // allocate share mem for each thread
      sD[threadIdx.x] = make_hipFloatComplex(0, 0);

      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        sD[threadIdx.x] = in[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();

      if (blockDim.x >= 1024) {
        if (threadIdx.x < 512) {
          sD[threadIdx.x].x += sD[threadIdx.x + 512].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 512].y;
        }
        __syncthreads();
      }
      if (blockDim.x >= 512) {
        if (threadIdx.x < 256) {
          sD[threadIdx.x].x += sD[threadIdx.x + 256].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 256].y;
        }
        __syncthreads();
      }
      if (blockDim.x >= 256) {
        if (threadIdx.x < 128) {
          sD[threadIdx.x].x += sD[threadIdx.x + 128].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 128].y;
        }
        __syncthreads();
      }
      if (blockDim.x >= 128) {
        if (threadIdx.x < 64) {
          sD[threadIdx.x].x += sD[threadIdx.x + 64].x;
          sD[threadIdx.x].y += sD[threadIdx.x + 64].y;
        }
        __syncthreads();
      }

      if (threadIdx.x < 32) warp_unroll(sD, threadIdx.x);
      __syncthreads();

      if (threadIdx.x == 0) out[blockIdx.x] = sD[0];  // write to global for block
    }

    template <class T>
    void swap(T*& a, T*& b) {
      T* tmp = a;
      a = b;
      b = tmp;
    }

    template <class T>
    void cuReduce_gpu_generic(T* out, T* in, const cytnx_uint64& Nelem) {
      cytnx_uint64 Nelems = Nelem;
      cytnx_uint64 NBlocks;

      NBlocks = Nelems / _TNinB_REDUCE_;
      if (Nelems % _TNinB_REDUCE_) NBlocks += 1;

      // alloc mem for each block:
      T* dblk;
      // std::cout << NBlocks*sizeof(cytnx_double) << std::endl;
      checkCudaErrors(hipMalloc((void**)&dblk, NBlocks * sizeof(T)));
      if (NBlocks == 1) {
        cuReduce_kernel<<<NBlocks, _TNinB_REDUCE_>>>(out, in, Nelems);
      } else {
        cuReduce_kernel<<<NBlocks, _TNinB_REDUCE_>>>(dblk, in, Nelems);
      }
      Nelems = NBlocks;

      while (Nelems > 1) {
        NBlocks = Nelems / _TNinB_REDUCE_;
        if (Nelems % _TNinB_REDUCE_) NBlocks += 1;

        if (NBlocks == 1) {
          cuReduce_kernel<<<NBlocks, _TNinB_REDUCE_>>>(out, dblk, Nelems);
        } else {
          T* dblk2;
          hipMalloc((void**)&dblk2, NBlocks * sizeof(T));
          // do something:
          cuReduce_kernel<<<NBlocks, _TNinB_REDUCE_>>>(dblk2, dblk, Nelems);

          swap(dblk2, dblk);  // swap new data to old data, and free the old
          hipFree(dblk2);
        }
        Nelems = NBlocks;
      }
      hipFree(dblk);
    }

    void cuReduce_gpu_d(double* out, double* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }

    void cuReduce_gpu_f(float* out, float* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }

    void cuReduce_gpu_i64(cytnx_int64* out, cytnx_int64* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }
    void cuReduce_gpu_u64(cytnx_uint64* out, cytnx_uint64* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }

    void cuReduce_gpu_i32(cytnx_int32* out, cytnx_int32* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }
    void cuReduce_gpu_u32(cytnx_uint32* out, cytnx_uint32* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }
    void cuReduce_gpu_i16(cytnx_int16* out, cytnx_int16* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }
    void cuReduce_gpu_u16(cytnx_uint16* out, cytnx_uint16* in, const cytnx_uint64& Nelem) {
      cuReduce_gpu_generic(out, in, Nelem);
    }

    void cuReduce_gpu_cf(cytnx_complex64* out, cytnx_complex64* in, const cytnx_uint64& Nelem) {
      cytnx_uint64 Nelems = Nelem;
      cytnx_uint64 NBlocks;

      NBlocks = Nelems / _TNinB_REDUCE_;
      if (Nelems % _TNinB_REDUCE_) NBlocks += 1;

      // alloc mem for each block:
      hipFloatComplex* dblk;
      // std::cout << NBlocks*sizeof(cytnx_double) << std::endl;
      hipMalloc((void**)&dblk, NBlocks * sizeof(hipFloatComplex));

      if (NBlocks == 1) {
        cuReduce_kernel_cf<<<NBlocks, _TNinB_REDUCE_>>>((hipFloatComplex*)out, (hipFloatComplex*)in,
                                                        Nelems);
      } else {
        cuReduce_kernel_cf<<<NBlocks, _TNinB_REDUCE_>>>(dblk, (hipFloatComplex*)in, Nelems);
      }
      Nelems = NBlocks;

      while (Nelems > 1) {
        NBlocks = Nelems / _TNinB_REDUCE_;
        if (Nelems % _TNinB_REDUCE_) NBlocks += 1;

        if (NBlocks == 1) {
          cuReduce_kernel_cf<<<NBlocks, _TNinB_REDUCE_>>>((hipFloatComplex*)out, dblk, Nelems);
        } else {
          hipFloatComplex* dblk2;
          hipMalloc((void**)&dblk2, NBlocks * sizeof(hipFloatComplex));
          // do something:
          cuReduce_kernel_cf<<<NBlocks, _TNinB_REDUCE_>>>(dblk2, dblk, Nelems);

          swap(dblk2, dblk);  // swap new data to old data, and free the old
          hipFree(dblk2);
        }
        Nelems = NBlocks;
      }
      hipFree(dblk);
    }

    void cuReduce_gpu_cd(cytnx_complex128* out, cytnx_complex128* in, const cytnx_uint64& Nelem) {
      cytnx_uint64 Nelems = Nelem;
      cytnx_uint64 NBlocks;

      NBlocks = Nelems / _TNinB_REDUCE_;
      if (Nelems % _TNinB_REDUCE_) NBlocks += 1;

      // alloc mem for each block:
      hipDoubleComplex* dblk;
      // std::cout << NBlocks*sizeof(cytnx_double) << std::endl;
      hipMalloc((void**)&dblk, NBlocks * sizeof(hipDoubleComplex));

      if (NBlocks == 1) {
        cuReduce_kernel_cd<<<NBlocks, _TNinB_REDUCE_>>>((hipDoubleComplex*)out, (hipDoubleComplex*)in,
                                                        Nelems);
      } else {
        cuReduce_kernel_cd<<<NBlocks, _TNinB_REDUCE_>>>(dblk, (hipDoubleComplex*)in, Nelems);
      }
      Nelems = NBlocks;

      while (Nelems > 1) {
        NBlocks = Nelems / _TNinB_REDUCE_;
        if (Nelems % _TNinB_REDUCE_) NBlocks += 1;

        if (NBlocks == 1) {
          cuReduce_kernel_cd<<<NBlocks, _TNinB_REDUCE_>>>((hipDoubleComplex*)out, dblk, Nelems);
        } else {
          hipDoubleComplex* dblk2;
          hipMalloc((void**)&dblk2, NBlocks * sizeof(hipDoubleComplex));
          // do something:
          cuReduce_kernel_cd<<<NBlocks, _TNinB_REDUCE_>>>(dblk2, dblk, Nelems);

          swap(dblk2, dblk);  // swap new data to old data, and free the old
          hipFree(dblk2);
        }
        Nelems = NBlocks;
      }
      hipFree(dblk);
    }

  }  // namespace utils_internal

}  // namespace cytnx
