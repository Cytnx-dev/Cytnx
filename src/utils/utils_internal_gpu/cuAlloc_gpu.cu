#include "cuAlloc_gpu.hpp"

using namespace std;

namespace cytnx{
    namespace utils_internal{
    #ifdef UNI_GPU
        //void* Calloc_cpu(const cytnx_uint64 &N, const cytnx_uint64 &perelem_bytes){
        //    return calloc(M,perelem_bytes);
        //}
        void* cuMalloc_gpu(const cytnx_uint64 &bytes){
            void* ptr;
            checkCudaErrors(hipMallocManaged(&ptr,bytes));
            return ptr;
        }
    #endif
    }//utils_internal
}//cytnx
