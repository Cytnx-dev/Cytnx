#include "cuAlloc_gpu.hpp"

using namespace std;

namespace cytnx {
  namespace utils_internal {
#ifdef UNI_GPU
    // void* Calloc_cpu(const cytnx_uint64 &N, const cytnx_uint64 &perelem_bytes){
    //     return calloc(M,perelem_bytes);
    // }
    void* cuCalloc_gpu(const cytnx_uint64& N, const cytnx_uint64& perelem_bytes) {
      void* ptr;
      checkCudaErrors(hipMallocManaged((void**)&ptr, perelem_bytes * N));
      checkCudaErrors(hipMemset(ptr, 0, perelem_bytes * N));
      return ptr;
    }
    void* cuMalloc_gpu(const cytnx_uint64& bytes) {
      void* ptr;
      checkCudaErrors(hipMallocManaged(&ptr, bytes));
      return ptr;
    }
#endif
  }  // namespace utils_internal
}  // namespace cytnx
