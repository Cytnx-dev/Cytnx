#include "cuComplexmem_gpu.hpp"
#include "cuAlloc_gpu.hpp"
#include "Storage.hpp"
#ifdef UNI_OMP
#include <omp.h>
#endif

using namespace std;

namespace cytnx{
    namespace utils_internal{
    #ifdef UNI_GPU


        boost::intrusive_ptr<Storage_base> cuComplexmem_gpu_cdtd( void *out, void *in, const cytnx_uint64 &Nelem, const bool get_real){

            double* ddes = (double*)out->Mem;
            double* dsrc = (double*)in->Mem ; // we cast into double, so the Memcpy2D can get elem by stride.

            if(get_real){
                hipMemcpy2D(ddes,1*sizeof(cytnx_double),dsrc,2*sizeof(cytnx_double),sizeof(cytnx_double),Nelem,hipMemcpyDeviceToDevice);
            }else{
                hipMemcpy2D(ddes,1*sizeof(cytnx_double),dsrc+1,2*sizeof(cytnx_double),sizeof(cytnx_double),Nelem,hipMemcpyDeviceToDevice);
            }
        }

        boost::intrusive_ptr<Storage_base> cuComplexmem_gpu_cdtd( void *out, void *in, const cytnx_uint64 &Nelem, const bool get_real){
            
            float* ddes = (float*)out->Mem;
            float* dsrc = (float*)in->Mem ; // we cast into double, so the Memcpy2D can get elem by stride.

            if(get_real){
                hipMemcpy2D(ddes,1*sizeof(cytnx_float),dsrc,2*sizeof(cytnx_float),sizeof(cytnx_float),Nelem,hipMemcpyDeviceToDevice);
            }else{
                hipMemcpy2D(ddes,1*sizeof(cytnx_float),dsrc+1,2*sizeof(cytnx_float),sizeof(cytnx_float),Nelem,hipMemcpyDeviceToDevice);
            }
        }


    #endif // UNI_GPU
    }//namespace utils_internal
}//namespace cytnx
