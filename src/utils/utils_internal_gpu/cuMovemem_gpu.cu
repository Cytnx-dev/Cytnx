#include "hip/hip_runtime.h"
#include "utils/utils_internal_gpu/cuMovemem_gpu.hpp"
#include "utils/utils_internal_gpu/cuAlloc_gpu.hpp"
#include "Storage.hpp"
#ifdef UNI_OMP
#include <omp.h>
#endif

using namespace std;

namespace tor10{
    namespace utils_internal{
    #ifdef UNI_GPU
        template<class T>
        __global__ void cuMovemem_kernel(T* ddes, T*dsrc, tor10_uint64* accu_old, tor10_uint64* permuted_accu_new, tor10_uint32 rank, tor10_uint64 Nelem){
                extern __shared__ tor10_uint64 SHaccu[];

                tor10_uint64 ids;
                ///copy to share mem:
                if(rank<=blockDim.x){
                    if(threadIdx.x<rank){
                        SHaccu[threadIdx.x] = accu_old[threadIdx.x];
                        SHaccu[threadIdx.x+rank] = permuted_accu_new[threadIdx.x];
                    }
                }else{
                    tor10_uint32 Np=rank/blockDim.x;
                    if(rank%blockDim.x) Np+=1;
                    for(tor10_uint32 i=0;i<Np;i++){
                        ids = i*blockDim.x + threadIdx.x;
                        if(ids < rank){
                            SHaccu[ids] = accu_old[ids];
                            SHaccu[ids+rank] = permuted_accu_new[ids];
                        }
                    }
                }
                __syncthreads();

                tor10_uint64 tid = blockIdx.x*blockDim.x + threadIdx.x;
                ids = 0;
                for(tor10_uint32 i=0;i<rank;i++){
                    ids += (tid/SHaccu[i])*SHaccu[rank+i];
                    tid = tid%SHaccu[i];
                }
                if(blockIdx.x*blockDim.x+threadIdx.x<Nelem) ddes[ids] = dsrc[blockIdx.x*blockDim.x+threadIdx.x];

        }
        
        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_cd(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.ComplexDouble,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type ComplexDouble",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            hipDoubleComplex *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (hipDoubleComplex*)cuMalloc_gpu(sizeof(hipDoubleComplex)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(hipDoubleComplex*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new ComplexDoubleStorage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(tor10_complex128)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }

        }

        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_cf(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.ComplexFloat,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type ComplexFloat",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            hipFloatComplex *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (hipFloatComplex*)cuMalloc_gpu(sizeof(hipFloatComplex)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(hipFloatComplex*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new ComplexFloatStorage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(tor10_complex64)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }

        }
        
        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_d(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.Double,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type Double",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            double *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (double*)cuMalloc_gpu(sizeof(double)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(double*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new DoubleStorage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(double)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }
        }
        
        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_f(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.Float,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type Float",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            float *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (float*)cuMalloc_gpu(sizeof(float)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(float*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new FloatStorage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(float)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }

        }

        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i64(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.Int64,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type Int64",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            tor10_int64 *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (tor10_int64*)cuMalloc_gpu(sizeof(tor10_int64)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(tor10_int64*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new Int64Storage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(tor10_int64)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }


       }

        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u64(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.Uint64,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type Uint64",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            tor10_uint64 *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (tor10_uint64*)cuMalloc_gpu(sizeof(tor10_uint64)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(tor10_uint64*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new Uint64Storage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(tor10_uint64)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }
        }

        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i32(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.Int32,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type Int32",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            tor10_int32 *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (tor10_int32*)cuMalloc_gpu(sizeof(tor10_int32)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(tor10_int32*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new Int32Storage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(tor10_int32)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }
        }

        boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u32(boost::intrusive_ptr<Storage_base> &in, const std::vector<tor10_uint64> &old_shape, const std::vector<tor10_uint64>&mapper, const std::vector<tor10_uint64> &invmapper, const bool is_inplace){
            #ifdef UNI_DEBUG
            tor10_error_msg(in->dtype != tor10type.Uint32,"[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type Uint32",in->dtype_str().c_str());
            tor10_error_msg(in->device == tor10device.cpu,"%s", "[DEBUG][internal error] in.device is on cpu but all cuda function.");
            #endif

            

            std::vector<tor10_uint64> newshape(old_shape.size());
            for(tor10_uint64 i=0;i<old_shape.size();i++)
                newshape[i] = old_shape[mapper[i]];

            std::vector<tor10_uint64> shifter_old(old_shape.size());
            std::vector<tor10_uint64> shifter_new(old_shape.size());

            tor10_uint64 accu_old=1,accu_new=1;
            for(tor10_int64 i=old_shape.size()-1;i>=0;i--){
                shifter_old[i] = accu_old;
                shifter_new[i] = accu_new;
                accu_old*=old_shape[i];
                accu_new*=newshape[i];
            }
            std::vector<tor10_uint64> old_inds(old_shape.size());

            std::vector<tor10_uint64> permuted_shifter_new(old_shape.size());
            for(unsigned int i=0;i<old_shape.size();i++)
                permuted_shifter_new[i] = shifter_new[invmapper[i]];

            ///allocate a GPU for psn-vec/so-vec/tmp des-vec
            tor10_uint64 *dshifter_old, *dperm_shifter_new;
            tor10_uint32 *dtmp;
            tor10_uint64 Nelem = accu_old;        

            hipSetDevice(in->device); // ensure the following allocation on the same device as src.
            checkCudaErrors(hipMalloc((void**)&dshifter_old, sizeof(tor10_uint64)*shifter_old.size()));
            checkCudaErrors(hipMalloc((void**)&dperm_shifter_new, sizeof(tor10_uint64)*permuted_shifter_new.size()));
            dtmp = (tor10_uint32*)cuMalloc_gpu(sizeof(tor10_uint32)*Nelem); 

            /// copy psn-vec/so-vec to device
            checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0], sizeof(tor10_uint64)*permuted_shifter_new.size(),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0], sizeof(tor10_uint64)*shifter_old.size(),hipMemcpyHostToDevice));


            ///calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
            tor10_uint64 NBlocks = Nelem/256;
            if(Nelem%256){
                NBlocks+=1;
            }
            cuMovemem_kernel<<< NBlocks,256,shifter_old.size()*2*sizeof(tor10_uint64) >>>(dtmp,(tor10_uint32*)in->Mem,dshifter_old,dperm_shifter_new,old_shape.size(),Nelem);


            ///house keeping:
            checkCudaErrors(hipFree(dshifter_old));
            checkCudaErrors(hipFree(dperm_shifter_new));

            boost::intrusive_ptr<Storage_base> out(new Uint32Storage());
            if(is_inplace){

                ///cpy back:
                checkCudaErrors(hipMemcpy(in->Mem,dtmp, sizeof(tor10_uint32)*Nelem,hipMemcpyDeviceToDevice));
                hipFree(dtmp);
                return out;

            }else{

                out->_Init_byptr(dtmp,Nelem);
                return out;
            }
        }

    #endif // UNI_GPU
    }//namespace utils_internal
}//namespace tor10
