#include "hip/hip_runtime.h"
#include "cuMovemem_gpu.hpp"
#include "cuAlloc_gpu.hpp"
#include "Storage.hpp"
#include <algorithm>
#include "utils/vec_print.hpp"

#ifdef UNI_GPU
  #ifdef UNI_CUTT
    #include "cutt.h"
  #endif

  #ifdef UNI_CUTENSOR
    #include "hiptensor.h"
  #endif

#endif

using namespace std;

namespace cytnx {
  namespace utils_internal {

#ifdef UNI_GPU
    template <class BidirectionalIterator>
    void reverse_perm(BidirectionalIterator first, BidirectionalIterator last, int N) {
      while ((first != last) && (first != --last)) {
        *first = (N - 1) - *first;
        *last = (N - 1) - *last;
        std::iter_swap(first, last);
        ++first;
      }
      if (N % 2) *first = (N - 1) - *first;
    }

    template <class T>
    __global__ void cuMovemem_kernel(T *ddes, T *dsrc, cytnx_uint64 *accu_old,
                                     cytnx_uint64 *permuted_accu_new, cytnx_uint32 rank,
                                     cytnx_uint64 Nelem) {
      extern __shared__ cytnx_uint64 SHaccu[];

      cytnx_uint64 ids;
      /// copy to share mem:
      if (rank <= blockDim.x) {
        if (threadIdx.x < rank) {
          SHaccu[threadIdx.x] = accu_old[threadIdx.x];
          SHaccu[threadIdx.x + rank] = permuted_accu_new[threadIdx.x];
        }
      } else {
        cytnx_uint32 Np = rank / blockDim.x;
        if (rank % blockDim.x) Np += 1;
        for (cytnx_uint32 i = 0; i < Np; i++) {
          ids = i * blockDim.x + threadIdx.x;
          if (ids < rank) {
            SHaccu[ids] = accu_old[ids];
            SHaccu[ids + rank] = permuted_accu_new[ids];
          }
        }
      }
      __syncthreads();

      cytnx_uint64 tid = blockIdx.x * blockDim.x + threadIdx.x;
      ids = 0;
      for (cytnx_uint32 i = 0; i < rank; i++) {
        ids += (tid / SHaccu[i]) * SHaccu[rank + i];
        tid = tid % SHaccu[i];
      }
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem)
        ddes[ids] = dsrc[blockIdx.x * blockDim.x + threadIdx.x];
    }

    // T is the cytnx type, cuT is the cuda type. For all types they should be the same except for
    // hipDoubleComplex and hipFloatComplex.
    template <class T, class cuT>
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_general(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
  #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
  #endif

      std::vector<cytnx_uint64> newshape(old_shape.size());
      for (cytnx_uint64 i = 0; i < old_shape.size(); i++) newshape[i] = old_shape[mapper[i]];

      std::vector<cytnx_uint64> shifter_old(old_shape.size());
      std::vector<cytnx_uint64> shifter_new(old_shape.size());

      cytnx_uint64 accu_old = 1, accu_new = 1;
      for (cytnx_int64 i = old_shape.size() - 1; i >= 0; i--) {
        shifter_old[i] = accu_old;
        shifter_new[i] = accu_new;
        accu_old *= old_shape[i];
        accu_new *= newshape[i];
      }
      std::vector<cytnx_uint64> old_inds(old_shape.size());

      std::vector<cytnx_uint64> permuted_shifter_new(old_shape.size());
      for (unsigned int i = 0; i < old_shape.size(); i++)
        permuted_shifter_new[i] = shifter_new[invmapper[i]];

      /// allocate a GPU for psn-vec/so-vec/tmp des-vec
      cytnx_uint64 *dshifter_old, *dperm_shifter_new;
      cuT *dtmp;
      cytnx_uint64 Nelem = accu_old;

      hipSetDevice(in->device);  // ensure the following allocation on the same device as src.
      checkCudaErrors(
        hipMalloc((void **)&dshifter_old, sizeof(cytnx_uint64) * shifter_old.size()));
      checkCudaErrors(hipMalloc((void **)&dperm_shifter_new,
                                 sizeof(cytnx_uint64) * permuted_shifter_new.size()));
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->cap);

      /// copy psn-vec/so-vec to device
      checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0],
                                 sizeof(cytnx_uint64) * permuted_shifter_new.size(),
                                 hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0],
                                 sizeof(cytnx_uint64) * shifter_old.size(),
                                 hipMemcpyHostToDevice));

      /// calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
      cytnx_uint64 NBlocks = Nelem / 256;
      if (Nelem % 256) {
        NBlocks += 1;
      }
      cuMovemem_kernel<<<NBlocks, 256, shifter_old.size() * 2 * sizeof(cytnx_uint64)>>>(
        dtmp, (cuT *)in->Mem, dshifter_old, dperm_shifter_new, old_shape.size(), Nelem);

      /// house keeping:
      checkCudaErrors(hipFree(dshifter_old));
      checkCudaErrors(hipFree(dperm_shifter_new));

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->Mem, dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device, true, in->cap);
        return out;
      }
    }

  #ifdef UNI_CUTT
    template <class T, class cuT>
    boost::intrusive_ptr<Storage_base> cuMovemem_cutt_gpu(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
    #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_cutt with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
    #endif

      cuT *dtmp;
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->cap);
      cytnx_uint64 Nelem = in->len;

      std::vector<int> perm(mapper.begin(), mapper.end());
      std::vector<int> size(old_shape.begin(), old_shape.end());
      std::reverse(size.begin(), size.end());  // matching API CUTT
      reverse_perm(perm.begin(), perm.end(), perm.size());  // matching API CUTT

      cuttHandle plan;
      cuttPlan(&plan, perm.size(), size.data(), perm.data(), sizeof(cuT), 0);
      cuttExecute(plan, in->Mem, dtmp);

      cuttDestroy(plan);

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->Mem, dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device, true, in->cap);
        return out;
      }
    }
  #endif

  #ifdef UNI_CUTENSOR
    template <class T, class cuT>  // T: cpu type, cuT: gpu type, cutnT: cntensor type
    boost::intrusive_ptr<Storage_base> cuMovemem_cutensor_gpu(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace, hipDataType type_in, hipDataType type_out,
      hipDataType type_one, const cuT &ONE) {
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
    #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_cutt with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
    #endif

      cuT *dtmp;
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->cap);
      cytnx_uint64 Nelem = in->len;

      std::vector<int> perm(mapper.begin(), mapper.end());
      std::vector<int64_t> size(old_shape.begin(), old_shape.end());
      std::vector<int> ori(perm.size());
      for (int i = 0; i < ori.size(); i++) ori[i] = i;

      std::vector<int64_t> new_size(perm.size());
      for (int i = 0; i < new_size.size(); i++) {
        new_size[i] = size[perm[i]];
      }
      std::reverse(size.begin(), size.end());  // matching API
      std::reverse(perm.begin(), perm.end());  // matching API
      std::reverse(new_size.begin(), new_size.end());  // matching API
      std::reverse(ori.begin(), ori.end());  // matching API

      hiptensorHandle_t handle;
      checkCudaErrors(cutensorInit(&handle));

      hiptensorTensorDescriptor_t descA;
      checkCudaErrors(hiptensorInitTensorDescriptor(&handle, &descA, size.size(), size.data(),
                                                   NULL /* stride */, type_in,
                                                   HIPTENSOR_OP_IDENTITY));

      hiptensorTensorDescriptor_t descC;
      checkCudaErrors(hiptensorInitTensorDescriptor(&handle, &descC, new_size.size(),
                                                   new_size.data(), NULL /* stride */, type_out,
                                                   HIPTENSOR_OP_IDENTITY));

      checkCudaErrors(hiptensorPermutation(&handle, &ONE, (cuT *)in->Mem, &descA, ori.data(), dtmp,
                                          &descC, perm.data(), type_one, 0 /* stream */));

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->Mem, dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device, true, in->cap);
        return out;
      }
    }
  #endif

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_cd(boost::intrusive_ptr<Storage_base> &in,
                                                        const std::vector<cytnx_uint64> &old_shape,
                                                        const std::vector<cytnx_uint64> &mapper,
                                                        const std::vector<cytnx_uint64> &invmapper,
                                                        const bool is_inplace) {
  #ifdef UNI_CUTENSOR
      return cuMovemem_cutensor_gpu<cytnx_complex128, hipDoubleComplex>(
        in, old_shape, mapper, invmapper, is_inplace, HIP_C_64F, HIP_C_64F, HIP_C_64F,
        make_hipDoubleComplex(1, 0));
  #else
      return cuMovemem_gpu_general<cytnx_complex128, hipDoubleComplex>(in, old_shape, mapper,
                                                                      invmapper, is_inplace);
  #endif
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_cf(boost::intrusive_ptr<Storage_base> &in,
                                                        const std::vector<cytnx_uint64> &old_shape,
                                                        const std::vector<cytnx_uint64> &mapper,
                                                        const std::vector<cytnx_uint64> &invmapper,
                                                        const bool is_inplace) {
  #if defined(UNO_CUTENSOR)
      return cuMovemem_cutensor_gpu<cytnx_complex64, hipFloatComplex>(
        in, old_shape, mapper, invmapper, is_inplace, HIP_C_32F, HIP_C_32F, HIP_C_32F,
        make_hipFloatComplex(1, 0));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_complex64, hipFloatComplex>(in, old_shape, mapper, invmapper,
                                                                 is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_complex64, hipFloatComplex>(in, old_shape, mapper,
                                                                    invmapper, is_inplace);
  #endif
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_d(boost::intrusive_ptr<Storage_base> &in,
                                                       const std::vector<cytnx_uint64> &old_shape,
                                                       const std::vector<cytnx_uint64> &mapper,
                                                       const std::vector<cytnx_uint64> &invmapper,
                                                       const bool is_inplace) {
  #if defined(UNI_CUTENSOR)
      return cuMovemem_cutensor_gpu<double, double>(in, old_shape, mapper, invmapper, is_inplace,
                                                    HIP_R_64F, HIP_R_64F, HIP_R_64F, double(1));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_double, cytnx_double>(in, old_shape, mapper, invmapper,
                                                            is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_double, cytnx_double>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
  #endif
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_f(boost::intrusive_ptr<Storage_base> &in,
                                                       const std::vector<cytnx_uint64> &old_shape,
                                                       const std::vector<cytnx_uint64> &mapper,
                                                       const std::vector<cytnx_uint64> &invmapper,
                                                       const bool is_inplace) {
  #if defined(UNI_CUTENSOR)
      return cuMovemem_cutensor_gpu<float, float>(in, old_shape, mapper, invmapper, is_inplace,
                                                  HIP_R_32F, HIP_R_32F, HIP_R_32F, float(1));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_float, cytnx_float>(in, old_shape, mapper, invmapper,
                                                          is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_float, cytnx_float>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
  #endif
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i64(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_int64, cytnx_int64>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u64(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_uint64, cytnx_uint64>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i32(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_int32, cytnx_int32>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u32(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_uint32, cytnx_uint32>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u16(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_uint16, cytnx_uint16>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i16(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_int16, cytnx_int16>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_b(boost::intrusive_ptr<Storage_base> &in,
                                                       const std::vector<cytnx_uint64> &old_shape,
                                                       const std::vector<cytnx_uint64> &mapper,
                                                       const std::vector<cytnx_uint64> &invmapper,
                                                       const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_bool, cytnx_bool>(in, old_shape, mapper, invmapper,
                                                           is_inplace);
    }

#endif  // UNI_GPU
  }  // namespace utils_internal
}  // namespace cytnx
