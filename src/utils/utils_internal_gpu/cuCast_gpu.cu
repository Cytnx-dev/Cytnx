#include "hip/hip_runtime.h"
#include "utils/utils_internal_gpu/cuCast_gpu.hpp"
#include "Storage.hpp"
#ifdef UNI_OMP
#include <omp.h>
#endif

using namespace std;
namespace cytnx{
    namespace utils_internal{

        cuCast_gpu_interface::cuCast_gpu_interface(){
            UElemCast_gpu = vector<vector<ElemCast_io_gpu> >(N_Type,vector<ElemCast_io_gpu>(N_Type,NULL));

            UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.ComplexDouble] = cuCast_gpu_cdtcd;
            UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.ComplexFloat ] = cuCast_gpu_cdtcf;
            //UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.Double       ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.Float        ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.Int64        ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.Uint64       ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.Int32        ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexDouble][cytnxtype.Uint32       ] = cuCast_gpu_invalid;

            UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.ComplexDouble] = cuCast_gpu_cftcd;
            UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.ComplexFloat ] = cuCast_gpu_cftcf;
            //UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.Double       ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.Float        ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.Int64        ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.Uint64       ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.Int32        ] = cuCast_gpu_invalid;
            //UElemCast_gpu[cytnxtype.ComplexFloat][cytnxtype.Uint32       ] = cuCast_gpu_invalid;

            UElemCast_gpu[cytnxtype.Double][cytnxtype.ComplexDouble] = cuCast_gpu_dtcd;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.ComplexFloat ] = cuCast_gpu_dtcf;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.Double       ] = cuCast_gpu_dtd;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.Float        ] = cuCast_gpu_dtf;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.Int64        ] = cuCast_gpu_dti64;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.Uint64       ] = cuCast_gpu_dtu64;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.Int32        ] = cuCast_gpu_dti32;
            UElemCast_gpu[cytnxtype.Double][cytnxtype.Uint32       ] = cuCast_gpu_dtu32;

            UElemCast_gpu[cytnxtype.Float][cytnxtype.ComplexDouble] = cuCast_gpu_ftcd;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.ComplexFloat ] = cuCast_gpu_ftcf;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.Double       ] = cuCast_gpu_ftd;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.Float        ] = cuCast_gpu_ftf;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.Int64        ] = cuCast_gpu_fti64;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.Uint64       ] = cuCast_gpu_ftu64;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.Int32        ] = cuCast_gpu_fti32;
            UElemCast_gpu[cytnxtype.Float][cytnxtype.Uint32       ] = cuCast_gpu_ftu32;

            UElemCast_gpu[cytnxtype.Int64][cytnxtype.ComplexDouble] = cuCast_gpu_i64tcd;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.ComplexFloat ] = cuCast_gpu_i64tcf;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.Double       ] = cuCast_gpu_i64td;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.Float        ] = cuCast_gpu_i64tf;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.Int64        ] = cuCast_gpu_i64ti64;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.Uint64       ] = cuCast_gpu_i64tu64;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.Int32        ] = cuCast_gpu_i64ti32;
            UElemCast_gpu[cytnxtype.Int64][cytnxtype.Uint32       ] = cuCast_gpu_i64tu32;

            UElemCast_gpu[cytnxtype.Uint64][cytnxtype.ComplexDouble] = cuCast_gpu_u64tcd;
            UElemCast_gpu[cytnxtype.Uint64][cytnxtype.ComplexFloat ] = cuCast_gpu_u64tcf;
            UElemCast_gpu[cytnxtype.Uint64][cytnxtype.Double       ] = cuCast_gpu_u64td;
        }
        utils_internal::cuCast_gpu_interface cuCast_gpu; // interface object. 

        //=======================================================================

        __global__ void cuCastElem_kernel_cd2cf(const hipDoubleComplex *src, hipFloatComplex *des, cytnx_uint64 Nelem){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                des[blockIdx.x*blockDim.x + threadIdx.x] = hipComplexDoubleToFloat(src[blockIdx.x*blockDim.x + threadIdx.x]);
            }
        }

        __global__ void cuCastElem_kernel_cf2cd(const hipFloatComplex *src, hipDoubleComplex *des, cytnx_uint64 Nelem){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                des[blockIdx.x*blockDim.x + threadIdx.x] = hipComplexFloatToDouble(src[blockIdx.x*blockDim.x + threadIdx.x]);
            }
        }
        
        template<class T>
        __global__ void cuCastElem_kernel_r2cf(const T *src, hipFloatComplex *des, cytnx_uint64 Nelem){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                des[blockIdx.x*blockDim.x + threadIdx.x].x = src[blockIdx.x*blockDim.x + threadIdx.x];
            }
        }
        template<class T2>
        __global__ void cuCastElem_kernel_r2cd(const T2 *src, hipDoubleComplex *des, cytnx_uint64 Nelem){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                des[blockIdx.x*blockDim.x + threadIdx.x].x = src[blockIdx.x*blockDim.x + threadIdx.x];
            }
        }

        template<class T3,class T4>
        __global__ void cuCastElem_kernel_r2r(const T3 *src, T4 *des, cytnx_uint64 Nelem){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                des[blockIdx.x*blockDim.x + threadIdx.x] = src[blockIdx.x*blockDim.x + threadIdx.x];
            }
        }

        //========================================================================
        void cuCast_gpu_cdtcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_complex128)*len_in,hipMemcpyDeviceToDevice)); 
        }

        void cuCast_gpu_cdtcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }

            hipDoubleComplex* _in = static_cast<hipDoubleComplex*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_cd2cf<<<NBlocks,512>>>(_in,_out,len_in);

        }

        void cuCast_gpu_cftcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            hipFloatComplex* _in = static_cast<hipFloatComplex*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);
            
            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_cf2cd<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_cftcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_complex64)*len_in,hipMemcpyDeviceToDevice)); 
        }


        void cuCast_gpu_dtcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){

            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cd<<<NBlocks,512>>>(_in,_out,len_in);

        }

        void cuCast_gpu_dtcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());    
                out->Init(len_in,alloc_device);
            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cf<<<NBlocks,512>>>(_in,_out,len_in);

        }

        void cuCast_gpu_dtd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){       
                out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_double)*len_in,hipMemcpyDeviceToDevice)); 

        }
        void cuCast_gpu_dtf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
                out -> Init(len_in);
            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            cytnx_float*  _out= static_cast<cytnx_float*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_dti64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
                out-> Init(len_in);
            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            cytnx_int64*  _out= static_cast<cytnx_int64*>(out->Mem);
            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_dtu64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            cytnx_uint64*  _out= static_cast<cytnx_uint64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_dti32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
                out->Init(len_in,alloc_device);

            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            cytnx_int32*  _out= static_cast<cytnx_int32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_dtu32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
            cytnx_uint32*  _out= static_cast<cytnx_uint32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);

        }

        void cuCast_gpu_ftcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cd<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_ftcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cf<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_ftd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            cytnx_double*  _out= static_cast<cytnx_double*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_ftf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_float)*len_in,hipMemcpyDeviceToDevice)); 
        }
        void cuCast_gpu_fti64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            cytnx_int64*  _out= static_cast<cytnx_int64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_ftu64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            cytnx_uint64*  _out= static_cast<cytnx_uint64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_fti32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            cytnx_int32*  _out= static_cast<cytnx_int32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_ftu32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
            cytnx_uint32*  _out= static_cast<cytnx_uint32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_i64tcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cd<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i64tcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cf<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_i64td(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            cytnx_double*  _out= static_cast<cytnx_double*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_i64tf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            cytnx_float*  _out= static_cast<cytnx_float*>(out->Mem);


            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_i64ti64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_int64)*len_in,hipMemcpyDeviceToDevice)); 

        }
        void cuCast_gpu_i64tu64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            cytnx_uint64*  _out= static_cast<cytnx_uint64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i64ti32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            cytnx_int32*  _out= static_cast<cytnx_int32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i64tu32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
            cytnx_uint32*  _out= static_cast<cytnx_uint32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_u64tcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cd<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_u64tcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cf<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_u64td(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            cytnx_double*  _out= static_cast<cytnx_double*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u64tf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            cytnx_float*  _out= static_cast<cytnx_float*>(out->Mem);


            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u64ti64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            cytnx_int64*  _out= static_cast<cytnx_int64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u64tu64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_uint64)*len_in,hipMemcpyDeviceToDevice)); 
           
        }
        void cuCast_gpu_u64ti32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            cytnx_int32*  _out= static_cast<cytnx_int32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u64tu32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
            cytnx_uint32*  _out= static_cast<cytnx_uint32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_i32tcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cd<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i32tcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cf<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i32td(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            cytnx_double*  _out= static_cast<cytnx_double*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i32tf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            cytnx_float*  _out= static_cast<cytnx_float*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i32ti64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            cytnx_int64*  _out= static_cast<cytnx_int64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i32tu64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            cytnx_uint64*  _out= static_cast<cytnx_uint64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_i32ti32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_int32)*len_in,hipMemcpyDeviceToDevice)); 
        }
        void cuCast_gpu_i32tu32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
            cytnx_uint32*  _out= static_cast<cytnx_uint32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }

        void cuCast_gpu_u32tcd(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            hipDoubleComplex*  _out= static_cast<hipDoubleComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cd<<<NBlocks,512>>>(_in,_out,len_in);

        }
        void cuCast_gpu_u32tcf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            hipFloatComplex*  _out= static_cast<hipFloatComplex*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2cf<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u32td(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            cytnx_double*  _out= static_cast<cytnx_double*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u32tf(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            cytnx_float*  _out= static_cast<cytnx_float*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u32ti64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            cytnx_int64*  _out= static_cast<cytnx_int64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u32tu64(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            cytnx_uint64*  _out= static_cast<cytnx_uint64*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u32ti32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
                out->Init(len_in,alloc_device);
            }
            cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
            cytnx_int32*  _out= static_cast<cytnx_int32*>(out->Mem);

            cytnx_uint64 NBlocks = len_in/512;
            if(len_in%512) NBlocks+=1;
            cuCastElem_kernel_r2r<<<NBlocks,512>>>(_in,_out,len_in);
        }
        void cuCast_gpu_u32tu32(const boost::intrusive_ptr<Storage_base>& in, boost::intrusive_ptr<Storage_base>& out, const unsigned long long &len_in, const int &alloc_device){
            if(alloc_device>=0){
                out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
                out->Init(len_in,alloc_device);
            }
            checkCudaErrors(hipMemcpy(out->Mem,in->Mem,sizeof(cytnx_uint32)*len_in,hipMemcpyDeviceToDevice)); 

        }
    }//namespace utils_internal
}//namespace cytnx
