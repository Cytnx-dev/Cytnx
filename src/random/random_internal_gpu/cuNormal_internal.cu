#include "cuNormal_internal.hpp"

namespace cytnx {
  namespace random_internal {

    void cuRng_normal_cd(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->Mem);

      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormalDouble(gen, rptr, in->len * 2, a, b);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_normal_cf(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->Mem);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormal(gen, rptr, in->len * 2, a, b);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_normal_d(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                        const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->Mem);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormalDouble(gen, rptr, in->len, a, b);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_normal_f(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                        const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->Mem);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormal(gen, rptr, in->len, a, b);

      hiprandDestroyGenerator(gen);
    }

  }  // namespace random_internal
}  // namespace cytnx
