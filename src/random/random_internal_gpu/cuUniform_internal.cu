#include "cuUniform_internal.hpp"

namespace cytnx {
  namespace random_internal {

    void cuRng_uniform_cd(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                          const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->Mem);

      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniformDouble(gen, rptr, in->len * 2);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_uniform_cf(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                          const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->Mem);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniform(gen, rptr, in->len * 2);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_uniform_d(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->Mem);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniformDouble(gen, rptr, in->len);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_uniform_f(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->Mem);
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniform(gen, rptr, in->len);

      hiprandDestroyGenerator(gen);
    }

  }  // namespace random_internal
}  // namespace cytnx
