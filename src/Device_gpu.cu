#include "Device.hpp"
#include "cytnx_error.hpp"

using namespace std;
namespace cytnx {

#ifdef UNI_GPU
  void Device_class::hipDeviceSynchronize() { hipDeviceSynchronize(); }
#else
  // See Device.cpp
#endif

}  // namespace cytnx
