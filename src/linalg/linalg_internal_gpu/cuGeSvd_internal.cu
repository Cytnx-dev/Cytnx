#include "hip/hip_runtime.h"
#include "linalg/linalg_internal_gpu/cuGeSvd_internal.hpp"

namespace cytnx {

  namespace linalg_internal {

    /// cuGeSvd
    void cuGeSvd_internal_cd(const boost::intrusive_ptr<Storage_base> &in,
                             boost::intrusive_ptr<Storage_base> &U,
                             boost::intrusive_ptr<Storage_base> &vT,
                             boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                             const cytnx_int64 &N) {
      assert(sizeof(hipDoubleComplex) == sizeof(cytnx_complex128));
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      hipDoubleComplex *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(hipDoubleComplex)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(hipDoubleComplex) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 max = std::max(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      if (N < M) {
        ldA = M, ldu = M, ldvT = min;
      }
      cytnx_int32 lwork = 0;

      void *UMem, *vTMem;
      if (U->Mem) {
        UMem = U->Mem;
      } else {
        if (jobu == 'S') checkCudaErrors(hipMalloc(&UMem, max * max * sizeof(hipDoubleComplex)));
      }
      if (vT->Mem) {
        vTMem = vT->Mem;
      } else {
        if (jobv == 'S') checkCudaErrors(hipMalloc(&vTMem, max * max * sizeof(hipDoubleComplex)));
      }

      // query working space :
      checkCudaErrors(hipsolverDnZgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      hipDoubleComplex *work;
      cytnx_double *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(hipDoubleComplex)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_double64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      if (N >= M) {
        checkCudaErrors(hipsolverDnZgesvd(cusolverH, jobv, jobu, N, M, (hipDoubleComplex *)Mij, ldA,
                                         (cytnx_double *)S->Mem, (hipDoubleComplex *)vTMem, ldu,
                                         (hipDoubleComplex *)UMem, ldvT, work, lwork, rwork,
                                         devinfo));
      } else {
        checkCudaErrors(hipsolverDnZgesvd(cusolverH, jobu, jobv, M, N, (hipDoubleComplex *)Mij, ldA,
                                         (cytnx_double *)S->Mem, (hipDoubleComplex *)UMem, ldu,
                                         (hipDoubleComplex *)vTMem, ldvT, work, lwork, rwork,
                                         devinfo));
        if (U->dtype != Type.Void)
          U->Move_memory_({(cytnx_uint64)min, (cytnx_uint64)M}, {1, 0}, {1, 0});
        // linalg_internal::cuConj_inplace_internal_cd(U,M*min);
        if (vT->dtype != Type.Void)
          vT->Move_memory_({(cytnx_uint64)N, (cytnx_uint64)min}, {1, 0}, {1, 0});
        linalg_internal::cuConj_inplace_internal_cd(vT, N * min);
      }
      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));
      cytnx_error_msg(
        info != 0, "%s %d %s", "Error in cuBlas function 'hipsolverDnZgesvd': cuBlas INFO = ", info,
        "If info>0, possibly svd not converge, if info<0, see cusolver manual for more info.");

      checkCudaErrors(hipFree(work));
      checkCudaErrors(hipFree(Mij));
      if (UMem != nullptr and U->dtype == Type.Void) {
        checkCudaErrors(hipFree(UMem));
      }
      if (vTMem != nullptr and vT->dtype == Type.Void) {
        checkCudaErrors(hipFree(vTMem));
      }
      checkCudaErrors(hipFree(devinfo));
      checkCudaErrors(hipsolverDnDestroy(cusolverH));
    }
    void cuGeSvd_internal_cf(const boost::intrusive_ptr<Storage_base> &in,
                             boost::intrusive_ptr<Storage_base> &U,
                             boost::intrusive_ptr<Storage_base> &vT,
                             boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                             const cytnx_int64 &N) {
      assert(sizeof(hipFloatComplex) == sizeof(cytnx_complex64));
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      hipFloatComplex *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(hipFloatComplex)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_complex64) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 max = std::max(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      if (N < M) {
        ldA = M, ldu = M, ldvT = min;
      }
      cytnx_int32 lwork = 0;

      void *UMem, *vTMem;
      if (U->Mem) {
        UMem = U->Mem;
      } else {
        if (jobu == 'S') checkCudaErrors(hipMalloc(&UMem, max * max * sizeof(hipFloatComplex)));
      }
      if (vT->Mem) {
        vTMem = vT->Mem;
      } else {
        if (jobv == 'S') checkCudaErrors(hipMalloc(&vTMem, max * max * sizeof(hipFloatComplex)));
      }

      // query working space :
      checkCudaErrors(hipsolverDnCgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      hipFloatComplex *work;
      cytnx_float *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(hipFloatComplex)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_float64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      if (N >= M) {
        checkCudaErrors(hipsolverDnCgesvd(
          cusolverH, jobv, jobu, N, M, (hipFloatComplex *)Mij, ldA, (cytnx_float *)S->Mem,
          (hipFloatComplex *)vTMem, ldu, (hipFloatComplex *)UMem, ldvT, work, lwork, rwork, devinfo));
      } else {
        checkCudaErrors(hipsolverDnCgesvd(
          cusolverH, jobu, jobv, M, N, (hipFloatComplex *)Mij, ldA, (cytnx_float *)S->Mem,
          (hipFloatComplex *)UMem, ldu, (hipFloatComplex *)vTMem, ldvT, work, lwork, rwork, devinfo));
        if (U->dtype != Type.Void)
          U->Move_memory_({(cytnx_uint64)min, (cytnx_uint64)M}, {1, 0}, {1, 0});
        // linalg_internal::cuConj_inplace_internal_cf(U,M*min);
        if (vT->dtype != Type.Void)
          vT->Move_memory_({(cytnx_uint64)N, (cytnx_uint64)min}, {1, 0}, {1, 0});
        linalg_internal::cuConj_inplace_internal_cf(vT, N * min);
      }

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));
      cytnx_error_msg(
        info != 0, "%s %d %s", "Error in cuBlas function 'hipsolverDnCgesvd': cuBlas INFO = ", info,
        "If info>0, possibly svd not converge, if info<0, see cusolver manual for more info.");

      checkCudaErrors(hipFree(work));
      checkCudaErrors(hipFree(Mij));
      if (UMem != nullptr and U->dtype == Type.Void) {
        checkCudaErrors(hipFree(UMem));
      }
      if (vTMem != nullptr and vT->dtype == Type.Void) {
        checkCudaErrors(hipFree(vTMem));
      }
      checkCudaErrors(hipFree(devinfo));
      checkCudaErrors(hipsolverDnDestroy(cusolverH));
    }
    void cuGeSvd_internal_d(const boost::intrusive_ptr<Storage_base> &in,
                            boost::intrusive_ptr<Storage_base> &U,
                            boost::intrusive_ptr<Storage_base> &vT,
                            boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                            const cytnx_int64 &N) {
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_double *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(cytnx_double)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_double) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 max = std::max(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      if (N < M) {
        ldA = M, ldu = M, ldvT = min;
      }
      cytnx_int32 lwork = 0;

      void *UMem, *vTMem;
      if (U->Mem) {
        UMem = U->Mem;
      } else {
        if (jobu == 'S') checkCudaErrors(hipMalloc(&UMem, max * max * sizeof(cytnx_double)));
      }
      if (vT->Mem) {
        vTMem = vT->Mem;
      } else {
        if (jobv == 'S') checkCudaErrors(hipMalloc(&vTMem, max * max * sizeof(cytnx_double)));
      }

      // query working space :
      checkCudaErrors(hipsolverDnDgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      cytnx_double *work;
      cytnx_double *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(cytnx_double)));
      checkCudaErrors(hipMalloc((void **)&rwork, (min - 1) * sizeof(cytnx_double)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      if (N >= M) {
        hipsolverDnDgesvd(cusolverH, jobv, jobu, N, M, (cytnx_double *)Mij, ldA,
                         (cytnx_double *)S->Mem, (cytnx_double *)vTMem, ldu, (cytnx_double *)UMem,
                         ldvT, work, lwork, rwork, devinfo);
      } else {
        hipsolverDnDgesvd(cusolverH, jobu, jobv, M, N, (cytnx_double *)Mij, ldA,
                         (cytnx_double *)S->Mem, (cytnx_double *)UMem, ldu, (cytnx_double *)vTMem,
                         ldvT, work, lwork, rwork, devinfo);
        if (U->dtype != Type.Void)
          U->Move_memory_({(cytnx_uint64)min, (cytnx_uint64)M}, {1, 0}, {1, 0});
        if (vT->dtype != Type.Void)
          vT->Move_memory_({(cytnx_uint64)N, (cytnx_uint64)min}, {1, 0}, {1, 0});
      }

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(
        info != 0, "%s %d %s", "Error in cuBlas function 'hipsolverDnDgesvd': cuBlas INFO = ", info,
        "If info>0, possibly svd not converge, if info<0, see cusolver manual for more info.");

      checkCudaErrors(hipFree(work));
      checkCudaErrors(hipFree(Mij));
      if (UMem != nullptr and U->dtype == Type.Void) {
        checkCudaErrors(hipFree(UMem));
      }
      if (vTMem != nullptr and vT->dtype == Type.Void) {
        checkCudaErrors(hipFree(vTMem));
      }
      checkCudaErrors(hipFree(devinfo));
      checkCudaErrors(hipsolverDnDestroy(cusolverH));
    }
    void cuGeSvd_internal_f(const boost::intrusive_ptr<Storage_base> &in,
                            boost::intrusive_ptr<Storage_base> &U,
                            boost::intrusive_ptr<Storage_base> &vT,
                            boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                            const cytnx_int64 &N) {
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_float *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(cytnx_float)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_float) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 max = std::max(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      if (N < M) {
        ldA = M, ldu = M, ldvT = min;
      }
      cytnx_int32 lwork = 0;

      void *UMem, *vTMem;
      if (U->Mem) {
        UMem = U->Mem;
      } else {
        if (jobu == 'S') checkCudaErrors(hipMalloc(&UMem, max * max * sizeof(cytnx_float)));
      }
      if (vT->Mem) {
        vTMem = vT->Mem;
      } else {
        if (jobv == 'S') checkCudaErrors(hipMalloc(&vTMem, max * max * sizeof(cytnx_float)));
      }

      // query working space :
      checkCudaErrors(hipsolverDnSgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      cytnx_float *work;
      cytnx_float *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(cytnx_float)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_float64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      if (N >= M) {
        checkCudaErrors(hipsolverDnSgesvd(cusolverH, jobv, jobu, N, M, (cytnx_float *)Mij, ldA,
                                         (cytnx_float *)S->Mem, (cytnx_float *)vTMem, ldu,
                                         (cytnx_float *)UMem, ldvT, work, lwork, rwork, devinfo));
      } else {
        checkCudaErrors(hipsolverDnSgesvd(cusolverH, jobu, jobv, M, N, (cytnx_float *)Mij, ldA,
                                         (cytnx_float *)S->Mem, (cytnx_float *)UMem, ldu,
                                         (cytnx_float *)vTMem, ldvT, work, lwork, rwork, devinfo));
        if (U->dtype != Type.Void)
          U->Move_memory_({(cytnx_uint64)min, (cytnx_uint64)M}, {1, 0}, {1, 0});
        if (vT->dtype != Type.Void)
          vT->Move_memory_({(cytnx_uint64)N, (cytnx_uint64)min}, {1, 0}, {1, 0});
      }

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));
      cytnx_error_msg(
        info != 0, "%s %d %s", "Error in cuBlas function 'hipsolverDnSgesvd': cuBlas INFO = ", info,
        "If info>0, possibly svd not converge, if info<0, see cusolver manual for more info.");

      checkCudaErrors(hipFree(work));
      checkCudaErrors(hipFree(Mij));
      if (UMem != nullptr and U->dtype == Type.Void) {
        checkCudaErrors(hipFree(UMem));
      }
      if (vTMem != nullptr and vT->dtype == Type.Void) {
        checkCudaErrors(hipFree(vTMem));
      }
      checkCudaErrors(hipFree(devinfo));
      checkCudaErrors(hipsolverDnDestroy(cusolverH));
    }

  }  // namespace linalg_internal
}  // namespace cytnx
