#include "cuVectordot_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "lapack_wrapper.hpp"

namespace cytnx{



    namespace linalg_internal{
        using namespace std;
        void cuVectordot_internal_cd(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){

            hipblasHandle_t cublasH = NULL;
            checkCudaErrors(hipblasCreate(&cublasH));

            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            hipDoubleComplex *_Rin = (hipDoubleComplex*)Rin->Mem;
            
            _out[0] = make_hipDoubleComplex(0.,0.);
            unsigned long long remain = len;
            unsigned long long bias = 0; 
            unsigned int TotSeg = (len/INT_MAX)+1;
            int cnt = 0;
            cytnx_int32 ONE = 1;
            cytnx_int32 MAXX = INT_MAX; 
            hipDoubleComplex *dacres;
            hipMallocManaged((void**)&dacres,sizeof(hipDoubleComplex)*TotSeg);
            hipMemset(dacres,0,sizeof(hipDoubleComplex)*TotSeg);

            while(remain!=0){
                cout << "cnt"<< endl;
                if(remain>=INT_MAX) MAXX = INT_MAX;
                else MAXX = remain;
                
                if(is_conj)
                    checkCudaErrors(hipblasZdotc(cublasH,MAXX,&_Lin[bias],ONE,&_Rin[bias],ONE,dacres+cnt));
                else
                    checkCudaErrors(hipblasZdotu(cublasH,MAXX,&_Lin[bias],ONE,&_Rin[bias],ONE,dacres+cnt));
                
                remain -= MAXX;
                bias += MAXX;
                cnt += 1;
            }

            cytnx_complex128 *hacres = (cytnx_complex128*)malloc(sizeof(cytnx_complex128)*TotSeg);
            hipMemcpy((hipDoubleComplex*)hacres,dacres,sizeof(cytnx_complex128)*TotSeg,hipMemcpyDeviceToHost);
            for(int i=1;i<TotSeg;i++){
                hacres[0] += hacres[i];
            }
            _out[0] = make_hipDoubleComplex(hacres[0].real(),hacres[0].imag());


            free(hacres);
            hipFree(dacres);
            hipblasDestroy(cublasH);

        }
        void cuVectordot_internal_cf(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            hipblasHandle_t cublasH = NULL;
            checkCudaErrors(hipblasCreate(&cublasH));

            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            hipFloatComplex *_Rin = (hipFloatComplex*)Rin->Mem;
            
            _out[0] = make_hipFloatComplex(0.,0.);
            unsigned long long remain = len;
            unsigned long long bias = 0; 
            unsigned int TotSeg = (len/INT_MAX)+1;
            int cnt = 0;
            cytnx_int32 ONE = 1;
            cytnx_int32 MAXX = INT_MAX; 
            hipFloatComplex *dacres;
            hipMallocManaged((void**)&dacres,sizeof(hipFloatComplex)*TotSeg);
            hipMemset(dacres,0,sizeof(hipFloatComplex)*TotSeg);

            while(remain!=0){
                if(remain>=INT_MAX) MAXX = INT_MAX;
                else MAXX = remain;
                
                if(is_conj)
                    checkCudaErrors(hipblasCdotc(cublasH,MAXX,&_Lin[bias],ONE,&_Rin[bias],ONE,dacres+cnt));
                else
                    checkCudaErrors(hipblasCdotu(cublasH,MAXX,&_Lin[bias],ONE,&_Rin[bias],ONE,dacres+cnt));
                
                remain -= MAXX;
                bias += MAXX;
                cnt += 1;
            }

            cytnx_complex64 *hacres = (cytnx_complex64*)malloc(sizeof(cytnx_complex64)*TotSeg);
            hipMemcpy((hipFloatComplex*)hacres,dacres,sizeof(cytnx_complex64)*TotSeg,hipMemcpyDeviceToHost);
            for(int i=1;i<TotSeg;i++){
                hacres[0] += hacres[i];
            }
            _out[0] = make_hipFloatComplex(hacres[0].real(),hacres[0].imag());


            free(hacres);
            hipFree(dacres);
            hipblasDestroy(cublasH);


        }
        void cuVectordot_internal_d(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            hipblasHandle_t cublasH = NULL;
            checkCudaErrors(hipblasCreate(&cublasH));

            cytnx_double *_out = (cytnx_double*)out->Mem;
            cytnx_double *_Lin = (cytnx_double*)Lin->Mem;
            cytnx_double *_Rin = (cytnx_double*)Rin->Mem;

            _out[0] = 0;
            unsigned long long remain = len;
            unsigned long long bias = 0; 
            cytnx_int32 ONE = 1;
            cytnx_int32 MAXX = INT_MAX; 
            cytnx_double *acres;
            hipMalloc((void**)&acres,sizeof(cytnx_double));

            while(remain!=0){
                if(remain>=INT_MAX) MAXX = INT_MAX;
                else MAXX = remain;
                
                checkCudaErrors(hipblasDdot(cublasH,MAXX,&_Lin[bias],ONE,&_Rin[bias],ONE,acres));
                
                _out[0] += acres[0];
                remain -= MAXX;
                bias += MAXX;
            }
            hipFree(acres);
            hipblasDestroy(cublasH);

        }
        void cuVectordot_internal_f(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            hipblasHandle_t cublasH = NULL;
            checkCudaErrors(hipblasCreate(&cublasH));

            cytnx_float *_out = (cytnx_float*)out->Mem;
            cytnx_float *_Lin = (cytnx_float*)Lin->Mem;
            cytnx_float *_Rin = (cytnx_float*)Rin->Mem;

            _out[0] = 0;
            unsigned long long remain = len;
            unsigned long long bias = 0; 
            cytnx_int32 ONE = 1;
            cytnx_int32 MAXX = INT_MAX; 
            cytnx_float *acres;
            hipMalloc((void**)&acres,sizeof(cytnx_float));

            while(remain!=0){
                if(remain>=INT_MAX) MAXX = INT_MAX;
                else MAXX = remain;
                
                checkCudaErrors(hipblasSdot(cublasH,MAXX,&_Lin[bias],ONE,&_Rin[bias],ONE,acres));
                
                _out[0] += acres[0];
                remain -= MAXX;
                bias += MAXX;
            }
            hipFree(acres);
            hipblasDestroy(cublasH);
        }
        void cuVectordot_internal_i64(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_i64][FATAL Invalid internal call.] No internal function for vectordot of int64 type.%s","\n");
        }
        void cuVectordot_internal_u64(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_u64][FATAL Invalid internal call.] No internal function for vectordot of uint64 type.%s","\n");
        }
        void cuVectordot_internal_i32(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_i32][FATAL Invalid internal call.] No internal function for vectordot of int32 type.%s","\n");
        }
        void cuVectordot_internal_u32(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_u32][FATAL Invalid internal call.] No internal function for vectordot of uint32 type.%s","\n");
        }
        void cuVectordot_internal_i16(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_i16][FATAL Invalid internal call.] No internal function for vectordot of int16 type.%s","\n");
        }
        void cuVectordot_internal_u16(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_u16][FATAL Invalid internal call.] No internal function for vectordot of uint16 type.%s","\n");
        }
        void cuVectordot_internal_b(boost::intrusive_ptr<Storage_base> & out, const boost::intrusive_ptr<Storage_base> & Lin, const boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len, const bool &is_conj){
            cytnx_error_msg(1,"[ERROR][cuVectordot_internal_b][FATAL Invalid internal call.] No internal function for vectordot of bool type.%s","\n");
        }

    }//namespace linalg_internal
}//namespace cytnx



