#include "hip/hip_runtime.h"
#include "linalg/linalg_internal_gpu/cuMul_internal.hpp"
#include "utils/utils_internal.hpp"

#ifdef UNI_OMP
    #include <omp.h>
#endif

namespace tor10{

    namespace linalg_internal{

        //====================================================================
        //generic R+R kernel
        template<class T1,class T2,class T3>
        __global__ void cuMul_rconst_kernel(T1 *out, const T2 *ptr, const tor10_uint64 Nelem, const T3 val){
              if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                  out[blockIdx.x*blockDim.x + threadIdx.x] = ptr[blockIdx.x*blockDim.x + threadIdx.x] * val;
              }
              __syncthreads();
         }
        
        template<class T1,class T2,class T3>
        __global__ void cuMul_lconst_kernel(T1 *out, const T2 val, const tor10_uint64 Nelem, const T3 *ptr){
              if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                  out[blockIdx.x*blockDim.x + threadIdx.x] = val * ptr[blockIdx.x*blockDim.x + threadIdx.x];
              }
              __syncthreads();
         }
        
        template<class T1,class T2,class T3>
        __global__ void cuMul_tn_kernel(T1 *out, const T2 *val, const tor10_uint64 Nelem, const T3 *ptr){
              if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                  out[blockIdx.x*blockDim.x + threadIdx.x] = val[blockIdx.x*blockDim.x + threadIdx.x] * ptr[blockIdx.x*blockDim.x + threadIdx.x];
              }
              __syncthreads();
        }

        //=====================================================================

        /// cuMul
        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const hipDoubleComplex val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],val);
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const hipDoubleComplex *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],val);
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const hipDoubleComplex *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],val[blockIdx.x*blockDim.x + threadIdx.x]);
            }
            __syncthreads();
        }
        void cuMul_internal_cdtcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            hipDoubleComplex *_Rin = (hipDoubleComplex*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;
                
           if(Lin->size()==1){
                cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
            }else if(Rin->size()==1){
                cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
            }else{
                cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
            }
        }



        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const hipFloatComplex val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],hipComplexFloatToDouble(val));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const hipFloatComplex *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,hipComplexFloatToDouble(ptr[blockIdx.x*blockDim.x + threadIdx.x]));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const hipFloatComplex *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],hipComplexFloatToDouble(val[blockIdx.x*blockDim.x + threadIdx.x]));
            }
            __syncthreads();
        }
        void cuMul_internal_cdtcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            hipFloatComplex *_Rin = (hipFloatComplex*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

            if(Lin->size()==1){
                cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
            }else if(Rin->size()==1){
                cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
            }else{
                cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
            }

        }

        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_double val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const tor10_double *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,make_hipDoubleComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_double *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cdtd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            tor10_double *_Rin = (tor10_double*)Rin->Mem;


            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

            if(Lin->size()==1){
                cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
            }else if(Rin->size()==1){
                cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
            }else{
                cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
            }


        }

        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_float val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const tor10_float *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,make_hipDoubleComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_float *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){                                                
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cdtf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            tor10_float *_Rin = (tor10_float*)Rin->Mem;


            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

            if(Lin->size()==1){
                cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
            }else if(Rin->size()==1){
                cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
            }else{
                cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
            }



        }


          __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_uint64 val){
              if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                  out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val,0));
              }
              __syncthreads();
          }
          __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const tor10_uint64 *ptr){
              if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                  out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,make_hipDoubleComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
              }
              __syncthreads();
          }
          __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_uint64 *val){
              if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                  out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
              }
              __syncthreads();
          }

        void cuMul_internal_cdtu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            tor10_uint64 *_Rin = (tor10_uint64*)Rin->Mem;


            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_uint32 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const tor10_uint32 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,make_hipDoubleComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_uint32 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cdtu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }



        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_int64 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const tor10_int64 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,make_hipDoubleComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_int64 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cdti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            tor10_int64 *_Rin = (tor10_int64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }


        }


        __global__ void cuMul_rconst_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_int32 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipDoubleComplex *out, const hipDoubleComplex val, const tor10_uint64 Nelem, const tor10_int32 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(val,make_hipDoubleComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipDoubleComplex *out, const hipDoubleComplex *ptr, const tor10_uint64 Nelem, const tor10_int32 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmul(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipDoubleComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cdti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipDoubleComplex *_out = (hipDoubleComplex*)out->Mem;
            hipDoubleComplex *_Lin = (hipDoubleComplex*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }

        void cuMul_internal_cftcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){

		    cuMul_internal_cdtcf(out,Rin,Lin,len);

	    }

        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const hipFloatComplex val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],val);
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_uint64 Nelem, const hipFloatComplex *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,ptr[blockIdx.x*blockDim.x + threadIdx.x]);
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const hipFloatComplex *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],val[blockIdx.x*blockDim.x + threadIdx.x]);
            }
            __syncthreads();
        }
        void cuMul_internal_cftcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            hipFloatComplex *_Rin = (hipFloatComplex*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const tor10_double val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_uint64 Nelem, const tor10_double *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,make_hipFloatComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const tor10_double *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cftd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            tor10_double *_Rin = (tor10_double*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }



        }

        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const tor10_float val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_uint64 Nelem, const tor10_float *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,make_hipFloatComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const tor10_float *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cftf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            tor10_float *_Rin = (tor10_float*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const tor10_uint64 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_uint64 Nelem, const tor10_uint64 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,make_hipFloatComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint64 Nelem, const tor10_uint64 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cftu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            tor10_uint64 *_Rin = (tor10_uint64*)Rin->Mem;
            
            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint32 Nelem, const tor10_uint32 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_uint32 Nelem, const tor10_uint32 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,make_hipFloatComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_uint32 Nelem, const tor10_uint32 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cftu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }


        }


        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_int64 Nelem, const tor10_int64 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_int64 Nelem, const tor10_int64 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,make_hipFloatComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_int64 Nelem, const tor10_int64 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cfti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            tor10_int64 *_Rin = (tor10_int64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }


        }


        __global__ void cuMul_rconst_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_int32 Nelem, const tor10_int32 val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val,0));
            }
            __syncthreads();
        }
        __global__ void cuMul_lconst_kernel(hipFloatComplex *out, const hipFloatComplex val, const tor10_int32 Nelem, const tor10_int32 *ptr){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(val,make_hipFloatComplex(ptr[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        __global__ void cuMul_tn_kernel(hipFloatComplex *out, const hipFloatComplex *ptr, const tor10_int32 Nelem, const tor10_int32 *val){
            if(blockIdx.x*blockDim.x + threadIdx.x < Nelem){
                out[blockIdx.x*blockDim.x + threadIdx.x] = hipCmulf(ptr[blockIdx.x*blockDim.x + threadIdx.x],make_hipFloatComplex(val[blockIdx.x*blockDim.x + threadIdx.x],0));
            }
            __syncthreads();
        }
        void cuMul_internal_cfti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            hipFloatComplex *_out = (hipFloatComplex*)out->Mem;
            hipFloatComplex *_Lin = (hipFloatComplex*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }

        void cuMul_internal_dtcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cdtd(out,Rin,Lin,len);
        }
        void cuMul_internal_dtcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cftd(out,Rin,Lin,len);
        }


        void cuMul_internal_dtd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_double *_out = (tor10_double*)out->Mem;
            tor10_double *_Lin = (tor10_double*)Lin->Mem;
            tor10_double *_Rin = (tor10_double*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }


        }
        void cuMul_internal_dtf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_double *_out = (tor10_double*)out->Mem;
            tor10_double *_Lin = (tor10_double*)Lin->Mem;
            tor10_float *_Rin = (tor10_float*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_dtu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){

            tor10_double *_out = (tor10_double*)out->Mem;
            tor10_double *_Lin = (tor10_double*)Lin->Mem;
            tor10_uint64 *_Rin = (tor10_uint64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_dtu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){

            tor10_double *_out = (tor10_double*)out->Mem;
            tor10_double *_Lin = (tor10_double*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_dti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){

            tor10_double *_out = (tor10_double*)out->Mem;
            tor10_double *_Lin = (tor10_double*)Lin->Mem;
            tor10_int64 *_Rin = (tor10_int64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_dti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){

            tor10_double *_out = (tor10_double*)out->Mem;
            tor10_double *_Lin = (tor10_double*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }

        void cuMul_internal_ftcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cdtf(out,Rin,Lin,len);
        }
        void cuMul_internal_ftcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cftf(out,Rin,Lin,len);
        }
        void cuMul_internal_ftd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_dtf(out,Rin,Lin,len);
        }
        void cuMul_internal_ftf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_float *_out = (tor10_float*)out->Mem;
            tor10_float *_Lin = (tor10_float*)Lin->Mem;
            tor10_float *_Rin = (tor10_float*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_ftu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_float *_out = (tor10_float*)out->Mem;
            tor10_float *_Lin = (tor10_float*)Lin->Mem;
            tor10_uint64 *_Rin = (tor10_uint64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_ftu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_float *_out = (tor10_float*)out->Mem;
            tor10_float *_Lin = (tor10_float*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_fti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_float *_out = (tor10_float*)out->Mem;
            tor10_float *_Lin = (tor10_float*)Lin->Mem;
            tor10_int64 *_Rin = (tor10_int64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_fti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_float *_out = (tor10_float*)out->Mem;
            tor10_float *_Lin = (tor10_float*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        void cuMul_internal_i64tcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cdti64(out,Rin,Lin,len);
        }
        void cuMul_internal_i64tcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cfti64(out,Rin,Lin,len);
        }
        void cuMul_internal_i64td(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_dti64(out,Rin,Lin,len);
        }
        void cuMul_internal_i64tf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_fti64(out,Rin,Lin,len);
        }
        void cuMul_internal_i64ti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_int64 *_out = (tor10_int64*)out->Mem;
            tor10_int64 *_Lin = (tor10_int64*)Lin->Mem;
            tor10_int64 *_Rin = (tor10_int64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_i64tu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_int64 *_out = (tor10_int64*)out->Mem;
            tor10_int64 *_Lin = (tor10_int64*)Lin->Mem;
            tor10_uint64 *_Rin = (tor10_uint64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_i64ti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_int64 *_out = (tor10_int64*)out->Mem;
            tor10_int64 *_Lin = (tor10_int64*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_i64tu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_int64 *_out = (tor10_int64*)out->Mem;
            tor10_int64 *_Lin = (tor10_int64*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        void cuMul_internal_u64tcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cdtu64(out,Rin,Lin,len);
        }
        void cuMul_internal_u64tcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cftu64(out,Rin,Lin,len);
        }
        void cuMul_internal_u64td(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_dtu64(out,Rin,Lin,len);
        }
        void cuMul_internal_u64tf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_ftu64(out,Rin,Lin,len);
        }
        void cuMul_internal_u64ti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_i64tu64(out,Rin,Lin,len);
        }
        void cuMul_internal_u64tu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_uint64 *_out = (tor10_uint64*)out->Mem;
            tor10_uint64 *_Lin = (tor10_uint64*)Lin->Mem;
            tor10_uint64 *_Rin = (tor10_uint64*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_u64ti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_uint64 *_out = (tor10_uint64*)out->Mem;
            tor10_uint64 *_Lin = (tor10_uint64*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }
        void cuMul_internal_u64tu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_uint64 *_out = (tor10_uint64*)out->Mem;
            tor10_uint64 *_Lin = (tor10_uint64*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }

        void cuMul_internal_i32tcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cdti32(out,Rin,Lin,len);

        }
        void cuMul_internal_i32tcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cfti32(out,Rin,Lin,len);

        }
        void cuMul_internal_i32td(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_dti32(out,Rin,Lin,len);

        }
        void cuMul_internal_i32tf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_fti32(out,Rin,Lin,len);

        }
        void cuMul_internal_i32ti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_i64ti32(out,Rin,Lin,len);

        }
        void cuMul_internal_i32tu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_u64ti32(out,Rin,Lin,len);

        }
        void cuMul_internal_i32ti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_int32 *_out = (tor10_int32*)out->Mem;
            tor10_int32 *_Lin = (tor10_int32*)Lin->Mem;
            tor10_int32 *_Rin = (tor10_int32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }
        void cuMul_internal_i32tu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_int32 *_out = (tor10_int32*)out->Mem;
            tor10_int32 *_Lin = (tor10_int32*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }

        }


        void cuMul_internal_u32tcd(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cdtu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32tcf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_cftu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32td(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_dtu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32tf(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_ftu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32ti64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_i64tu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32tu64(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_u64tu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32ti32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
             cuMul_internal_i32tu32(out,Rin,Lin,len);

        }
        void cuMul_internal_u32tu32(boost::intrusive_ptr<Storage_base> & out, boost::intrusive_ptr<Storage_base> & Lin, boost::intrusive_ptr<Storage_base> & Rin, const unsigned long long &len){
            tor10_uint32 *_out = (tor10_uint32*)out->Mem;
            tor10_uint32 *_Lin = (tor10_uint32*)Lin->Mem;
            tor10_uint32 *_Rin = (tor10_uint32*)Rin->Mem;

            tor10_uint32 NBlocks = len/512;
            if(len%512) NBlocks += 1;

              if(Lin->size()==1){
                  cuMul_lconst_kernel<<<NBlocks,512>>>(_out,_Lin[0],len,_Rin);
              }else if(Rin->size()==1){
                  cuMul_rconst_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin[0]);
              }else{
                  cuMul_tn_kernel<<<NBlocks,512>>>(_out,_Lin,len,_Rin);
              }
        }





    }//namespace linalg_internal
}//namespace tor10


