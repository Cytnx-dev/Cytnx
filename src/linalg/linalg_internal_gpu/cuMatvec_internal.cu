#include "hip/hip_runtime.h"
#include "cuMatvec_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

namespace cytnx {

  namespace linalg_internal {

    template <typename UniType>
    __global__ void cuMatVec_kernel(UniType *out, const UniType *inl, const UniType *inr,
                                    cytnx_int64 Ml, cytnx_int64 Nr) {
      UniType tmp = 0;
      cytnx_uint64 sid = blockIdx.x * blockDim.x + threadIdx.x;
      if (sid < cytnx_uint64(Ml)) {
        for (cytnx_int64 c = 0; c < Nr; c++) {
          tmp += inl[sid * Nr + c] * inr[c];
        }
        out[sid] = tmp;
      }
    }

    /// cuMatvec
    void cuMatvec_internal_cd(boost::intrusive_ptr<Storage_base> &out,
                              const boost::intrusive_ptr<Storage_base> &inl,
                              const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                              const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex128 alpha = cytnx_complex128(1, 0), beta = cytnx_complex128(0, 0);

      hipDoubleComplex *_out = (hipDoubleComplex *)out->Mem;
      hipDoubleComplex *_inl = (hipDoubleComplex *)inl->Mem;
      hipDoubleComplex *_inr = (hipDoubleComplex *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasZgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, (hipDoubleComplex *)&alpha,
      //                             _inl, blsMl, _inr, 1, (hipDoubleComplex *)&beta, _out, 1));
      checkCudaErrors(hipblasZgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, (hipDoubleComplex *)&alpha,
                                  _inl, blsMl, _inr, 1, (hipDoubleComplex *)&beta, _out, 1));

      hipblasDestroy(cublasH);
    }
    void cuMatvec_internal_cf(boost::intrusive_ptr<Storage_base> &out,
                              const boost::intrusive_ptr<Storage_base> &inl,
                              const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                              const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex64 alpha = cytnx_complex64(1, 0), beta = cytnx_complex64(0, 0);

      hipFloatComplex *_out = (hipFloatComplex *)out->Mem;
      hipFloatComplex *_inl = (hipFloatComplex *)inl->Mem;
      hipFloatComplex *_inr = (hipFloatComplex *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasCgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, (hipFloatComplex *)&alpha,
      //                             _inl, blsMl, _inr, 1, (hipFloatComplex *)&beta, _out, 1));
      checkCudaErrors(hipblasCgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, (hipFloatComplex *)&alpha,
                                  _inl, blsMl, _inr, 1, (hipFloatComplex *)&beta, _out, 1));

      hipblasDestroy(cublasH);
    }

    void cuMatvec_internal_d(boost::intrusive_ptr<Storage_base> &out,
                             const boost::intrusive_ptr<Storage_base> &inl,
                             const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                             const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_double alpha = 1, beta = 0;

      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_inl = (cytnx_double *)inl->Mem;
      cytnx_double *_inr = (cytnx_double *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasDgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, &alpha, _inl, blsMl, _inr,
      // 1, &beta, _out, 1));
      checkCudaErrors(hipblasDgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, &alpha, _inl, blsNr, _inr, 1,
                                  &beta, _out, 1));

      hipblasDestroy(cublasH);
    }
    void cuMatvec_internal_f(boost::intrusive_ptr<Storage_base> &out,
                             const boost::intrusive_ptr<Storage_base> &inl,
                             const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                             const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_float alpha = 1, beta = 0;

      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_inl = (cytnx_float *)inl->Mem;
      cytnx_float *_inr = (cytnx_float *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasSgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, &alpha, _inr, blsMl, _inl,
      // 1,
      //                             &beta, _out, 1));
      checkCudaErrors(hipblasSgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, &alpha, _inr, blsMl, _inl, 1,
                                  &beta, _out, 1));

      hipblasDestroy(cublasH);
    }
    void cuMatvec_internal_i64(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_inl = (cytnx_int64 *)inl->Mem;
      cytnx_int64 *_inr = (cytnx_int64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_u64(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_inl = (cytnx_uint64 *)inl->Mem;
      cytnx_uint64 *_inr = (cytnx_uint64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_i32(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_inl = (cytnx_int64 *)inl->Mem;
      cytnx_int64 *_inr = (cytnx_int64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_u32(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_inl = (cytnx_uint64 *)inl->Mem;
      cytnx_uint64 *_inr = (cytnx_uint64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_i16(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_int16 *_inl = (cytnx_int16 *)inl->Mem;
      cytnx_int16 *_inr = (cytnx_int16 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_u16(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_uint16 *_out = (cytnx_uint16 *)out->Mem;
      cytnx_uint16 *_inl = (cytnx_uint16 *)inl->Mem;
      cytnx_uint16 *_inr = (cytnx_uint16 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_b(boost::intrusive_ptr<Storage_base> &out,
                             const boost::intrusive_ptr<Storage_base> &inl,
                             const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                             const cytnx_int64 &Nr) {
      cytnx_bool *_out = (cytnx_bool *)out->Mem;
      cytnx_bool *_inl = (cytnx_bool *)inl->Mem;
      cytnx_bool *_inr = (cytnx_bool *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
