#include "hip/hip_runtime.h"
#include "linalg/linalg_internal_gpu/cuSvd_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "lapack_wrapper.hpp"

namespace cytnx {

  namespace linalg_internal {

    /// cuSvd
    void cuSvd_internal_cd(const boost::intrusive_ptr<Storage_base> &in,
                           boost::intrusive_ptr<Storage_base> &U,
                           boost::intrusive_ptr<Storage_base> &vT,
                           boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                           const cytnx_int64 &N) {
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      hipDoubleComplex *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(hipDoubleComplex)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_complex128) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      cytnx_int32 lwork = 0;

      // query working space :
      checkCudaErrors(hipsolverDnZgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      hipDoubleComplex *work;
      cytnx_double *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(hipDoubleComplex)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_double64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      checkCudaErrors(hipsolverDnZgesvd(
        cusolverH, jobv, jobu, N, M, Mij, ldA, (cytnx_double *)S->Mem, (hipDoubleComplex *)vT->Mem,
        ldu, (hipDoubleComplex *)U->Mem, ldvT, work, lwork, rwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'hipsolverDnZgesvd': cuBlas INFO = ", info);

      hipFree(work);
      hipFree(Mij);
      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }
    void cuSvd_internal_cf(const boost::intrusive_ptr<Storage_base> &in,
                           boost::intrusive_ptr<Storage_base> &U,
                           boost::intrusive_ptr<Storage_base> &vT,
                           boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                           const cytnx_int64 &N) {
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      hipFloatComplex *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(hipFloatComplex)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_complex128) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      cytnx_int32 lwork = 0;

      // query working space :
      checkCudaErrors(hipsolverDnCgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      hipFloatComplex *work;
      cytnx_float *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(hipFloatComplex)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_float64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      checkCudaErrors(hipsolverDnCgesvd(cusolverH, jobv, jobu, N, M, Mij, ldA, (cytnx_float *)S->Mem,
                                       (hipFloatComplex *)vT->Mem, ldu, (hipFloatComplex *)U->Mem,
                                       ldvT, work, lwork, rwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'hipsolverDnCgesvd': cuBlas INFO = ", info);

      hipFree(work);
      hipFree(Mij);
      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }
    void cuSvd_internal_d(const boost::intrusive_ptr<Storage_base> &in,
                          boost::intrusive_ptr<Storage_base> &U,
                          boost::intrusive_ptr<Storage_base> &vT,
                          boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                          const cytnx_int64 &N) {
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_double *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(cytnx_double)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_double) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      cytnx_int32 lwork = 0;

      // query working space :
      checkCudaErrors(hipsolverDnDgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      cytnx_double *work;
      cytnx_double *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(cytnx_double)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_double64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      checkCudaErrors(hipsolverDnDgesvd(cusolverH, jobv, jobu, N, M, Mij, ldA,
                                       (cytnx_double *)S->Mem, (cytnx_double *)vT->Mem, ldu,
                                       (cytnx_double *)U->Mem, ldvT, work, lwork, rwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'hipsolverDnDgesvd': cuBlas INFO = ", info);

      hipFree(work);
      hipFree(Mij);
      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }
    void cuSvd_internal_f(const boost::intrusive_ptr<Storage_base> &in,
                          boost::intrusive_ptr<Storage_base> &U,
                          boost::intrusive_ptr<Storage_base> &vT,
                          boost::intrusive_ptr<Storage_base> &S, const cytnx_int64 &M,
                          const cytnx_int64 &N) {
      signed char jobu, jobv;

      // if U and vT are NULL ptr, then it will not be computed.
      jobu = (U->dtype == Type.Void) ? 'N' : 'S';
      jobv = (vT->dtype == Type.Void) ? 'N' : 'S';

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_float *Mij;
      checkCudaErrors(hipMalloc((void **)&Mij, M * N * sizeof(cytnx_float)));
      checkCudaErrors(
        hipMemcpy(Mij, in->Mem, sizeof(cytnx_float) * M * N, hipMemcpyDeviceToDevice));

      cytnx_int32 min = std::min(M, N);
      cytnx_int32 ldA = N, ldu = N, ldvT = min;
      cytnx_int32 lwork = 0;

      // query working space :
      checkCudaErrors(hipsolverDnSgesvd_bufferSize(cusolverH, N, M, &lwork));

      // allocate working space:
      cytnx_float *work;
      cytnx_float *rwork = NULL;
      checkCudaErrors(hipMalloc((void **)&work, lwork * sizeof(cytnx_float)));
      // checkCudaErrors(hipMalloc((void**)&rwork,(min-1)*sizeof(cytnx_float64)));

      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipMemset(devinfo, 0, sizeof(cytnx_int32)));

      cytnx_int32 info;
      /// compute:
      checkCudaErrors(hipsolverDnSgesvd(cusolverH, jobv, jobu, N, M, Mij, ldA, (cytnx_float *)S->Mem,
                                       (cytnx_float *)vT->Mem, ldu, (cytnx_float *)U->Mem, ldvT,
                                       work, lwork, rwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'hipsolverDnSgesvd': cuBlas INFO = ", info);

      hipFree(work);
      hipFree(Mij);
      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
