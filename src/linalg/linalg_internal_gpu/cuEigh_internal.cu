#include "cuEigh_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

namespace cytnx {

  namespace linalg_internal {

    /// cuEigh
    void cuEigh_internal_cd(const boost::intrusive_ptr<Storage_base> &in,
                            boost::intrusive_ptr<Storage_base> &e,
                            boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
      if (v->dtype == Type.Void) jobz = HIPSOLVER_EIG_MODE_NOVECTOR;

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_complex128 *tA;
      if (v != NULL) {
        tA = (cytnx_complex128 *)v->Mem;
        checkCudaErrors(hipMemcpy(v->Mem, in->Mem, sizeof(cytnx_complex128) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      } else {
        checkCudaErrors(hipMalloc((void **)&tA, cytnx_uint64(L) * L * sizeof(cytnx_complex128)));
        checkCudaErrors(hipMemcpy(tA, in->Mem, sizeof(cytnx_complex128) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      }

      // query buffer:
      cytnx_int32 lwork = 0;
      cytnx_int32 b32L = L;
      checkCudaErrors(hipsolverDnZheevd_bufferSize(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L,
                                                  (hipDoubleComplex *)tA, b32L,
                                                  (cytnx_double *)e->Mem, &lwork));

      // allocate working space:
      cytnx_complex128 *work;
      checkCudaErrors(hipMalloc((void **)&work, sizeof(cytnx_complex128) * lwork));

      // call :
      cytnx_int32 info;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipsolverDnZheevd(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L,
                                       (hipDoubleComplex *)tA, b32L, (cytnx_double *)e->Mem,
                                       (hipDoubleComplex *)work, lwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'hipsolverDnZheevd': cuBlas INFO = ", info);

      hipFree(work);
      if (v->dtype == Type.Void) hipFree(tA);

      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }
    void cuEigh_internal_cf(const boost::intrusive_ptr<Storage_base> &in,
                            boost::intrusive_ptr<Storage_base> &e,
                            boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
      if (v->dtype == Type.Void) jobz = HIPSOLVER_EIG_MODE_NOVECTOR;

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_complex64 *tA;
      if (v != NULL) {
        tA = (cytnx_complex64 *)v->Mem;
        checkCudaErrors(hipMemcpy(v->Mem, in->Mem, sizeof(cytnx_complex64) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      } else {
        checkCudaErrors(hipMalloc((void **)&tA, cytnx_uint64(L) * L * sizeof(cytnx_complex64)));
        checkCudaErrors(hipMemcpy(tA, in->Mem, sizeof(cytnx_complex64) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      }

      // query buffer:
      cytnx_int32 lwork = 0;
      cytnx_int32 b32L = L;
      checkCudaErrors(hipsolverDnCheevd_bufferSize(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L,
                                                  (hipFloatComplex *)tA, b32L, (cytnx_float *)e->Mem,
                                                  &lwork));

      // allocate working space:
      cytnx_complex64 *work;
      checkCudaErrors(hipMalloc((void **)&work, sizeof(cytnx_complex64) * lwork));

      // call :
      cytnx_int32 info;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipsolverDnCheevd(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L,
                                       (hipFloatComplex *)tA, b32L, (cytnx_float *)e->Mem,
                                       (hipFloatComplex *)work, lwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'hipsolverDnZheevd': cuBlas INFO = ", info);

      hipFree(work);
      if (v->dtype == Type.Void) hipFree(tA);

      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }
    void cuEigh_internal_d(const boost::intrusive_ptr<Storage_base> &in,
                           boost::intrusive_ptr<Storage_base> &e,
                           boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
      if (v->dtype == Type.Void) jobz = HIPSOLVER_EIG_MODE_NOVECTOR;

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_double *tA;
      if (v->dtype != Type.Void) {
        tA = (cytnx_double *)v->Mem;
        checkCudaErrors(hipMemcpy(v->Mem, in->Mem, sizeof(cytnx_double) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      } else {
        checkCudaErrors(hipMalloc((void **)&tA, cytnx_uint64(L) * L * sizeof(cytnx_double)));
        checkCudaErrors(hipMemcpy(tA, in->Mem, sizeof(cytnx_double) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      }

      // query buffer:
      cytnx_int32 lwork = 0;
      cytnx_int32 b32L = L;
      checkCudaErrors(hipsolverDnDsyevd_bufferSize(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L, tA,
                                                  b32L, (cytnx_double *)e->Mem, &lwork));

      // allocate working space:
      cytnx_double *work;
      checkCudaErrors(hipMalloc((void **)&work, sizeof(cytnx_double) * lwork));

      // call :
      cytnx_int32 info;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipsolverDnDsyevd(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L, tA, b32L,
                                       (cytnx_double *)e->Mem, work, lwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'cusolverDnDsysevd': cuBlas INFO = ", info);

      hipFree(work);
      if (v->dtype == Type.Void) hipFree(tA);

      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }
    void cuEigh_internal_f(const boost::intrusive_ptr<Storage_base> &in,
                           boost::intrusive_ptr<Storage_base> &e,
                           boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
      if (v->dtype == Type.Void) jobz = HIPSOLVER_EIG_MODE_NOVECTOR;

      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_float *tA;
      if (v->dtype != Type.Void) {
        tA = (cytnx_float *)v->Mem;
        checkCudaErrors(hipMemcpy(v->Mem, in->Mem, sizeof(cytnx_float) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      } else {
        checkCudaErrors(hipMalloc((void **)&tA, cytnx_uint64(L) * L * sizeof(cytnx_float)));
        checkCudaErrors(hipMemcpy(tA, in->Mem, sizeof(cytnx_float) * cytnx_uint64(L) * L,
                                   hipMemcpyDeviceToDevice));
      }

      // query buffer:
      cytnx_int32 lwork = 0;
      cytnx_int32 b32L = L;
      checkCudaErrors(hipsolverDnSsyevd_bufferSize(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L, tA,
                                                  b32L, (cytnx_float *)e->Mem, &lwork));

      // allocate working space:
      cytnx_float *work;
      checkCudaErrors(hipMalloc((void **)&work, sizeof(cytnx_float) * lwork));

      // call :
      cytnx_int32 info;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      checkCudaErrors(hipsolverDnSsyevd(cusolverH, jobz, HIPBLAS_FILL_MODE_UPPER, b32L, tA, b32L,
                                       (cytnx_float *)e->Mem, work, lwork, devinfo));

      // get info
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));
      cytnx_error_msg(info != 0, "%s %d",
                      "Error in cuBlas function 'cusolverDnDsysevd': cuBlas INFO = ", info);

      hipFree(work);
      if (v->dtype == Type.Void) hipFree(tA);

      hipFree(devinfo);
      hipsolverDnDestroy(cusolverH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
