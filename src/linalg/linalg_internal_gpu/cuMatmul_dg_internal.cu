#include "hip/hip_runtime.h"
#include "cuMatmul_dg_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

// this use dgmm

namespace cytnx {

  namespace linalg_internal {

    template <typename UniType>
    __global__ void cuMatMul_dg_kernel_diagL(UniType *out, const UniType *inl, const UniType *inr,
                                             cytnx_int64 Ml, cytnx_int64 Comm, cytnx_int64 Nr) {
      // UniType tmp=0;
      cytnx_uint64 sid = blockIdx.x * blockDim.x + threadIdx.x;
      if (sid < cytnx_uint64(Ml) * Nr) {
        out[sid] = inl[(sid / Nr)] * inr[sid];
      }
    }

    template <typename UniType>
    __global__ void cuMatMul_dg_kernel_diagR(UniType *out, const UniType *inl, const UniType *inr,
                                             cytnx_int64 Ml, cytnx_int64 Comm, cytnx_int64 Nr) {
      // UniType tmp=0;
      cytnx_uint64 sid = blockIdx.x * blockDim.x + threadIdx.x;
      if (sid < cytnx_uint64(Ml) * Nr) {
        out[sid] = inl[sid] * inr[sid % Nr];
      }
    }

    /// cuMatmul_dg
    void cuMatmul_dg_internal_cd(boost::intrusive_ptr<Storage_base> &out,
                                 const boost::intrusive_ptr<Storage_base> &inl,
                                 const boost::intrusive_ptr<Storage_base> &inr,
                                 const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                 const cytnx_int64 &Nr, const int &diag_L) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      // cytnx_complex128 alpha = cytnx_complex128(1,0), beta=cytnx_complex128(0,0);

      hipDoubleComplex *_out = (hipDoubleComplex *)out->Mem;
      hipDoubleComplex *_inl = (hipDoubleComplex *)inl->Mem;
      hipDoubleComplex *_inr = (hipDoubleComplex *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;  //, blsComm=Comm;
      if (diag_L)
        checkCudaErrors(
          hipblasZdgmm(cublasH, HIPBLAS_SIDE_RIGHT, blsNr, blsMl, _inr, blsNr, _inl, 1, _out, blsNr));
      else
        checkCudaErrors(
          hipblasZdgmm(cublasH, HIPBLAS_SIDE_LEFT, blsNr, blsMl, _inl, blsNr, _inr, 1, _out, blsNr));

      hipblasDestroy(cublasH);
    }
    void cuMatmul_dg_internal_cf(boost::intrusive_ptr<Storage_base> &out,
                                 const boost::intrusive_ptr<Storage_base> &inl,
                                 const boost::intrusive_ptr<Storage_base> &inr,
                                 const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                 const cytnx_int64 &Nr, const int &diag_L) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      // cytnx_complex64 alpha = cytnx_complex64(1,0), beta=cytnx_complex64(0,0);

      hipFloatComplex *_out = (hipFloatComplex *)out->Mem;
      hipFloatComplex *_inl = (hipFloatComplex *)inl->Mem;
      hipFloatComplex *_inr = (hipFloatComplex *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;  //, blsComm=Comm;
      if (diag_L)
        checkCudaErrors(
          hipblasCdgmm(cublasH, HIPBLAS_SIDE_RIGHT, blsNr, blsMl, _inr, blsNr, _inl, 1, _out, blsNr));
      else
        checkCudaErrors(
          hipblasCdgmm(cublasH, HIPBLAS_SIDE_LEFT, blsNr, blsMl, _inl, blsNr, _inr, 1, _out, blsNr));

      hipblasDestroy(cublasH);
    }

    void cuMatmul_dg_internal_d(boost::intrusive_ptr<Storage_base> &out,
                                const boost::intrusive_ptr<Storage_base> &inl,
                                const boost::intrusive_ptr<Storage_base> &inr,
                                const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                const cytnx_int64 &Nr, const int &diag_L) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      // cytnx_double alpha = 1, beta=0;

      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_inl = (cytnx_double *)inl->Mem;
      cytnx_double *_inr = (cytnx_double *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;  //, blsComm=Comm;
      // std::cout << blsMl << " " << blsNr << " " << blsComm << std::endl;
      // std::cout << diag_L << std::endl;

      if (diag_L)
        checkCudaErrors(
          hipblasDdgmm(cublasH, HIPBLAS_SIDE_RIGHT, blsNr, blsMl, _inr, blsNr, _inl, 1, _out, blsNr));
      else
        checkCudaErrors(
          hipblasDdgmm(cublasH, HIPBLAS_SIDE_LEFT, blsNr, blsMl, _inl, blsNr, _inr, 1, _out, blsNr));

      hipblasDestroy(cublasH);
    }
    void cuMatmul_dg_internal_f(boost::intrusive_ptr<Storage_base> &out,
                                const boost::intrusive_ptr<Storage_base> &inl,
                                const boost::intrusive_ptr<Storage_base> &inr,
                                const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                const cytnx_int64 &Nr, const int &diag_L) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      // cytnx_float alpha = 1, beta=0;

      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_inl = (cytnx_float *)inl->Mem;
      cytnx_float *_inr = (cytnx_float *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;  //, blsComm=Comm;
      if (diag_L)
        checkCudaErrors(
          hipblasSdgmm(cublasH, HIPBLAS_SIDE_RIGHT, blsNr, blsMl, _inr, blsNr, _inl, 1, _out, blsNr));
      else
        checkCudaErrors(
          hipblasSdgmm(cublasH, HIPBLAS_SIDE_LEFT, blsNr, blsMl, _inl, blsNr, _inr, 1, _out, blsNr));

      hipblasDestroy(cublasH);
    }
    void cuMatmul_dg_internal_i64(boost::intrusive_ptr<Storage_base> &out,
                                  const boost::intrusive_ptr<Storage_base> &inl,
                                  const boost::intrusive_ptr<Storage_base> &inr,
                                  const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                  const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_inl = (cytnx_int64 *)inl->Mem;
      cytnx_int64 *_inr = (cytnx_int64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;
      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }
    void cuMatmul_dg_internal_u64(boost::intrusive_ptr<Storage_base> &out,
                                  const boost::intrusive_ptr<Storage_base> &inl,
                                  const boost::intrusive_ptr<Storage_base> &inr,
                                  const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                  const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_inl = (cytnx_uint64 *)inl->Mem;
      cytnx_uint64 *_inr = (cytnx_uint64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;

      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }
    void cuMatmul_dg_internal_i32(boost::intrusive_ptr<Storage_base> &out,
                                  const boost::intrusive_ptr<Storage_base> &inl,
                                  const boost::intrusive_ptr<Storage_base> &inr,
                                  const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                  const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_inl = (cytnx_int64 *)inl->Mem;
      cytnx_int64 *_inr = (cytnx_int64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;

      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }
    void cuMatmul_dg_internal_u32(boost::intrusive_ptr<Storage_base> &out,
                                  const boost::intrusive_ptr<Storage_base> &inl,
                                  const boost::intrusive_ptr<Storage_base> &inr,
                                  const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                  const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_inl = (cytnx_uint64 *)inl->Mem;
      cytnx_uint64 *_inr = (cytnx_uint64 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;

      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }
    void cuMatmul_dg_internal_i16(boost::intrusive_ptr<Storage_base> &out,
                                  const boost::intrusive_ptr<Storage_base> &inl,
                                  const boost::intrusive_ptr<Storage_base> &inr,
                                  const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                  const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_int16 *_inl = (cytnx_int16 *)inl->Mem;
      cytnx_int16 *_inr = (cytnx_int16 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;

      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }
    void cuMatmul_dg_internal_u16(boost::intrusive_ptr<Storage_base> &out,
                                  const boost::intrusive_ptr<Storage_base> &inl,
                                  const boost::intrusive_ptr<Storage_base> &inr,
                                  const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                  const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_uint16 *_out = (cytnx_uint16 *)out->Mem;
      cytnx_uint16 *_inl = (cytnx_uint16 *)inl->Mem;
      cytnx_uint16 *_inr = (cytnx_uint16 *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;

      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }
    void cuMatmul_dg_internal_b(boost::intrusive_ptr<Storage_base> &out,
                                const boost::intrusive_ptr<Storage_base> &inl,
                                const boost::intrusive_ptr<Storage_base> &inr,
                                const cytnx_int64 &Ml, const cytnx_int64 &Comm,
                                const cytnx_int64 &Nr, const int &diag_L) {
      cytnx_bool *_out = (cytnx_bool *)out->Mem;
      cytnx_bool *_inl = (cytnx_bool *)inl->Mem;
      cytnx_bool *_inr = (cytnx_bool *)inr->Mem;

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml) * Nr) / 512;
      if ((cytnx_uint64(Ml) * Nr) % 512) Nblocks += 1;

      if (diag_L)
        cuMatMul_dg_kernel_diagL<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
      else
        cuMatMul_dg_kernel_diagR<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Comm, Nr);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
