#include "hip/hip_runtime.h"
#include "cuGetElems_gpu.hpp"

namespace cytnx {
  namespace utils_internal {

    template <class T>
    __global__ void cuGetElems_kernel(T* d_out, T* d_in, cytnx_uint64* offj, cytnx_uint64* new_offj,
                                      cytnx_uint64* locators, cytnx_uint64* picksize,
                                      cytnx_uint64 rank, cytnx_uint64 TotalElem) {
      if (blockIdx.x * blockDim.x + threadIdx.x < TotalElem) {
        cytnx_uint64 Loc = 0;
        cytnx_uint64 tmpn = blockIdx.x * blockDim.x + threadIdx.x;
        cytnx_uint64 offset = 0;
        for (cytnx_uint32 r = 0; r < rank; r++) {
          if (picksize[r])
            Loc += locators[offset + cytnx_uint64(tmpn / new_offj[r])] * offj[r];
          else
            Loc += cytnx_uint64(tmpn / new_offj[r]) * offj[r];
          tmpn %= new_offj[r];
          offset += picksize[r];
        }
        d_out[blockIdx.x * blockDim.x + threadIdx.x] = d_in[Loc];
      }
    }

    void cuGetElems_gpu_cd(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                           const std::vector<cytnx_uint64>& new_offj,
                           const std::vector<std::vector<cytnx_uint64>>& locators,
                           const cytnx_uint64& TotalElem) {
      // Start copy elem:
      hipDoubleComplex* elem_ptr_ = static_cast<hipDoubleComplex*>(in);
      hipDoubleComplex* new_elem_ptr_ = static_cast<hipDoubleComplex*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_cf(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                           const std::vector<cytnx_uint64>& new_offj,
                           const std::vector<std::vector<cytnx_uint64>>& locators,
                           const cytnx_uint64& TotalElem) {
      // Start copy elem:
      hipFloatComplex* elem_ptr_ = static_cast<hipFloatComplex*>(in);
      hipFloatComplex* new_elem_ptr_ = static_cast<hipFloatComplex*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_d(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                          const std::vector<cytnx_uint64>& new_offj,
                          const std::vector<std::vector<cytnx_uint64>>& locators,
                          const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_double* elem_ptr_ = static_cast<cytnx_double*>(in);
      cytnx_double* new_elem_ptr_ = static_cast<cytnx_double*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_f(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                          const std::vector<cytnx_uint64>& new_offj,
                          const std::vector<std::vector<cytnx_uint64>>& locators,
                          const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_float* elem_ptr_ = static_cast<cytnx_float*>(in);
      cytnx_float* new_elem_ptr_ = static_cast<cytnx_float*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_i64(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                            const std::vector<cytnx_uint64>& new_offj,
                            const std::vector<std::vector<cytnx_uint64>>& locators,
                            const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_int64* elem_ptr_ = static_cast<cytnx_int64*>(in);
      cytnx_int64* new_elem_ptr_ = static_cast<cytnx_int64*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_u64(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                            const std::vector<cytnx_uint64>& new_offj,
                            const std::vector<std::vector<cytnx_uint64>>& locators,
                            const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_uint64* elem_ptr_ = static_cast<cytnx_uint64*>(in);
      cytnx_uint64* new_elem_ptr_ = static_cast<cytnx_uint64*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_i32(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                            const std::vector<cytnx_uint64>& new_offj,
                            const std::vector<std::vector<cytnx_uint64>>& locators,
                            const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_int32* elem_ptr_ = static_cast<cytnx_int32*>(in);
      cytnx_int32* new_elem_ptr_ = static_cast<cytnx_int32*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_u32(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                            const std::vector<cytnx_uint64>& new_offj,
                            const std::vector<std::vector<cytnx_uint64>>& locators,
                            const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_uint32* elem_ptr_ = static_cast<cytnx_uint32*>(in);
      cytnx_uint32* new_elem_ptr_ = static_cast<cytnx_uint32*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_i16(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                            const std::vector<cytnx_uint64>& new_offj,
                            const std::vector<std::vector<cytnx_uint64>>& locators,
                            const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_int16* elem_ptr_ = static_cast<cytnx_int16*>(in);
      cytnx_int16* new_elem_ptr_ = static_cast<cytnx_int16*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_u16(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                            const std::vector<cytnx_uint64>& new_offj,
                            const std::vector<std::vector<cytnx_uint64>>& locators,
                            const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_uint16* elem_ptr_ = static_cast<cytnx_uint16*>(in);
      cytnx_uint16* new_elem_ptr_ = static_cast<cytnx_uint16*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }
    void cuGetElems_gpu_b(void* out, void* in, const std::vector<cytnx_uint64>& offj,
                          const std::vector<cytnx_uint64>& new_offj,
                          const std::vector<std::vector<cytnx_uint64>>& locators,
                          const cytnx_uint64& TotalElem) {
      // Start copy elem:
      cytnx_bool* elem_ptr_ = static_cast<cytnx_bool*>(in);
      cytnx_bool* new_elem_ptr_ = static_cast<cytnx_bool*>(out);

      // create on device:
      cytnx_uint64* d_offj;
      checkCudaErrors(hipMalloc((void**)&d_offj, sizeof(cytnx_uint64) * offj.size()));
      checkCudaErrors(
        hipMemcpy(d_offj, &offj[0], sizeof(cytnx_uint64) * offj.size(), hipMemcpyHostToDevice));

      cytnx_uint64* d_new_offj;
      checkCudaErrors(hipMalloc((void**)&d_new_offj, sizeof(cytnx_uint64) * new_offj.size()));
      checkCudaErrors(hipMemcpy(d_new_offj, &new_offj[0], sizeof(cytnx_uint64) * new_offj.size(),
                                 hipMemcpyHostToDevice));

      std::vector<cytnx_uint64> composit_locators, picksize;
      cytnx_uint64 Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        picksize.push_back(locators[i].size());
        Nte += locators[i].size();
      }
      composit_locators.resize(Nte);
      Nte = 0;
      for (cytnx_uint32 i = 0; i < locators.size(); i++) {
        memcpy(&composit_locators[Nte], &(locators[i][0]),
               sizeof(cytnx_uint64) * locators[i].size());
        Nte += locators[i].size();
      }

      cytnx_uint64* d_locators;
      checkCudaErrors(
        hipMalloc((void**)&d_locators, sizeof(cytnx_uint64) * composit_locators.size()));
      checkCudaErrors(hipMemcpy(d_locators, &composit_locators[0],
                                 sizeof(cytnx_uint64) * composit_locators.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64* d_picksize;
      checkCudaErrors(hipMalloc((void**)&d_picksize, sizeof(cytnx_uint64) * picksize.size()));
      checkCudaErrors(hipMemcpy(d_picksize, &picksize[0], sizeof(cytnx_uint64) * picksize.size(),
                                 hipMemcpyHostToDevice));

      cytnx_uint64 NBlocks = TotalElem / 256;
      if (TotalElem % 256) NBlocks += 1;
      cuGetElems_kernel<<<NBlocks, 256>>>(new_elem_ptr_, elem_ptr_, d_offj, d_new_offj, d_locators,
                                          d_picksize, offj.size(), TotalElem);

      hipFree(d_offj);
      hipFree(d_new_offj);
      hipFree(d_locators);
      hipFree(d_picksize);
    }

  }  // namespace utils_internal
}  // namespace cytnx
