#include "hip/hip_runtime.h"
#include "cuMovemem_gpu.hpp"
#include "cuAlloc_gpu.hpp"
#include "backend/Storage.hpp"
#include <algorithm>
#include "utils/vec_print.hpp"

#ifdef UNI_GPU
  #ifdef UNI_CUTT
    #include "cutt.h"
  #endif

  #ifdef UNI_CUTENSOR
    #include "hiptensor.h"
  #endif

#endif

using namespace std;

namespace cytnx {
  namespace utils_internal {

#ifdef UNI_GPU
    template <class BidirectionalIterator>
    void reverse_perm(BidirectionalIterator first, BidirectionalIterator last, int N) {
      while ((first != last) && (first != --last)) {
        *first = (N - 1) - *first;
        *last = (N - 1) - *last;
        std::iter_swap(first, last);
        ++first;
      }
      if (N % 2) *first = (N - 1) - *first;
    }

    template <class T>
    __global__ void cuMovemem_kernel(T *ddes, T *dsrc, cytnx_uint64 *accu_old,
                                     cytnx_uint64 *permuted_accu_new, cytnx_uint32 rank,
                                     cytnx_uint64 Nelem) {
      extern __shared__ cytnx_uint64 SHaccu[];

      cytnx_uint64 ids;
      /// copy to share mem:
      if (rank <= blockDim.x) {
        if (threadIdx.x < rank) {
          SHaccu[threadIdx.x] = accu_old[threadIdx.x];
          SHaccu[threadIdx.x + rank] = permuted_accu_new[threadIdx.x];
        }
      } else {
        cytnx_uint32 Np = rank / blockDim.x;
        if (rank % blockDim.x) Np += 1;
        for (cytnx_uint32 i = 0; i < Np; i++) {
          ids = i * blockDim.x + threadIdx.x;
          if (ids < rank) {
            SHaccu[ids] = accu_old[ids];
            SHaccu[ids + rank] = permuted_accu_new[ids];
          }
        }
      }
      __syncthreads();

      cytnx_uint64 tid = blockIdx.x * blockDim.x + threadIdx.x;
      ids = 0;
      for (cytnx_uint32 i = 0; i < rank; i++) {
        ids += (tid / SHaccu[i]) * SHaccu[rank + i];
        tid = tid % SHaccu[i];
      }
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem)
        ddes[ids] = dsrc[blockIdx.x * blockDim.x + threadIdx.x];
    }

    // T is the cytnx type, cuT is the cuda type. For all types they should be the same except for
    // hipDoubleComplex and hipFloatComplex.
    template <class T, class cuT>
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_general(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
  #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
  #endif

      std::vector<cytnx_uint64> newshape(old_shape.size());
      for (cytnx_uint64 i = 0; i < old_shape.size(); i++) newshape[i] = old_shape[mapper[i]];

      std::vector<cytnx_uint64> shifter_old(old_shape.size());
      std::vector<cytnx_uint64> shifter_new(old_shape.size());

      cytnx_uint64 accu_old = 1, accu_new = 1;
      for (cytnx_int64 i = old_shape.size() - 1; i >= 0; i--) {
        shifter_old[i] = accu_old;
        shifter_new[i] = accu_new;
        accu_old *= old_shape[i];
        accu_new *= newshape[i];
      }
      std::vector<cytnx_uint64> old_inds(old_shape.size());

      std::vector<cytnx_uint64> permuted_shifter_new(old_shape.size());
      for (unsigned int i = 0; i < old_shape.size(); i++)
        permuted_shifter_new[i] = shifter_new[invmapper[i]];

      /// allocate a GPU for psn-vec/so-vec/tmp des-vec
      cytnx_uint64 *dshifter_old, *dperm_shifter_new;
      cuT *dtmp;
      cytnx_uint64 Nelem = accu_old;

      hipSetDevice(in->device);  // ensure the following allocation on the same device as src.
      checkCudaErrors(
        hipMalloc((void **)&dshifter_old, sizeof(cytnx_uint64) * shifter_old.size()));
      checkCudaErrors(hipMalloc((void **)&dperm_shifter_new,
                                 sizeof(cytnx_uint64) * permuted_shifter_new.size()));
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->cap);

      /// copy psn-vec/so-vec to device
      checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0],
                                 sizeof(cytnx_uint64) * permuted_shifter_new.size(),
                                 hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0],
                                 sizeof(cytnx_uint64) * shifter_old.size(),
                                 hipMemcpyHostToDevice));

      /// calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
      cytnx_uint64 NBlocks = Nelem / 256;
      if (Nelem % 256) {
        NBlocks += 1;
      }
      cuMovemem_kernel<<<NBlocks, 256, shifter_old.size() * 2 * sizeof(cytnx_uint64)>>>(
        dtmp, (cuT *)in->Mem, dshifter_old, dperm_shifter_new, old_shape.size(), Nelem);

      /// house keeping:
      checkCudaErrors(hipFree(dshifter_old));
      checkCudaErrors(hipFree(dperm_shifter_new));

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->Mem, dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipFree(dtmp));
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device, true, in->cap);
        return out;
      }
    }

  #ifdef UNI_CUTT
    template <class T, class cuT>
    boost::intrusive_ptr<Storage_base> cuMovemem_cutt_gpu(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
    #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_cutt with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
    #endif

      cuT *dtmp;
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->cap);
      cytnx_uint64 Nelem = in->len;

      std::vector<int> perm(mapper.begin(), mapper.end());
      std::vector<int> size(old_shape.begin(), old_shape.end());
      std::reverse(size.begin(), size.end());  // matching API CUTT
      reverse_perm(perm.begin(), perm.end(), perm.size());  // matching API CUTT

      cuttHandle plan;
      cuttPlan(&plan, perm.size(), size.data(), perm.data(), sizeof(cuT), 0);
      cuttExecute(plan, in->Mem, dtmp);

      cuttDestroy(plan);

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->Mem, dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device, true, in->cap);
        return out;
      }
    }
  #endif

  #ifdef UNI_CUTENSOR
    template <class T, class cuT>  // T: cpu type, cuT: gpu type, cutnT: cntensor type
    boost::intrusive_ptr<Storage_base> cuMovemem_cutensor_gpu(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace, hiptensorDataType_t type_in, hiptensorDataType_t type_out,
      const hiptensorComputeDescriptor_t descCompute, const cuT &ONE) {
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
    #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_cutt with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
    #endif

      cuT *dtmp;
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->cap);
      cytnx_uint64 Nelem = in->len;

      std::vector<int> perm(mapper.begin(), mapper.end());
      std::vector<int64_t> size(old_shape.begin(), old_shape.end());
      std::vector<int> ori(perm.size());
      for (int i = 0; i < ori.size(); i++) ori[i] = i;

      std::vector<int64_t> new_size(perm.size());
      for (int i = 0; i < new_size.size(); i++) {
        new_size[i] = size[perm[i]];
      }
      std::reverse(size.begin(), size.end());  // matching API
      std::reverse(perm.begin(), perm.end());  // matching API
      std::reverse(new_size.begin(), new_size.end());  // matching API
      std::reverse(ori.begin(), ori.end());  // matching API

      hiptensorHandle_t handle;
      checkCudaErrors(hiptensorCreate(&handle));

      // This is the default alignment of hipMalloc() and may also be the default alignment of
      // hipMallocManaged()
      cytnx_uint64 defaultAlignment = 256;
      hiptensorTensorDescriptor_t descA;
      checkCudaErrors(hiptensorCreateTensorDescriptor(handle, &descA, size.size(), size.data(),
                                                     NULL /* stride */, type_in, defaultAlignment));

      hiptensorTensorDescriptor_t descC;
      checkCudaErrors(hiptensorCreateTensorDescriptor(handle, &descC, new_size.size(),
                                                     new_size.data(), NULL /* stride */, type_out,
                                                     defaultAlignment));
      // TODO: verify the type of ONE matches descCompute
      hiptensorOperationDescriptor_t desc;
      checkCudaErrors(hiptensorCreatePermutation(
        handle, &desc, descA, ori.data(), HIPTENSOR_OP_IDENTITY, descC, perm.data(), descCompute));

      const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

      hiptensorPlanPreference_t planPref;
      checkCudaErrors(
        hiptensorCreatePlanPreference(handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE));

      hiptensorPlan_t plan;
      checkCudaErrors(
        hiptensorCreatePlan(handle, &plan, desc, planPref, 0 /* workspaceSizeLimit */));

      checkCudaErrors(hiptensorPermute(handle, plan, &ONE, (cuT *)in->Mem, dtmp, 0 /* stream */));

      checkCudaErrors(hiptensorDestroyTensorDescriptor(descA));
      checkCudaErrors(hiptensorDestroyTensorDescriptor(descC));
      checkCudaErrors(hiptensorDestroyPlanPreference(planPref));
      checkCudaErrors(hiptensorDestroyPlan(plan));
      checkCudaErrors(hiptensorDestroy(handle));

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->Mem, dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device, true, in->cap);
        return out;
      }
    }
  #endif

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_cd(boost::intrusive_ptr<Storage_base> &in,
                                                        const std::vector<cytnx_uint64> &old_shape,
                                                        const std::vector<cytnx_uint64> &mapper,
                                                        const std::vector<cytnx_uint64> &invmapper,
                                                        const bool is_inplace) {
  #ifdef UNI_CUTENSOR
      return cuMovemem_cutensor_gpu<cytnx_complex128, hipDoubleComplex>(
        in, old_shape, mapper, invmapper, is_inplace, HIPTENSOR_C_64F, HIPTENSOR_C_64F,
        HIPTENSOR_COMPUTE_DESC_64F, make_hipDoubleComplex(1, 0));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_complex128, hipDoubleComplex>(in, old_shape, mapper, invmapper,
                                                                   is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_complex128, hipDoubleComplex>(in, old_shape, mapper,
                                                                      invmapper, is_inplace);
  #endif
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_cf(boost::intrusive_ptr<Storage_base> &in,
                                                        const std::vector<cytnx_uint64> &old_shape,
                                                        const std::vector<cytnx_uint64> &mapper,
                                                        const std::vector<cytnx_uint64> &invmapper,
                                                        const bool is_inplace) {
  #if defined(UNI_CUTENSOR)
      return cuMovemem_cutensor_gpu<cytnx_complex64, hipFloatComplex>(
        in, old_shape, mapper, invmapper, is_inplace, HIPTENSOR_C_32F, HIPTENSOR_C_32F,
        HIPTENSOR_COMPUTE_DESC_32F, make_hipFloatComplex(1, 0));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_complex64, hipFloatComplex>(in, old_shape, mapper, invmapper,
                                                                 is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_complex64, hipFloatComplex>(in, old_shape, mapper,
                                                                    invmapper, is_inplace);
  #endif
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_d(boost::intrusive_ptr<Storage_base> &in,
                                                       const std::vector<cytnx_uint64> &old_shape,
                                                       const std::vector<cytnx_uint64> &mapper,
                                                       const std::vector<cytnx_uint64> &invmapper,
                                                       const bool is_inplace) {
  #if defined(UNI_CUTENSOR)
      return cuMovemem_cutensor_gpu<double, double>(in, old_shape, mapper, invmapper, is_inplace,
                                                    HIPTENSOR_R_64F, HIPTENSOR_R_64F,
                                                    HIPTENSOR_COMPUTE_DESC_64F, double(1));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_double, cytnx_double>(in, old_shape, mapper, invmapper,
                                                            is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_double, cytnx_double>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
  #endif
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_f(boost::intrusive_ptr<Storage_base> &in,
                                                       const std::vector<cytnx_uint64> &old_shape,
                                                       const std::vector<cytnx_uint64> &mapper,
                                                       const std::vector<cytnx_uint64> &invmapper,
                                                       const bool is_inplace) {
  #if defined(UNI_CUTENSOR)
      return cuMovemem_cutensor_gpu<float, float>(in, old_shape, mapper, invmapper, is_inplace,
                                                  HIPTENSOR_R_32F, HIPTENSOR_R_32F,
                                                  HIPTENSOR_COMPUTE_DESC_32F, float(1));
  #elif defined(UNI_CUTT)
      return cuMovemem_cutt_gpu<cytnx_float, cytnx_float>(in, old_shape, mapper, invmapper,
                                                          is_inplace);
  #else
      return cuMovemem_gpu_general<cytnx_float, cytnx_float>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
  #endif
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i64(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_int64, cytnx_int64>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u64(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_uint64, cytnx_uint64>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i32(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_int32, cytnx_int32>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
    }

    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u32(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_uint32, cytnx_uint32>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_u16(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_uint16, cytnx_uint16>(in, old_shape, mapper, invmapper,
                                                               is_inplace);
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_i16(boost::intrusive_ptr<Storage_base> &in,
                                                         const std::vector<cytnx_uint64> &old_shape,
                                                         const std::vector<cytnx_uint64> &mapper,
                                                         const std::vector<cytnx_uint64> &invmapper,
                                                         const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_int16, cytnx_int16>(in, old_shape, mapper, invmapper,
                                                             is_inplace);
    }
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_b(boost::intrusive_ptr<Storage_base> &in,
                                                       const std::vector<cytnx_uint64> &old_shape,
                                                       const std::vector<cytnx_uint64> &mapper,
                                                       const std::vector<cytnx_uint64> &invmapper,
                                                       const bool is_inplace) {
      return cuMovemem_gpu_general<cytnx_bool, cytnx_bool>(in, old_shape, mapper, invmapper,
                                                           is_inplace);
    }

#endif  // UNI_GPU
  }  // namespace utils_internal
}  // namespace cytnx
