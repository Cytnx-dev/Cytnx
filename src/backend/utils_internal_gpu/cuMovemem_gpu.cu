#include "hip/hip_runtime.h"
#include "cuMovemem_gpu.hpp"

#include <algorithm>
#include <vector>
#include <type_traits>

#include "boost/smart_ptr/intrusive_ptr.hpp"
#include "hip/hip_runtime_api.h"

#include "backend/Storage.hpp"
#include "cuAlloc_gpu.hpp"
#include "Type.hpp"

#ifdef UNI_GPU
  #ifdef UNI_CUTT
    #include "cutt.h"
  #endif

  #ifdef UNI_CUTENSOR
    #include "hiptensor.h"
  #endif

#endif

using namespace std;

namespace cytnx {
  namespace utils_internal {

#ifdef UNI_GPU

    /**
     * A helper class that retrieves corresponding complex type defined in CUDA for the complex
     * dtype.
     */
    template <class DType>
    struct ToCudaTypeMap {
      typedef DType type;
    };

    template <>
    struct ToCudaTypeMap<cytnx_complex128> {
      typedef hipDoubleComplex type;
    };
    template <>
    struct ToCudaTypeMap<cytnx_complex64> {
      typedef hipFloatComplex type;
    };

    template <class DType>
    using ToCudaType = typename ToCudaTypeMap<DType>::type;

    template <class BidirectionalIterator>
    void reverse_perm(BidirectionalIterator first, BidirectionalIterator last, int N) {
      while ((first != last) && (first != --last)) {
        *first = (N - 1) - *first;
        *last = (N - 1) - *last;
        std::iter_swap(first, last);
        ++first;
      }
      if (N % 2) *first = (N - 1) - *first;
    }

    template <class T>
    __global__ void cuMovemem_kernel(T *ddes, T *dsrc, cytnx_uint64 *accu_old,
                                     cytnx_uint64 *permuted_accu_new, cytnx_uint32 rank,
                                     cytnx_uint64 Nelem) {
      extern __shared__ cytnx_uint64 SHaccu[];

      cytnx_uint64 ids;
      /// copy to share mem:
      if (rank <= blockDim.x) {
        if (threadIdx.x < rank) {
          SHaccu[threadIdx.x] = accu_old[threadIdx.x];
          SHaccu[threadIdx.x + rank] = permuted_accu_new[threadIdx.x];
        }
      } else {
        cytnx_uint32 Np = rank / blockDim.x;
        if (rank % blockDim.x) Np += 1;
        for (cytnx_uint32 i = 0; i < Np; i++) {
          ids = i * blockDim.x + threadIdx.x;
          if (ids < rank) {
            SHaccu[ids] = accu_old[ids];
            SHaccu[ids + rank] = permuted_accu_new[ids];
          }
        }
      }
      __syncthreads();

      cytnx_uint64 tid = blockIdx.x * blockDim.x + threadIdx.x;
      ids = 0;
      for (cytnx_uint32 i = 0; i < rank; i++) {
        ids += (tid / SHaccu[i]) * SHaccu[rank + i];
        tid = tid % SHaccu[i];
      }
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem)
        ddes[ids] = dsrc[blockIdx.x * blockDim.x + threadIdx.x];
    }

    // T is the cytnx type, cuT is the cuda type. For all types they should be the same except for
    // hipDoubleComplex and hipFloatComplex.
    template <class T>
    boost::intrusive_ptr<Storage_base> cuMovemem_gpu_general(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      using cuT = ToCudaType<T>;
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
  #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype() != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_gpu with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device() == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
  #endif

      std::vector<cytnx_uint64> newshape(old_shape.size());
      for (cytnx_uint64 i = 0; i < old_shape.size(); i++) newshape[i] = old_shape[mapper[i]];

      std::vector<cytnx_uint64> shifter_old(old_shape.size());
      std::vector<cytnx_uint64> shifter_new(old_shape.size());

      cytnx_uint64 accu_old = 1, accu_new = 1;
      for (cytnx_int64 i = old_shape.size() - 1; i >= 0; i--) {
        shifter_old[i] = accu_old;
        shifter_new[i] = accu_new;
        accu_old *= old_shape[i];
        accu_new *= newshape[i];
      }
      std::vector<cytnx_uint64> old_inds(old_shape.size());

      std::vector<cytnx_uint64> permuted_shifter_new(old_shape.size());
      for (unsigned int i = 0; i < old_shape.size(); i++)
        permuted_shifter_new[i] = shifter_new[invmapper[i]];

      /// allocate a GPU for psn-vec/so-vec/tmp des-vec
      cytnx_uint64 *dshifter_old, *dperm_shifter_new;
      cuT *dtmp;
      cytnx_uint64 Nelem = accu_old;

      hipSetDevice(in->device());  // ensure the following allocation on the same device as src.
      checkCudaErrors(
        hipMalloc((void **)&dshifter_old, sizeof(cytnx_uint64) * shifter_old.size()));
      checkCudaErrors(hipMalloc((void **)&dperm_shifter_new,
                                 sizeof(cytnx_uint64) * permuted_shifter_new.size()));
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->capacity());

      /// copy psn-vec/so-vec to device
      checkCudaErrors(hipMemcpy(dperm_shifter_new, &permuted_shifter_new[0],
                                 sizeof(cytnx_uint64) * permuted_shifter_new.size(),
                                 hipMemcpyHostToDevice));
      checkCudaErrors(hipMemcpy(dshifter_old, &shifter_old[0],
                                 sizeof(cytnx_uint64) * shifter_old.size(),
                                 hipMemcpyHostToDevice));

      /// calculate how many blocks, and shared mem size, thpb fixed at 256 (need fine tune)
      cytnx_uint64 NBlocks = Nelem / 256;
      if (Nelem % 256) {
        NBlocks += 1;
      }
      cuMovemem_kernel<<<NBlocks, 256, shifter_old.size() * 2 * sizeof(cytnx_uint64)>>>(
        dtmp, (cuT *)in->data(), dshifter_old, dperm_shifter_new, old_shape.size(), Nelem);

      /// house keeping:
      checkCudaErrors(hipFree(dshifter_old));
      checkCudaErrors(hipFree(dperm_shifter_new));

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->data(), dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        checkCudaErrors(hipFree(dtmp));
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device(), true, in->capacity());
        return out;
      }
    }

  #ifdef UNI_CUTT
    template <class T>
    boost::intrusive_ptr<Storage_base> cuMovemem_cutt_gpu(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      using cuT = ToCudaType<T>;
      T proxy;
      unsigned int dtype_T = Type_class::cy_typeid(proxy);
    #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype() != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_cutt with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device() == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
    #endif

      cuT *dtmp;
      dtmp = (cuT *)cuMalloc_gpu(sizeof(cuT) * in->capacity());
      cytnx_uint64 Nelem = in->size();
      std::vector<int> perm(mapper.begin(), mapper.end());
      std::vector<int> size(old_shape.begin(), old_shape.end());
      std::reverse(size.begin(), size.end());  // matching API CUTT
      reverse_perm(perm.begin(), perm.end(), perm.size());  // matching API CUTT

      cuttHandle plan;
      cuttPlan(&plan, perm.size(), size.data(), perm.data(), sizeof(cuT), 0);
      cuttExecute(plan, in->data(), dtmp);

      cuttDestroy(plan);

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(hipMemcpy(in->data(), dtmp, sizeof(T) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return in;
      } else {
        out->_Init_byptr(dtmp, Nelem, in->device(), true, in->capacity());
        return out;
      }
    }
  #endif

  #ifdef UNI_CUTENSOR
    template <class DType>
    boost::intrusive_ptr<Storage_base> cuMovemem_cutensor_gpu(
      boost::intrusive_ptr<Storage_base> &in, const std::vector<cytnx_uint64> &old_shape,
      const std::vector<cytnx_uint64> &mapper, const std::vector<cytnx_uint64> &invmapper,
      const bool is_inplace) {
      using CudaType = ToCudaType<DType>;

      unsigned int dtype_T = Type_class::cy_typeid(DType());
    #ifdef UNI_DEBUG
      cytnx_error_msg(
        in->dtype() != dtype_T,
        "[DEBUG][internal error] in.dtype_str is [%s] but call cuMovemem_cutt with type %s",
        in->dtype_str().c_str(), Type.getname(dtype_T));
      cytnx_error_msg(in->device() == Device.cpu, "%s",
                      "[DEBUG][internal error] in.device is on cpu but all cuda function.");
    #endif

      CudaType *dtmp =
        reinterpret_cast<CudaType *>(cuMalloc_gpu(sizeof(CudaType) * in->capacity()));
      cytnx_uint64 Nelem = in->size();

      std::vector<int> perm(mapper.begin(), mapper.end());
      std::vector<int64_t> size(old_shape.begin(), old_shape.end());
      std::vector<int> ori(perm.size());
      for (int i = 0; i < ori.size(); i++) ori[i] = i;

      std::vector<int64_t> new_size(perm.size());
      for (int i = 0; i < new_size.size(); i++) {
        new_size[i] = size[perm[i]];
      }
      std::reverse(size.begin(), size.end());  // matching API
      std::reverse(perm.begin(), perm.end());  // matching API
      std::reverse(new_size.begin(), new_size.end());  // matching API
      std::reverse(ori.begin(), ori.end());  // matching API

      hiptensorDataType_t cutensor_data_type;
      hiptensorComputeDescriptor_t compute_descriptor;
      CudaType one;
      if constexpr (std::is_same_v<cytnx_complex128, DType>) {
        cutensor_data_type = HIPTENSOR_C_64F;
        compute_descriptor = HIPTENSOR_COMPUTE_DESC_64F;
        one = make_hipDoubleComplex(1, 0);
      } else if constexpr (std::is_same_v<cytnx_complex64, DType>) {
        cutensor_data_type = HIPTENSOR_C_32F;
        compute_descriptor = HIPTENSOR_COMPUTE_DESC_32F;
        one = make_hipFloatComplex(1, 0);
      } else if constexpr (std::is_same_v<cytnx_double, DType>) {
        cutensor_data_type = HIPTENSOR_R_64F;
        compute_descriptor = HIPTENSOR_COMPUTE_DESC_64F;
        one = 1;
      } else if constexpr (std::is_same_v<cytnx_float, DType>) {
        cutensor_data_type = HIPTENSOR_R_32F;
        compute_descriptor = HIPTENSOR_COMPUTE_DESC_32F;
        one = 1;
      }

      hiptensorHandle_t handle;
      checkCudaErrors(hiptensorCreate(&handle));

      // This is the default alignment of hipMalloc() and may also be the default alignment of
      // hipMallocManaged()
      cytnx_uint64 defaultAlignment = 256;
      hiptensorTensorDescriptor_t descA;
      checkCudaErrors(hiptensorCreateTensorDescriptor(handle, &descA, size.size(), size.data(),
                                                     NULL /* stride */, cutensor_data_type,
                                                     defaultAlignment));

      hiptensorTensorDescriptor_t descC;
      checkCudaErrors(hiptensorCreateTensorDescriptor(handle, &descC, new_size.size(),
                                                     new_size.data(), NULL /* stride */,
                                                     cutensor_data_type, defaultAlignment));
      hiptensorOperationDescriptor_t desc;
      checkCudaErrors(hiptensorCreatePermutation(handle, &desc, descA, ori.data(),
                                                HIPTENSOR_OP_IDENTITY, descC, perm.data(),
                                                compute_descriptor));

      const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

      hiptensorPlanPreference_t planPref;
      checkCudaErrors(
        hiptensorCreatePlanPreference(handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE));

      hiptensorPlan_t plan;
      checkCudaErrors(
        hiptensorCreatePlan(handle, &plan, desc, planPref, 0 /* workspaceSizeLimit */));

      checkCudaErrors(hiptensorPermute(handle, plan, &one, reinterpret_cast<CudaType *>(in->data()),
                                      dtmp, 0 /* stream */));

      checkCudaErrors(hiptensorDestroyTensorDescriptor(descA));
      checkCudaErrors(hiptensorDestroyTensorDescriptor(descC));
      checkCudaErrors(hiptensorDestroyPlanPreference(planPref));
      checkCudaErrors(hiptensorDestroyPlan(plan));
      checkCudaErrors(hiptensorDestroy(handle));

      boost::intrusive_ptr<Storage_base> out = __SII.USIInit[dtype_T]();
      if (is_inplace) {
        /// cpy back:
        checkCudaErrors(
          hipMemcpy(in->data(), dtmp, sizeof(DType) * Nelem, hipMemcpyDeviceToDevice));
        hipFree(dtmp);
        return out;

      } else {
        out->_Init_byptr(dtmp, Nelem, in->device(), true, in->capacity());
        return out;
      }
    }
  #endif

    template <typename DType>
    boost::intrusive_ptr<Storage_base> MoveMemoryGpu(boost::intrusive_ptr<Storage_base> &in,
                                                     const std::vector<cytnx_uint64> &old_shape,
                                                     const std::vector<cytnx_uint64> &mapper,
                                                     const std::vector<cytnx_uint64> &invmapper,
                                                     bool is_inplace) {
      if constexpr (is_complex_v<DType> || std::is_floating_point_v<DType>) {
  #if defined(UNI_CUTENSOR)
        return cuMovemem_cutensor_gpu<DType>(in, old_shape, mapper, invmapper, is_inplace);
  #elif defined(UNI_CUTT)
        return cuMovemem_cutt_gpu<DType>(in, old_shape, mapper, invmapper, is_inplace);
  #else
        return cuMovemem_gpu_general<DType>(in, old_shape, mapper, invmapper, is_inplace);
  #endif
      } else {
        return cuMovemem_gpu_general<DType>(in, old_shape, mapper, invmapper, is_inplace);
      }
    }

    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_complex128>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_complex64>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_double>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_float>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_uint64>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_int64>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_uint32>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_int32>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_uint16>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_int16>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
    template boost::intrusive_ptr<Storage_base> MoveMemoryGpu<cytnx_bool>(
      boost::intrusive_ptr<Storage_base> &, const std::vector<cytnx_uint64> &,
      const std::vector<cytnx_uint64> &, const std::vector<cytnx_uint64> &, bool);
#endif  // UNI_GPU
  }  // namespace utils_internal
}  // namespace cytnx
