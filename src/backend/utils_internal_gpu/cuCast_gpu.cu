#include "hip/hip_runtime.h"
#include "cuCast_gpu.hpp"
#include "backend/Storage.hpp"
#ifdef UNI_OMP
  #include <omp.h>
#endif

using namespace std;
namespace cytnx {
  namespace utils_internal {

    __global__ void cuCastElem_kernel_cd2cf(const hipDoubleComplex* src, hipFloatComplex* des,
                                            cytnx_uint64 Nelem) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        des[blockIdx.x * blockDim.x + threadIdx.x] =
          hipComplexDoubleToFloat(src[blockIdx.x * blockDim.x + threadIdx.x]);
      }
    }

    __global__ void cuCastElem_kernel_cf2cd(const hipFloatComplex* src, hipDoubleComplex* des,
                                            cytnx_uint64 Nelem) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        des[blockIdx.x * blockDim.x + threadIdx.x] =
          hipComplexFloatToDouble(src[blockIdx.x * blockDim.x + threadIdx.x]);
      }
    }

    template <class T>
    __global__ void cuCastElem_kernel_r2cf(const T* src, hipFloatComplex* des, cytnx_uint64 Nelem) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        des[blockIdx.x * blockDim.x + threadIdx.x].x = src[blockIdx.x * blockDim.x + threadIdx.x];
      }
    }
    template <class T2>
    __global__ void cuCastElem_kernel_r2cd(const T2* src, hipDoubleComplex* des,
                                           cytnx_uint64 Nelem) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        des[blockIdx.x * blockDim.x + threadIdx.x].x = src[blockIdx.x * blockDim.x + threadIdx.x];
      }
    }

    template <class T3, class T4>
    __global__ void cuCastElem_kernel_r2r(const T3* src, T4* des, cytnx_uint64 Nelem) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        des[blockIdx.x * blockDim.x + threadIdx.x] = src[blockIdx.x * blockDim.x + threadIdx.x];
      }
    }

    //========================================================================
    void cuCast_gpu_cdtcd(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_complex128) * len_in, hipMemcpyDeviceToDevice));
    }

    void cuCast_gpu_cdtcf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }

      hipDoubleComplex* _in = static_cast<hipDoubleComplex*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_cd2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    void cuCast_gpu_cftcd(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      hipFloatComplex* _in = static_cast<hipFloatComplex*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_cf2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    void cuCast_gpu_cftcf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_complex64) * len_in, hipMemcpyDeviceToDevice));
    }

    //-----------------------------
    void cuCast_gpu_dtcd(const boost::intrusive_ptr<Storage_base>& in,
                         boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                         const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    void cuCast_gpu_dtcf(const boost::intrusive_ptr<Storage_base>& in,
                         boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                         const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    void cuCast_gpu_dtd(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_double) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_dtf(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dti64(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dtu64(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dti32(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dtu32(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dti16(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dtu16(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_dtb(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_double* _in = static_cast<cytnx_double*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    //-----------------------------
    void cuCast_gpu_ftcd(const boost::intrusive_ptr<Storage_base>& in,
                         boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                         const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    void cuCast_gpu_ftcf(const boost::intrusive_ptr<Storage_base>& in,
                         boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                         const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_ftd(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_ftf(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_float) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_fti64(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_ftu64(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_fti32(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_ftu32(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_fti16(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_ftu16(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_ftb(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_float* _in = static_cast<cytnx_float*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    //------------------------
    void cuCast_gpu_i64tcd(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64tcf(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    void cuCast_gpu_i64td(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64tf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64ti64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_int64) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_i64tu64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64ti32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64tu32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64ti16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64tu16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i64tb(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int64* _in = static_cast<cytnx_int64*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    //-------------------------------
    void cuCast_gpu_u64tcd(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64tcf(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64td(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64tf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64ti64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64tu64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_uint64) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_u64ti32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64tu32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64ti16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64tu16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u64tb(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint64* _in = static_cast<cytnx_uint64*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }

    //--------------------------------
    void cuCast_gpu_i32tcd(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32tcf(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32td(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32tf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32ti64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32tu64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32ti32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_int32) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_i32tu32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32tu16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32ti16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i32tb(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int32* _in = static_cast<cytnx_int32*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    //----------------------------
    void cuCast_gpu_u32tcd(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32tcf(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32td(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32tf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32ti64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32tu64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32ti32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32tu32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_uint32) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_u32tu16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32ti16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u32tb(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint32* _in = static_cast<cytnx_uint32*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    //------------------------------
    void cuCast_gpu_u16tcd(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16tcf(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16td(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16tf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16ti64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16tu64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16ti32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16tu32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16tu16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_uint16) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_u16ti16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_u16tb(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_uint16* _in = static_cast<cytnx_uint16*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    //------------------------------
    void cuCast_gpu_i16tcd(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16tcf(const boost::intrusive_ptr<Storage_base>& in,
                           boost::intrusive_ptr<Storage_base>& out,
                           const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16td(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16tf(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16ti64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16tu64(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16ti32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16tu32(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16tu16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_i16ti16(const boost::intrusive_ptr<Storage_base>& in,
                            boost::intrusive_ptr<Storage_base>& out,
                            const unsigned long long& len_in, const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_int16) * len_in, hipMemcpyDeviceToDevice));
    }
    void cuCast_gpu_i16tb(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_int16* _in = static_cast<cytnx_int16*>(in->Mem);
      cytnx_bool* _out = static_cast<cytnx_bool*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    //------------------------------
    void cuCast_gpu_btcd(const boost::intrusive_ptr<Storage_base>& in,
                         boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                         const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexDoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      hipDoubleComplex* _out = static_cast<hipDoubleComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipDoubleComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cd<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btcf(const boost::intrusive_ptr<Storage_base>& in,
                         boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                         const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new ComplexFloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      hipFloatComplex* _out = static_cast<hipFloatComplex*>(out->Mem);
      hipMemset(out->Mem, 0, sizeof(hipFloatComplex) * len_in);
      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2cf<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btd(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new DoubleStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_double* _out = static_cast<cytnx_double*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btf(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new FloatStorage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_float* _out = static_cast<cytnx_float*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_bti64(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_int64* _out = static_cast<cytnx_int64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btu64(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint64Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_uint64* _out = static_cast<cytnx_uint64*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_bti32(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_int32* _out = static_cast<cytnx_int32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btu32(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint32Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_uint32* _out = static_cast<cytnx_uint32*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btu16(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Uint16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_uint16* _out = static_cast<cytnx_uint16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_bti16(const boost::intrusive_ptr<Storage_base>& in,
                          boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                          const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new Int16Storage());
        out->Init(len_in, alloc_device);
      }
      cytnx_bool* _in = static_cast<cytnx_bool*>(in->Mem);
      cytnx_int16* _out = static_cast<cytnx_int16*>(out->Mem);

      cytnx_uint64 NBlocks = len_in / 512;
      if (len_in % 512) NBlocks += 1;
      cuCastElem_kernel_r2r<<<NBlocks, 512>>>(_in, _out, len_in);
    }
    void cuCast_gpu_btb(const boost::intrusive_ptr<Storage_base>& in,
                        boost::intrusive_ptr<Storage_base>& out, const unsigned long long& len_in,
                        const int& alloc_device) {
      if (alloc_device >= 0) {
        out = boost::intrusive_ptr<Storage_base>(new BoolStorage());
        out->Init(len_in, alloc_device);
      }
      checkCudaErrors(
        hipMemcpy(out->Mem, in->Mem, sizeof(cytnx_bool) * len_in, hipMemcpyDeviceToDevice));
    }

  }  // namespace utils_internal
}  // namespace cytnx
