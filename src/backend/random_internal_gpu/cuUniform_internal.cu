#include "cuUniform_internal.hpp"

namespace cytnx {
  namespace random_internal {

    void cuRng_uniform_cd(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                          const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->data());

      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniformDouble(gen, rptr, in->size() * 2);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_uniform_cf(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                          const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->data());
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniform(gen, rptr, in->size() * 2);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_uniform_d(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->data());
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniformDouble(gen, rptr, in->size());

      hiprandDestroyGenerator(gen);
    }
    void cuRng_uniform_f(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->data());
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateUniform(gen, rptr, in->size());

      hiprandDestroyGenerator(gen);
    }

  }  // namespace random_internal
}  // namespace cytnx
