#include "cuNormal_internal.hpp"

namespace cytnx {
  namespace random_internal {

    void cuRng_normal_cd(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->data());

      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormalDouble(gen, rptr, in->size() * 2, a, b);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_normal_cf(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                         const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->data());
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormal(gen, rptr, in->size() * 2, a, b);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_normal_d(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                        const unsigned int &seed) {
      double *rptr = static_cast<double *>(in->data());
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormalDouble(gen, rptr, in->size(), a, b);

      hiprandDestroyGenerator(gen);
    }
    void cuRng_normal_f(boost::intrusive_ptr<Storage_base> &in, const double &a, const double &b,
                        const unsigned int &seed) {
      float *rptr = static_cast<float *>(in->data());
      hiprandGenerator_t gen;
      hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);

      // seed:
      hiprandSetPseudoRandomGeneratorSeed(gen, seed);

      // generate:
      hiprandGenerateNormal(gen, rptr, in->size(), a, b);

      hiprandDestroyGenerator(gen);
    }

  }  // namespace random_internal
}  // namespace cytnx
