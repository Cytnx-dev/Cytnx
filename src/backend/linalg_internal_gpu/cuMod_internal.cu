#include "hip/hip_runtime.h"
#include "cuMod_internal.hpp"
#include "../utils_internal_interface.hpp"
#include <hip/hip_runtime.h>
#ifdef UNI_OMP
  #include <omp.h>
#endif

namespace cytnx {

  namespace linalg_internal {

    //====================================================================
    // generic R+R kernel
    template <class T2, class T3>
    __global__ void cufMod_rconst_kernel(double *out, const T2 *ptr, const cytnx_uint64 Nelem,
                                         const T3 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          fmod(double(ptr[blockIdx.x * blockDim.x + threadIdx.x]), double(val));
      }
      __syncthreads();
    }

    template <class T2, class T3>
    __global__ void cufMod_lconst_kernel(double *out, const T2 val, const cytnx_uint64 Nelem,
                                         const T3 *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          fmod(double(val), double(ptr[blockIdx.x * blockDim.x + threadIdx.x]));
      }
      __syncthreads();
    }

    template <class T2, class T3>
    __global__ void cufMod_tn_kernel(double *out, const T2 *val, const cytnx_uint64 Nelem,
                                     const T3 *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          fmod(double(val[blockIdx.x * blockDim.x + threadIdx.x]),
               double(ptr[blockIdx.x * blockDim.x + threadIdx.x]));
      }
      __syncthreads();
    }

    //====================================================================
    // generic R+R kernel
    template <class T2, class T3>
    __global__ void cufModf_rconst_kernel(float *out, const T2 *ptr, const cytnx_uint64 Nelem,
                                          const T3 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          fmodf(ptr[blockIdx.x * blockDim.x + threadIdx.x], val);
      }
      __syncthreads();
    }

    template <class T2, class T3>
    __global__ void cufModf_lconst_kernel(float *out, const T2 val, const cytnx_uint64 Nelem,
                                          const T3 *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          fmodf(val, ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    template <class T2, class T3>
    __global__ void cufModf_tn_kernel(float *out, const T2 *val, const cytnx_uint64 Nelem,
                                      const T3 *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] = fmodf(
          val[blockIdx.x * blockDim.x + threadIdx.x], ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    //====================================================================
    // generic R+R kernel
    template <class T1, class T2, class T3>
    __global__ void cuMod_rconst_kernel(T1 *out, const T2 *ptr, const cytnx_uint64 Nelem,
                                        const T3 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          ptr[blockIdx.x * blockDim.x + threadIdx.x] % val;
      }
      __syncthreads();
    }

    template <class T1, class T2, class T3>
    __global__ void cuMod_lconst_kernel(T1 *out, const T2 val, const cytnx_uint64 Nelem,
                                        const T3 *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % ptr[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }

    template <class T1, class T2, class T3>
    __global__ void cuMod_tn_kernel(T1 *out, const T2 *val, const cytnx_uint64 Nelem,
                                    const T3 *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] % ptr[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }

    //=====================================================================
    void cuMod_internal_cdtcd(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdtcf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_cdtd(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_cdtf(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdtu64(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdtu32(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_cdti64(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdti32(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdti16(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdtu16(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cdtb(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    //------------------------------------

    void cuMod_internal_cftcd(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftcf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftd(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftf(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftu64(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftu32(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cfti64(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cfti32(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cfti16(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftu16(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_cftb(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    //------------------------------

    void cuMod_internal_dtcd(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_dtcf(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_dtd(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dtf(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dtu64(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dtu32(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dti64(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dti32(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dti16(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_dtu16(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    void cuMod_internal_dtb(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_Lin = (cytnx_double *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_double(_Rin[0]));
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //-----------------------------------

    void cuMod_internal_ftcd(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_ftcf(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_ftd(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_ftf(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_ftu64(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_ftu32(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_fti64(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_fti32(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_fti16(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_ftu16(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    void cuMod_internal_ftb(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_Lin = (cytnx_float *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_float(_Rin[0]));
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    // taag
    //----------------------------

    void cuMod_internal_i64tcd(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_i64tcf(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_i64td(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64tf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64ti64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64tu64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64ti32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64tu32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64ti16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i64tu16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    __global__ void cuMod_lconst_kernel(cytnx_int64 *out, const cytnx_int64 val,
                                        const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % cytnx_int64(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_int64 *out, const cytnx_int64 *val,
                                    const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] %
          cytnx_int64(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }
    void cuMod_internal_i64tb(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int64 *_Lin = (cytnx_int64 *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_int64(_Rin[0]));
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //-----------------------------------------

    void cuMod_internal_u64tcd(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_u64tcf(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_u64td(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64tf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64ti64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64tu64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64ti32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64tu32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64ti16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u64tu16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    __global__ void cuMod_lconst_kernel(cytnx_uint64 *out, const cytnx_uint64 val,
                                        const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % cytnx_uint64(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_uint64 *out, const cytnx_uint64 *val,
                                    const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] %
          cytnx_uint64(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }
    void cuMod_internal_u64tb(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint64 *_Lin = (cytnx_uint64 *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_uint64(_Rin[0]));
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //-----------------------------------------

    void cuMod_internal_i32tcd(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_i32tcf(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_i32td(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32tf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32ti64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32tu64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32ti32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32tu32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32ti16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i32tu16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    __global__ void cuMod_lconst_kernel(cytnx_int32 *out, const cytnx_int32 val,
                                        const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % cytnx_int32(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_int32 *out, const cytnx_int32 *val,
                                    const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] %
          cytnx_int32(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }
    void cuMod_internal_i32tb(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_int32 *_Lin = (cytnx_int32 *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_int32(_Rin[0]));
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //---------------------------------------

    void cuMod_internal_u32tcd(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_u32tcf(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_u32td(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32tf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32ti64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32tu64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32ti32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32tu32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32ti16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u32tu16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    __global__ void cuMod_lconst_kernel(cytnx_uint32 *out, const cytnx_uint32 val,
                                        const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % cytnx_uint32(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_uint32 *out, const cytnx_uint32 *val,
                                    const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] %
          cytnx_uint32(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }
    void cuMod_internal_u32tb(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_uint32 *_Lin = (cytnx_uint32 *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_uint32(_Rin[0]));
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //---------------------------------------

    void cuMod_internal_i16tcd(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_i16tcf(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_i16td(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16tf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16ti64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16tu64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16ti32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16tu32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16ti16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_i16tu16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    __global__ void cuMod_lconst_kernel(cytnx_int16 *out, const cytnx_int16 val,
                                        const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % cytnx_int16(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_int16 *out, const cytnx_int16 *val,
                                    const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] %
          cytnx_int16(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }
    void cuMod_internal_i16tb(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_int16 *_Lin = (cytnx_int16 *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_int16(_Rin[0]));
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //---------------------------------------

    void cuMod_internal_u16tcd(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_u16tcf(boost::intrusive_ptr<Storage_base> &out,
                               boost::intrusive_ptr<Storage_base> &Lin,
                               boost::intrusive_ptr<Storage_base> &Rin,
                               const unsigned long long &len,
                               const std::vector<cytnx_uint64> &shape,
                               const std::vector<cytnx_uint64> &invmapper_L,
                               const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }
    void cuMod_internal_u16td(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16tf(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16ti64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16tu64(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16ti32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16tu32(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16ti16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    void cuMod_internal_u16tu16(boost::intrusive_ptr<Storage_base> &out,
                                boost::intrusive_ptr<Storage_base> &Lin,
                                boost::intrusive_ptr<Storage_base> &Rin,
                                const unsigned long long &len,
                                const std::vector<cytnx_uint64> &shape,
                                const std::vector<cytnx_uint64> &invmapper_L,
                                const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint16 *_out = (cytnx_uint16 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    __global__ void cuMod_lconst_kernel(cytnx_uint16 *out, const cytnx_uint16 val,
                                        const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val % cytnx_uint16(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_uint16 *out, const cytnx_uint16 *val,
                                    const cytnx_uint64 Nelem, const cytnx_bool *ptr) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          val[blockIdx.x * blockDim.x + threadIdx.x] %
          cytnx_uint16(ptr[blockIdx.x * blockDim.x + threadIdx.x]);
      }
      __syncthreads();
    }
    void cuMod_internal_u16tb(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint16 *_out = (cytnx_uint16 *)out->Mem;
      cytnx_uint16 *_Lin = (cytnx_uint16 *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, cytnx_uint16(_Rin[0]));
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    //---------------------------------------

    void cuMod_internal_btcd(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_btcf(boost::intrusive_ptr<Storage_base> &out,
                             boost::intrusive_ptr<Storage_base> &Lin,
                             boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                             const std::vector<cytnx_uint64> &shape,
                             const std::vector<cytnx_uint64> &invmapper_L,
                             const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_error_msg(true, "[cuMod] Cannot mod complex numbers%s", "\n");
    }

    void cuMod_internal_btd(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_double *_Rin = (cytnx_double *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_double(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cufMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    void cuMod_internal_btf(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_float *_Rin = (cytnx_float *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cufModf_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_float(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cufModf_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cufModf_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    __global__ void cuMod_rconst_kernel(cytnx_int64 *out, const cytnx_bool *ptr,
                                        const cytnx_uint64 Nelem, const cytnx_int64 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_int64(ptr[blockIdx.x * blockDim.x + threadIdx.x]) % val;
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_int64 *out, const cytnx_bool *ptr,
                                    const cytnx_uint64 Nelem, const cytnx_int64 *val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_int64(ptr[blockIdx.x * blockDim.x + threadIdx.x]) %
          val[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    void cuMod_internal_bti64(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int64 *_out = (cytnx_int64 *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_int64 *_Rin = (cytnx_int64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_int64(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    __global__ void cuMod_rconst_kernel(cytnx_uint64 *out, const cytnx_bool *ptr,
                                        const cytnx_uint64 Nelem, const cytnx_uint64 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_uint64(ptr[blockIdx.x * blockDim.x + threadIdx.x]) % val;
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_uint64 *out, const cytnx_bool *ptr,
                                    const cytnx_uint64 Nelem, const cytnx_uint64 *val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_uint64(ptr[blockIdx.x * blockDim.x + threadIdx.x]) %
          val[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    void cuMod_internal_btu64(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_uint64 *_Rin = (cytnx_uint64 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_uint64(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    __global__ void cuMod_rconst_kernel(cytnx_int32 *out, const cytnx_bool *ptr,
                                        const cytnx_uint64 Nelem, const cytnx_int32 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_int32(ptr[blockIdx.x * blockDim.x + threadIdx.x]) % val;
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_int32 *out, const cytnx_bool *ptr,
                                    const cytnx_uint64 Nelem, const cytnx_int32 *val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_int32(ptr[blockIdx.x * blockDim.x + threadIdx.x]) %
          val[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    void cuMod_internal_bti32(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int32 *_out = (cytnx_int32 *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_int32 *_Rin = (cytnx_int32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_int32(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    __global__ void cuMod_rconst_kernel(cytnx_uint32 *out, const cytnx_bool *ptr,
                                        const cytnx_uint64 Nelem, const cytnx_uint32 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_uint32(ptr[blockIdx.x * blockDim.x + threadIdx.x]) % val;
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_uint32 *out, const cytnx_bool *ptr,
                                    const cytnx_uint64 Nelem, const cytnx_uint32 *val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_uint32(ptr[blockIdx.x * blockDim.x + threadIdx.x]) %
          val[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    void cuMod_internal_btu32(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint32 *_out = (cytnx_uint32 *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_uint32 *_Rin = (cytnx_uint32 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_uint32(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    __global__ void cuMod_rconst_kernel(cytnx_int16 *out, const cytnx_bool *ptr,
                                        const cytnx_uint64 Nelem, const cytnx_int16 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_int16(ptr[blockIdx.x * blockDim.x + threadIdx.x]) % val;
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_int16 *out, const cytnx_bool *ptr,
                                    const cytnx_uint64 Nelem, const cytnx_int16 *val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_int16(ptr[blockIdx.x * blockDim.x + threadIdx.x]) %
          val[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    void cuMod_internal_bti16(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_int16 *_out = (cytnx_int16 *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_int16 *_Rin = (cytnx_int16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_int16(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }
    __global__ void cuMod_rconst_kernel(cytnx_uint16 *out, const cytnx_bool *ptr,
                                        const cytnx_uint64 Nelem, const cytnx_uint16 val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_uint16(ptr[blockIdx.x * blockDim.x + threadIdx.x]) % val;
      }
      __syncthreads();
    }

    __global__ void cuMod_tn_kernel(cytnx_uint16 *out, const cytnx_bool *ptr,
                                    const cytnx_uint64 Nelem, const cytnx_uint16 *val) {
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        out[blockIdx.x * blockDim.x + threadIdx.x] =
          cytnx_uint16(ptr[blockIdx.x * blockDim.x + threadIdx.x]) %
          val[blockIdx.x * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    void cuMod_internal_btu16(boost::intrusive_ptr<Storage_base> &out,
                              boost::intrusive_ptr<Storage_base> &Lin,
                              boost::intrusive_ptr<Storage_base> &Rin,
                              const unsigned long long &len, const std::vector<cytnx_uint64> &shape,
                              const std::vector<cytnx_uint64> &invmapper_L,
                              const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_uint16 *_out = (cytnx_uint16 *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_uint16 *_Rin = (cytnx_uint16 *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, cytnx_uint16(_Lin[0]), len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

    void cuMod_internal_btb(boost::intrusive_ptr<Storage_base> &out,
                            boost::intrusive_ptr<Storage_base> &Lin,
                            boost::intrusive_ptr<Storage_base> &Rin, const unsigned long long &len,
                            const std::vector<cytnx_uint64> &shape,
                            const std::vector<cytnx_uint64> &invmapper_L,
                            const std::vector<cytnx_uint64> &invmapper_R) {
      cytnx_bool *_out = (cytnx_bool *)out->Mem;
      cytnx_bool *_Lin = (cytnx_bool *)Lin->Mem;
      cytnx_bool *_Rin = (cytnx_bool *)Rin->Mem;

      cytnx_uint32 NBlocks = len / 512;
      if (len % 512) NBlocks += 1;

      if (Lin->size() == 1) {
        cuMod_lconst_kernel<<<NBlocks, 512>>>(_out, _Lin[0], len, _Rin);
      } else if (Rin->size() == 1) {
        cuMod_rconst_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin[0]);
      } else {
        cuMod_tn_kernel<<<NBlocks, 512>>>(_out, _Lin, len, _Rin);
      }
    }

  }  // namespace linalg_internal
}  // namespace cytnx
