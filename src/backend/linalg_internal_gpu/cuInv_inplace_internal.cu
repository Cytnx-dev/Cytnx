#include "hip/hip_runtime.h"
#include "cuInv_inplace_internal.hpp"
#include "../utils_internal_interface.hpp"

// #ifdef UNI_OMP
//     #include <omp.h>
// #endif

namespace cytnx {

  namespace linalg_internal {
    __global__ void cuInv_internal_kernel_d(cytnx_double *ten, const cytnx_uint64 Nelem,
                                            const double clip) {
      cytnx_double tmp;
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        tmp = ten[blockIdx.x * blockDim.x + threadIdx.x];
        ten[blockIdx.x * blockDim.x + threadIdx.x] = tmp < clip ? 0 : double(1) / tmp;
      }
      __syncthreads();
    }
    __global__ void cuInv_internal_kernel_f(cytnx_float *ten, const cytnx_uint64 Nelem,
                                            const float clip) {
      cytnx_float tmp;
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        tmp = ten[blockIdx.x * blockDim.x + threadIdx.x];
        ten[blockIdx.x * blockDim.x + threadIdx.x] = tmp < clip ? 0 : float(1) / tmp;
      }
      __syncthreads();
    }

    __global__ void cuInv_internal_kernel_cd(hipDoubleComplex *ten, const cytnx_uint64 Nelem,
                                             const double clip) {
      hipDoubleComplex tmp;
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        tmp = ten[blockIdx.x * blockDim.x + threadIdx.x];
        ten[blockIdx.x * blockDim.x + threadIdx.x] = (tmp.x * tmp.x + tmp.y * tmp.y) < clip
                                                       ? make_hipDoubleComplex(0., 0)
                                                       : hipCdiv(make_hipDoubleComplex(1., 0), tmp);
      }
      __syncthreads();
    }

    __global__ void cuInv_internal_kernel_cf(hipFloatComplex *ten, const cytnx_uint64 Nelem,
                                             const float clip) {
      hipFloatComplex tmp;
      if (blockIdx.x * blockDim.x + threadIdx.x < Nelem) {
        tmp = ten[blockIdx.x * blockDim.x + threadIdx.x];
        ten[blockIdx.x * blockDim.x + threadIdx.x] = (tmp.x * tmp.x + tmp.y * tmp.y) < clip
                                                       ? make_hipFloatComplex(0., 0)
                                                       : hipCdivf(make_hipFloatComplex(1., 0), tmp);
      }
      __syncthreads();
    }

  }  // namespace linalg_internal

}  // namespace cytnx

namespace cytnx {
  namespace linalg_internal {

    void cuInv_inplace_internal_d(boost::intrusive_ptr<Storage_base> &ten,
                                  const cytnx_uint64 &Nelem, const double &clip) {
      cytnx_uint32 NBlocks = Nelem / 512;
      if (Nelem % 512) NBlocks += 1;
      cuInv_internal_kernel_d<<<NBlocks, 512>>>((cytnx_double *)ten->Mem, Nelem, clip);
    }

    void cuInv_inplace_internal_f(boost::intrusive_ptr<Storage_base> &ten,
                                  const cytnx_uint64 &Nelem, const double &clip) {
      cytnx_uint32 NBlocks = Nelem / 512;
      if (Nelem % 512) NBlocks += 1;
      cuInv_internal_kernel_f<<<NBlocks, 512>>>((cytnx_float *)ten->Mem, Nelem, clip);
    }

    void cuInv_inplace_internal_cd(boost::intrusive_ptr<Storage_base> &ten,
                                   const cytnx_uint64 &Nelem, const double &clip) {
      cytnx_uint32 NBlocks = Nelem / 256;
      if (Nelem % 256) NBlocks += 1;
      cuInv_internal_kernel_cd<<<NBlocks, 256>>>((hipDoubleComplex *)ten->Mem, Nelem, clip);
    }

    void cuInv_inplace_internal_cf(boost::intrusive_ptr<Storage_base> &ten,
                                   const cytnx_uint64 &Nelem, const double &clip) {
      cytnx_uint32 NBlocks = Nelem / 256;
      if (Nelem % 256) NBlocks += 1;
      cuInv_internal_kernel_cf<<<NBlocks, 256>>>((hipFloatComplex *)ten->Mem, Nelem, clip);
    }

  }  // namespace linalg_internal

}  // namespace cytnx
