#include "cuGemm_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

namespace cytnx {

  namespace linalg_internal {
    void cuGemm_internal_cd(boost::intrusive_ptr<Storage_base> &out,
                            const boost::intrusive_ptr<Storage_base> &inl,
                            const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                            const cytnx_int64 &Comm, const cytnx_int64 &Nr, const Scalar &a,
                            const Scalar &b) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex128 alpha = complex128(a), beta = complex128(b);

      hipDoubleComplex *_out = (hipDoubleComplex *)out->Mem;
      hipDoubleComplex *_inl = (hipDoubleComplex *)inl->Mem;
      hipDoubleComplex *_inr = (hipDoubleComplex *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr, blsComm = Comm;
      checkCudaErrors(hipblasZgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, blsNr, blsMl, blsComm,
                                  (hipDoubleComplex *)&alpha, _inr, blsNr, _inl, blsComm,
                                  (hipDoubleComplex *)&beta, _out, blsNr));

      hipblasDestroy(cublasH);
    }
    void cuGemm_internal_cf(boost::intrusive_ptr<Storage_base> &out,
                            const boost::intrusive_ptr<Storage_base> &inl,
                            const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                            const cytnx_int64 &Comm, const cytnx_int64 &Nr, const Scalar &a,
                            const Scalar &b) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex64 alpha = complex64(a), beta = complex64(b);

      hipFloatComplex *_out = (hipFloatComplex *)out->Mem;
      hipFloatComplex *_inl = (hipFloatComplex *)inl->Mem;
      hipFloatComplex *_inr = (hipFloatComplex *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr, blsComm = Comm;
      checkCudaErrors(hipblasCgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, blsNr, blsMl, blsComm,
                                  (hipFloatComplex *)&alpha, _inr, blsNr, _inl, blsComm,
                                  (hipFloatComplex *)&beta, _out, blsNr));

      hipblasDestroy(cublasH);
    }

    void cuGemm_internal_d(boost::intrusive_ptr<Storage_base> &out,
                           const boost::intrusive_ptr<Storage_base> &inl,
                           const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                           const cytnx_int64 &Comm, const cytnx_int64 &Nr, const Scalar &a,
                           const Scalar &b) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_double alpha = double(a), beta = double(b);

      cytnx_double *_out = (cytnx_double *)out->Mem;
      cytnx_double *_inl = (cytnx_double *)inl->Mem;
      cytnx_double *_inr = (cytnx_double *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr, blsComm = Comm;
      checkCudaErrors(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, blsNr, blsMl, blsComm, &alpha,
                                  _inr, blsNr, _inl, blsComm, &beta, _out, blsNr));

      hipblasDestroy(cublasH);
    }
    void cuGemm_internal_f(boost::intrusive_ptr<Storage_base> &out,
                           const boost::intrusive_ptr<Storage_base> &inl,
                           const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                           const cytnx_int64 &Comm, const cytnx_int64 &Nr, const Scalar &a,
                           const Scalar &b) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_float alpha = float(a), beta = float(b);

      cytnx_float *_out = (cytnx_float *)out->Mem;
      cytnx_float *_inl = (cytnx_float *)inl->Mem;
      cytnx_float *_inr = (cytnx_float *)inr->Mem;

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr, blsComm = Comm;
      checkCudaErrors(hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, blsNr, blsMl, blsComm, &alpha,
                                  _inr, blsNr, _inl, blsComm, &beta, _out, blsNr));

      hipblasDestroy(cublasH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
