#include "cuDet_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

#include "../utils_internal_gpu/cuAlloc_gpu.hpp"

namespace cytnx {

  namespace linalg_internal {

    void cuDet_internal_cd(void* out, const boost::intrusive_ptr<Storage_base>& in,
                           const cytnx_uint64& L) {
      cytnx_complex128* od = (cytnx_complex128*)out;  // result on cpu!
      hipDoubleComplex* _in = (hipDoubleComplex*)utils_internal::cuMalloc_gpu(
        in->len * sizeof(hipDoubleComplex));  // unify mem.
      checkCudaErrors(
        hipMemcpy(_in, in->Mem, sizeof(cytnx_complex128) * in->len, hipMemcpyDeviceToDevice));

      hipsolverHandle_t cusolverH;
      hipsolverDnCreate(&cusolverH);

      int* devIpiv;
      int* devInfo;
      checkCudaErrors(hipMalloc((void**)&devIpiv, L * sizeof(int)));
      checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

      int workspace_size = 0;
      hipDoubleComplex* workspace = NULL;
      hipsolverDnZgetrf_bufferSize(cusolverH, L, L, _in, L, &workspace_size);
      checkCudaErrors(hipMalloc((void**)&workspace, workspace_size * sizeof(hipDoubleComplex)));

      hipsolverDnZgetrf(cusolverH, L, L, _in, L, workspace, devIpiv, devInfo);

      int info;
      checkCudaErrors(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
      // If the info > 0, that means the U factor is exactly singular, and the det is 0.
      cytnx_error_msg(info < 0, "[ERROR] hipsolverDnZgetrf fail with info= %d\n", info);

      // since we do unify mem, direct access element is possible:
      od[0] = 1;
      bool neg = 0;
      int* ipiv = new int[L];
      checkCudaErrors(hipMemcpy(ipiv, devIpiv, L * sizeof(int), hipMemcpyDeviceToHost));
      for (int i = 0; i < L; i++) {
        od[0] *= ((cytnx_complex128*)_in)[i * L + i];
        if (ipiv[i] != (i + 1)) neg = !neg;
      }
      delete[] ipiv;
      hipFree(devIpiv);
      hipFree(devInfo);
      hipFree(workspace);
      hipFree(_in);
      hipsolverDnDestroy(cusolverH);
      if (neg) od[0] *= -1;

      if (info > 0) od[0] = 0;
    }

    void cuDet_internal_cf(void* out, const boost::intrusive_ptr<Storage_base>& in,
                           const cytnx_uint64& L) {
      cytnx_complex64* od = (cytnx_complex64*)out;  // result on cpu!
      hipFloatComplex* _in = (hipFloatComplex*)utils_internal::cuMalloc_gpu(
        in->len * sizeof(hipFloatComplex));  // unify mem.
      checkCudaErrors(
        hipMemcpy(_in, in->Mem, sizeof(cytnx_complex64) * in->len, hipMemcpyDeviceToDevice));

      hipsolverHandle_t cusolverH;
      hipsolverDnCreate(&cusolverH);

      int* devIpiv;
      int* devInfo;
      checkCudaErrors(hipMalloc((void**)&devIpiv, L * sizeof(int)));
      checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

      int workspace_size = 0;
      hipFloatComplex* workspace = NULL;
      hipsolverDnCgetrf_bufferSize(cusolverH, L, L, _in, L, &workspace_size);
      checkCudaErrors(hipMalloc((void**)&workspace, workspace_size * sizeof(hipFloatComplex)));

      hipsolverDnCgetrf(cusolverH, L, L, _in, L, workspace, devIpiv, devInfo);

      int info;
      checkCudaErrors(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
      // If the info > 0, that means the U factor is exactly singular, and the det is 0.
      cytnx_error_msg(info < 0, "[ERROR] hipsolverDnCgetrf fail with info= %d\n", info);

      // since we do unify mem, direct access element is possible:
      od[0] = 1;
      bool neg = 0;
      int* ipiv = new int[L];
      checkCudaErrors(hipMemcpy(ipiv, devIpiv, L * sizeof(int), hipMemcpyDeviceToHost));
      for (int i = 0; i < L; i++) {
        od[0] *= ((cytnx_complex64*)_in)[i * L + i];
        if (ipiv[i] != (i + 1)) neg = !neg;
      }
      delete[] ipiv;
      hipFree(devIpiv);
      hipFree(devInfo);
      hipFree(workspace);
      hipFree(_in);
      hipsolverDnDestroy(cusolverH);
      if (neg) od[0] *= -1;

      if (info > 0) od[0] = 0;
    }

    void cuDet_internal_d(void* out, const boost::intrusive_ptr<Storage_base>& in,
                          const cytnx_uint64& L) {
      cytnx_double* od = (cytnx_double*)out;  // result on cpu!
      cytnx_double* _in =
        (cytnx_double*)utils_internal::cuMalloc_gpu(in->len * sizeof(cytnx_double));  // unify mem.
      checkCudaErrors(
        hipMemcpy(_in, in->Mem, sizeof(cytnx_double) * in->len, hipMemcpyDeviceToDevice));

      hipsolverHandle_t cusolverH;
      hipsolverDnCreate(&cusolverH);

      int* devIpiv;
      int* devInfo;
      checkCudaErrors(hipMalloc((void**)&devIpiv, L * sizeof(int)));
      checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

      int workspace_size = 0;
      cytnx_double* workspace = NULL;
      hipsolverDnDgetrf_bufferSize(cusolverH, L, L, _in, L, &workspace_size);
      checkCudaErrors(hipMalloc((void**)&workspace, workspace_size * sizeof(cytnx_double)));

      hipsolverDnDgetrf(cusolverH, L, L, _in, L, workspace, devIpiv, devInfo);

      int info;
      checkCudaErrors(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
      // If the info > 0, that means the U factor is exactly singular, and the det is 0.
      cytnx_error_msg(info < 0, "[ERROR] hipsolverDnDgetrf fail with info= %d\n", info);

      // since we do unify mem, direct access element is possible:
      od[0] = 1;
      bool neg = 0;
      int* ipiv = new int[L];
      checkCudaErrors(hipMemcpy(ipiv, devIpiv, L * sizeof(int), hipMemcpyDeviceToHost));
      for (int i = 0; i < L; i++) {
        od[0] *= _in[i * L + i];
        if (ipiv[i] != (i + 1)) neg = !neg;
      }
      delete[] ipiv;
      hipFree(devIpiv);
      hipFree(devInfo);
      hipFree(workspace);
      hipFree(_in);
      hipsolverDnDestroy(cusolverH);
      if (neg) od[0] *= -1;

      if (info > 0) od[0] = 0;
    }

    void cuDet_internal_f(void* out, const boost::intrusive_ptr<Storage_base>& in,
                          const cytnx_uint64& L) {
      cytnx_float* od = (cytnx_float*)out;  // result on cpu!
      cytnx_float* _in =
        (cytnx_float*)utils_internal::cuMalloc_gpu(in->len * sizeof(cytnx_float));  // unify mem.
      checkCudaErrors(
        hipMemcpy(_in, in->Mem, sizeof(cytnx_float) * in->len, hipMemcpyDeviceToDevice));

      hipsolverHandle_t cusolverH;
      hipsolverDnCreate(&cusolverH);

      int* devIpiv;
      int* devInfo;
      checkCudaErrors(hipMalloc((void**)&devIpiv, L * sizeof(int)));
      checkCudaErrors(hipMalloc((void**)&devInfo, sizeof(int)));

      int workspace_size = 0;
      cytnx_float* workspace = NULL;
      hipsolverDnSgetrf_bufferSize(cusolverH, L, L, _in, L, &workspace_size);
      checkCudaErrors(hipMalloc((void**)&workspace, workspace_size * sizeof(cytnx_float)));

      hipsolverDnSgetrf(cusolverH, L, L, _in, L, workspace, devIpiv, devInfo);

      int info;
      checkCudaErrors(hipMemcpy(&info, devInfo, sizeof(int), hipMemcpyDeviceToHost));
      // If the info > 0, that means the U factor is exactly singular, and the det is 0.
      cytnx_error_msg(info < 0, "[ERROR] hipsolverDnSgetrf fail with info= %d\n", info);

      // since we do unify mem, direct access element is possible:
      od[0] = 1;
      bool neg = 0;
      int* ipiv = new int[L];
      checkCudaErrors(hipMemcpy(ipiv, devIpiv, L * sizeof(int), hipMemcpyDeviceToHost));
      for (int i = 0; i < L; i++) {
        od[0] *= _in[i * L + i];
        if (ipiv[i] != (i + 1)) neg = !neg;
      }
      delete[] ipiv;
      hipFree(devIpiv);
      hipFree(devInfo);
      hipFree(workspace);
      hipFree(_in);
      hipsolverDnDestroy(cusolverH);
      if (neg) od[0] *= -1;

      if (info > 0) od[0] = 0;
    }

  }  // namespace linalg_internal
}  // namespace cytnx
