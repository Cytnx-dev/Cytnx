#include "cuGer_internal.hpp"
#include "../utils_internal_interface.hpp"

#include "backend/lapack_wrapper.hpp"

#ifdef UNI_OMP
  #include <omp.h>
#endif

namespace cytnx {
  namespace linalg_internal {

    void cuGer_internal_cd(boost::intrusive_ptr<Storage_base> &A,
                           const boost::intrusive_ptr<Storage_base> &x,
                           const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex128 alpha = complex128(a);

      hipDoubleComplex *_A = (hipDoubleComplex *)A->Mem;
      hipDoubleComplex *_x = (hipDoubleComplex *)x->Mem;
      hipDoubleComplex *_y = (hipDoubleComplex *)y->Mem;

      checkCudaErrors(hipblasZgeru(cublasH, y->size(), x->size(), (hipDoubleComplex *)&alpha, _y, 1,
                                  _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

    void cuGer_internal_cf(boost::intrusive_ptr<Storage_base> &A,
                           const boost::intrusive_ptr<Storage_base> &x,
                           const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex64 alpha = complex64(a);

      hipFloatComplex *_A = (hipFloatComplex *)A->Mem;
      hipFloatComplex *_x = (hipFloatComplex *)x->Mem;
      hipFloatComplex *_y = (hipFloatComplex *)y->Mem;

      checkCudaErrors(hipblasCgeru(cublasH, y->size(), x->size(), (hipFloatComplex *)&alpha, _y, 1,
                                  _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

    void cuGer_internal_d(boost::intrusive_ptr<Storage_base> &A,
                          const boost::intrusive_ptr<Storage_base> &x,
                          const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_double alpha = cytnx_double(a);

      cytnx_double *_A = (cytnx_double *)A->Mem;
      cytnx_double *_x = (cytnx_double *)x->Mem;
      cytnx_double *_y = (cytnx_double *)y->Mem;

      checkCudaErrors(
        hipblasDger(cublasH, y->size(), x->size(), &alpha, _y, 1, _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

    void cuGer_internal_f(boost::intrusive_ptr<Storage_base> &A,
                          const boost::intrusive_ptr<Storage_base> &x,
                          const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_float alpha = cytnx_float(a);

      cytnx_float *_A = (cytnx_float *)A->Mem;
      cytnx_float *_x = (cytnx_float *)x->Mem;
      cytnx_float *_y = (cytnx_float *)y->Mem;

      checkCudaErrors(
        hipblasSger(cublasH, y->size(), x->size(), &alpha, _y, 1, _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
