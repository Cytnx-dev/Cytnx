#include "cuGer_internal.hpp"
#include "../utils_internal_interface.hpp"

#include "backend/lapack_wrapper.hpp"

#ifdef UNI_OMP
  #include <omp.h>
#endif

namespace cytnx {
  namespace linalg_internal {

    void cuGer_internal_cd(boost::intrusive_ptr<Storage_base> &A,
                           const boost::intrusive_ptr<Storage_base> &x,
                           const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex128 alpha = complex128(a);

      hipDoubleComplex *_A = (hipDoubleComplex *)A->data();
      hipDoubleComplex *_x = (hipDoubleComplex *)x->data();
      hipDoubleComplex *_y = (hipDoubleComplex *)y->data();

      checkCudaErrors(hipblasZgeru(cublasH, y->size(), x->size(), (hipDoubleComplex *)&alpha, _y, 1,
                                  _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

    void cuGer_internal_cf(boost::intrusive_ptr<Storage_base> &A,
                           const boost::intrusive_ptr<Storage_base> &x,
                           const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex64 alpha = complex64(a);

      hipFloatComplex *_A = (hipFloatComplex *)A->data();
      hipFloatComplex *_x = (hipFloatComplex *)x->data();
      hipFloatComplex *_y = (hipFloatComplex *)y->data();

      checkCudaErrors(hipblasCgeru(cublasH, y->size(), x->size(), (hipFloatComplex *)&alpha, _y, 1,
                                  _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

    void cuGer_internal_d(boost::intrusive_ptr<Storage_base> &A,
                          const boost::intrusive_ptr<Storage_base> &x,
                          const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_double alpha = cytnx_double(a);

      cytnx_double *_A = (cytnx_double *)A->data();
      cytnx_double *_x = (cytnx_double *)x->data();
      cytnx_double *_y = (cytnx_double *)y->data();

      checkCudaErrors(
        hipblasDger(cublasH, y->size(), x->size(), &alpha, _y, 1, _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

    void cuGer_internal_f(boost::intrusive_ptr<Storage_base> &A,
                          const boost::intrusive_ptr<Storage_base> &x,
                          const boost::intrusive_ptr<Storage_base> &y, const Scalar &a) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_float alpha = cytnx_float(a);

      cytnx_float *_A = (cytnx_float *)A->data();
      cytnx_float *_x = (cytnx_float *)x->data();
      cytnx_float *_y = (cytnx_float *)y->data();

      checkCudaErrors(
        hipblasSger(cublasH, y->size(), x->size(), &alpha, _y, 1, _x, 1, _A, y->size()));

      hipblasDestroy(cublasH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
