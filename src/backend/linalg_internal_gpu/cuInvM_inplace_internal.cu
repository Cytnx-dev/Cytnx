#include "cuInvM_inplace_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

namespace cytnx {
  namespace linalg_internal {

    void cuInvM_inplace_internal_d(boost::intrusive_ptr<Storage_base> &ten, const cytnx_int64 &L) {
      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_int32 *ipiv;
      cytnx_int32 info;
      cytnx_int32 lwork = 0;
      cytnx_double *d_work = NULL;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&ipiv, (L + 1) * sizeof(cytnx_int32)));
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      // trf:
      checkCudaErrors(
        hipsolverDnDgetrf_bufferSize(cusolverH, L, L, (cytnx_double *)ten->Mem, L, &lwork));
      checkCudaErrors(hipMalloc((void **)&d_work, sizeof(cytnx_double) * lwork));

      checkCudaErrors(
        hipsolverDnDgetrf(cusolverH, L, L, (cytnx_double *)ten->Mem, L, d_work, ipiv, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnDgetrf': cuBlas INFO = ", info);

      // trs AX = B with B = I
      lwork = 0;
      cytnx_double *d_I;
      checkCudaErrors(hipMalloc((void **)&d_I, sizeof(cytnx_double) * L * L));
      cytnx_double *h_I = (cytnx_double *)calloc(L * L, sizeof(cytnx_double));
      for (auto i = 0; i < L; i++) h_I[i * L + i] = 1;

      checkCudaErrors(hipMemcpy(d_I, h_I, sizeof(cytnx_double) * L * L, hipMemcpyHostToDevice));

      checkCudaErrors(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, L, L, (cytnx_double *)ten->Mem, L,
                                       ipiv, d_I, L, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnDgetrs': cuBlas INFO = ", info);

      checkCudaErrors(
        hipMemcpy(ten->Mem, d_I, sizeof(cytnx_double) * L * L, hipMemcpyDeviceToDevice));

      hipFree(d_I);
      hipFree(d_work);
      hipFree(devinfo);
      hipFree(ipiv);
      free(h_I);
      hipsolverDnDestroy(cusolverH);
    }
    void cuInvM_inplace_internal_f(boost::intrusive_ptr<Storage_base> &ten, const cytnx_int64 &L) {
      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_int32 *ipiv;
      cytnx_int32 info;
      cytnx_int32 lwork = 0;
      cytnx_float *d_work = NULL;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&ipiv, (L + 1) * sizeof(cytnx_int32)));
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      // trf:
      checkCudaErrors(
        hipsolverDnSgetrf_bufferSize(cusolverH, L, L, (cytnx_float *)ten->Mem, L, &lwork));
      checkCudaErrors(hipMalloc((void **)&d_work, sizeof(cytnx_float) * lwork));

      checkCudaErrors(
        hipsolverDnSgetrf(cusolverH, L, L, (cytnx_float *)ten->Mem, L, d_work, ipiv, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnSgetrf': cuBlas INFO = ", info);

      // trs AX = B with B = I
      lwork = 0;
      cytnx_float *d_I;
      checkCudaErrors(hipMalloc((void **)&d_I, sizeof(cytnx_float) * L * L));
      cytnx_float *h_I = (cytnx_float *)calloc(L * L, sizeof(cytnx_float));
      for (auto i = 0; i < L; i++) h_I[i * L + i] = 1;

      checkCudaErrors(hipMemcpy(d_I, h_I, sizeof(cytnx_float) * L * L, hipMemcpyHostToDevice));

      checkCudaErrors(hipsolverDnSgetrs(cusolverH, HIPBLAS_OP_N, L, L, (cytnx_float *)ten->Mem, L,
                                       ipiv, d_I, L, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnSgetrs': cuBlas INFO = ", info);

      checkCudaErrors(
        hipMemcpy(ten->Mem, d_I, sizeof(cytnx_float) * L * L, hipMemcpyDeviceToDevice));

      hipFree(d_I);
      hipFree(d_work);
      hipFree(devinfo);
      hipFree(ipiv);
      free(h_I);
      hipsolverDnDestroy(cusolverH);
    }
    void cuInvM_inplace_internal_cd(boost::intrusive_ptr<Storage_base> &ten, const cytnx_int64 &L) {
      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_int32 *ipiv;
      cytnx_int32 info;
      cytnx_int32 lwork = 0;
      cytnx_complex128 *d_work = NULL;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&ipiv, (L + 1) * sizeof(cytnx_int32)));
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      // trf:
      checkCudaErrors(
        hipsolverDnZgetrf_bufferSize(cusolverH, L, L, (hipDoubleComplex *)ten->Mem, L, &lwork));
      checkCudaErrors(hipMalloc((void **)&d_work, sizeof(cytnx_complex128) * lwork));

      checkCudaErrors(hipsolverDnZgetrf(cusolverH, L, L, (hipDoubleComplex *)ten->Mem, L,
                                       (hipDoubleComplex *)d_work, ipiv, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnZgetrf': cuBlas INFO = ", info);

      // trs AX = B with B = I
      lwork = 0;
      cytnx_complex128 *d_I;
      checkCudaErrors(hipMalloc((void **)&d_I, sizeof(cytnx_complex128) * L * L));
      cytnx_complex128 *h_I = (cytnx_complex128 *)calloc(L * L, sizeof(cytnx_complex128));
      for (auto i = 0; i < L; i++) h_I[i * L + i] = cytnx_complex128(1, 0);

      checkCudaErrors(
        hipMemcpy(d_I, h_I, sizeof(cytnx_complex128) * L * L, hipMemcpyHostToDevice));

      checkCudaErrors(hipsolverDnZgetrs(cusolverH, HIPBLAS_OP_N, L, L, (hipDoubleComplex *)ten->Mem, L,
                                       ipiv, (hipDoubleComplex *)d_I, L, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnZgetrs': cuBlas INFO = ", info);

      checkCudaErrors(
        hipMemcpy(ten->Mem, d_I, sizeof(cytnx_complex128) * L * L, hipMemcpyDeviceToDevice));

      hipFree(d_I);
      hipFree(d_work);
      hipFree(devinfo);
      hipFree(ipiv);
      free(h_I);
      hipsolverDnDestroy(cusolverH);
    }

    void cuInvM_inplace_internal_cf(boost::intrusive_ptr<Storage_base> &ten, const cytnx_int64 &L) {
      // create handles:
      hipsolverHandle_t cusolverH = NULL;
      checkCudaErrors(hipsolverDnCreate(&cusolverH));

      cytnx_int32 *ipiv;
      cytnx_int32 info;
      cytnx_int32 lwork = 0;
      cytnx_complex64 *d_work = NULL;
      cytnx_int32 *devinfo;
      checkCudaErrors(hipMalloc((void **)&ipiv, (L + 1) * sizeof(cytnx_int32)));
      checkCudaErrors(hipMalloc((void **)&devinfo, sizeof(cytnx_int32)));
      // trf:
      checkCudaErrors(
        hipsolverDnCgetrf_bufferSize(cusolverH, L, L, (hipFloatComplex *)ten->Mem, L, &lwork));
      checkCudaErrors(hipMalloc((void **)&d_work, sizeof(cytnx_complex64) * lwork));

      checkCudaErrors(hipsolverDnCgetrf(cusolverH, L, L, (hipFloatComplex *)ten->Mem, L,
                                       (hipFloatComplex *)d_work, ipiv, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnCgetrf': cuBlas INFO = ", info);

      // trs AX = B with B = I
      lwork = 0;
      cytnx_complex64 *d_I;
      checkCudaErrors(hipMalloc((void **)&d_I, sizeof(cytnx_complex64) * L * L));
      cytnx_complex64 *h_I = (cytnx_complex64 *)calloc(L * L, sizeof(cytnx_complex64));
      for (auto i = 0; i < L; i++) h_I[i * L + i] = cytnx_complex64(1, 0);

      checkCudaErrors(
        hipMemcpy(d_I, h_I, sizeof(cytnx_complex64) * L * L, hipMemcpyHostToDevice));

      checkCudaErrors(hipsolverDnCgetrs(cusolverH, HIPBLAS_OP_N, L, L, (hipFloatComplex *)ten->Mem, L,
                                       ipiv, (hipFloatComplex *)d_I, L, devinfo));
      checkCudaErrors(hipMemcpy(&info, devinfo, sizeof(cytnx_int32), hipMemcpyDeviceToHost));

      cytnx_error_msg(info != 0, "%s %d",
                      "ERROR in cuSolver function 'hipsolverDnCgetrs': cuBlas INFO = ", info);

      checkCudaErrors(
        hipMemcpy(ten->Mem, d_I, sizeof(cytnx_complex64) * L * L, hipMemcpyDeviceToDevice));

      hipFree(d_I);
      hipFree(d_work);
      hipFree(devinfo);
      hipFree(ipiv);
      free(h_I);
      hipsolverDnDestroy(cusolverH);
    }

  }  // namespace linalg_internal

}  // namespace cytnx
