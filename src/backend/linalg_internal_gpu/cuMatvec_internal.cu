#include "hip/hip_runtime.h"
#include "cuMatvec_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

namespace cytnx {

  namespace linalg_internal {

    template <typename UniType>
    __global__ void cuMatVec_kernel(UniType *out, const UniType *inl, const UniType *inr,
                                    cytnx_int64 Ml, cytnx_int64 Nr) {
      UniType tmp = 0;
      cytnx_uint64 sid = blockIdx.x * blockDim.x + threadIdx.x;
      if (sid < cytnx_uint64(Ml)) {
        for (cytnx_int64 c = 0; c < Nr; c++) {
          tmp += inl[sid * Nr + c] * inr[c];
        }
        out[sid] = tmp;
      }
    }

    /// cuMatvec
    void cuMatvec_internal_cd(boost::intrusive_ptr<Storage_base> &out,
                              const boost::intrusive_ptr<Storage_base> &inl,
                              const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                              const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex128 alpha = cytnx_complex128(1, 0), beta = cytnx_complex128(0, 0);

      hipDoubleComplex *_out = (hipDoubleComplex *)out->data();
      hipDoubleComplex *_inl = (hipDoubleComplex *)inl->data();
      hipDoubleComplex *_inr = (hipDoubleComplex *)inr->data();

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasZgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, (hipDoubleComplex *)&alpha,
      //                             _inl, blsMl, _inr, 1, (hipDoubleComplex *)&beta, _out, 1));
      checkCudaErrors(hipblasZgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, (hipDoubleComplex *)&alpha,
                                  _inl, blsMl, _inr, 1, (hipDoubleComplex *)&beta, _out, 1));

      hipblasDestroy(cublasH);
    }
    void cuMatvec_internal_cf(boost::intrusive_ptr<Storage_base> &out,
                              const boost::intrusive_ptr<Storage_base> &inl,
                              const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                              const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_complex64 alpha = cytnx_complex64(1, 0), beta = cytnx_complex64(0, 0);

      hipFloatComplex *_out = (hipFloatComplex *)out->data();
      hipFloatComplex *_inl = (hipFloatComplex *)inl->data();
      hipFloatComplex *_inr = (hipFloatComplex *)inr->data();

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasCgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, (hipFloatComplex *)&alpha,
      //                             _inl, blsMl, _inr, 1, (hipFloatComplex *)&beta, _out, 1));
      checkCudaErrors(hipblasCgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, (hipFloatComplex *)&alpha,
                                  _inl, blsMl, _inr, 1, (hipFloatComplex *)&beta, _out, 1));

      hipblasDestroy(cublasH);
    }

    void cuMatvec_internal_d(boost::intrusive_ptr<Storage_base> &out,
                             const boost::intrusive_ptr<Storage_base> &inl,
                             const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                             const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_double alpha = 1, beta = 0;

      cytnx_double *_out = (cytnx_double *)out->data();
      cytnx_double *_inl = (cytnx_double *)inl->data();
      cytnx_double *_inr = (cytnx_double *)inr->data();

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasDgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, &alpha, _inl, blsMl, _inr,
      // 1, &beta, _out, 1));
      checkCudaErrors(hipblasDgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, &alpha, _inl, blsNr, _inr, 1,
                                  &beta, _out, 1));

      hipblasDestroy(cublasH);
    }
    void cuMatvec_internal_f(boost::intrusive_ptr<Storage_base> &out,
                             const boost::intrusive_ptr<Storage_base> &inl,
                             const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                             const cytnx_int64 &Nr) {
      // create handles:
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      cytnx_float alpha = 1, beta = 0;

      cytnx_float *_out = (cytnx_float *)out->data();
      cytnx_float *_inl = (cytnx_float *)inl->data();
      cytnx_float *_inr = (cytnx_float *)inr->data();

      // query working space :
      cytnx_int32 blsMl = Ml, blsNr = Nr;
      // checkCudaErrors(hipblasSgemv(cublasH, HIPBLAS_OP_T, blsMl, blsNr, &alpha, _inr, blsMl, _inl,
      // 1,
      //                             &beta, _out, 1));
      checkCudaErrors(hipblasSgemv(cublasH, HIPBLAS_OP_T, blsNr, blsMl, &alpha, _inl, blsNr, _inr, 1,
                                  &beta, _out, 1));

      hipblasDestroy(cublasH);
    }
    void cuMatvec_internal_i64(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_int64 *_out = (cytnx_int64 *)out->data();
      cytnx_int64 *_inl = (cytnx_int64 *)inl->data();
      cytnx_int64 *_inr = (cytnx_int64 *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_u64(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->data();
      cytnx_uint64 *_inl = (cytnx_uint64 *)inl->data();
      cytnx_uint64 *_inr = (cytnx_uint64 *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_i32(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_int64 *_out = (cytnx_int64 *)out->data();
      cytnx_int64 *_inl = (cytnx_int64 *)inl->data();
      cytnx_int64 *_inr = (cytnx_int64 *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_u32(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_uint64 *_out = (cytnx_uint64 *)out->data();
      cytnx_uint64 *_inl = (cytnx_uint64 *)inl->data();
      cytnx_uint64 *_inr = (cytnx_uint64 *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_i16(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_int16 *_out = (cytnx_int16 *)out->data();
      cytnx_int16 *_inl = (cytnx_int16 *)inl->data();
      cytnx_int16 *_inr = (cytnx_int16 *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_u16(boost::intrusive_ptr<Storage_base> &out,
                               const boost::intrusive_ptr<Storage_base> &inl,
                               const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                               const cytnx_int64 &Nr) {
      cytnx_uint16 *_out = (cytnx_uint16 *)out->data();
      cytnx_uint16 *_inl = (cytnx_uint16 *)inl->data();
      cytnx_uint16 *_inr = (cytnx_uint16 *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }
    void cuMatvec_internal_b(boost::intrusive_ptr<Storage_base> &out,
                             const boost::intrusive_ptr<Storage_base> &inl,
                             const boost::intrusive_ptr<Storage_base> &inr, const cytnx_int64 &Ml,
                             const cytnx_int64 &Nr) {
      cytnx_bool *_out = (cytnx_bool *)out->data();
      cytnx_bool *_inl = (cytnx_bool *)inl->data();
      cytnx_bool *_inr = (cytnx_bool *)inr->data();

      cytnx_uint64 Nblocks = (cytnx_uint64(Ml)) / 512;
      if ((cytnx_uint64(Ml)) % 512) Nblocks += 1;

      cuMatVec_kernel<<<Nblocks, 512>>>(_out, _inl, _inr, Ml, Nr);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
