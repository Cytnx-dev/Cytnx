#include "linalg/linalg_internal_gpu/cuEig_internal.hpp"
#include "cytnx_error.hpp"
#include "Type.hpp"
#include "backend/lapack_wrapper.hpp"

#include "cuAlloc_gpu.hpp"

namespace cytnx {

  namespace linalg_internal {

    /// cuEig
    void cuEig_internal_cd(const boost::intrusive_ptr<Storage_base> &in,
                           boost::intrusive_ptr<Storage_base> &e,
                           boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverHandle_t cusolverH = NULL;
      hipStream_t stream = NULL;
      hipsolverDnCreate(&cusolverH);
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
      hipsolverSetStream(cusolverH, stream);

      hipDoubleComplex *d_A, *d_W, *d_V;
      hipMalloc((void **)&d_A, sizeof(hipDoubleComplex) * L * L);
      hipMalloc((void **)&d_W, sizeof(hipDoubleComplex) * L);
      hipMemcpy(d_A, in->Mem, sizeof(hipDoubleComplex) * L * L, hipMemcpyHostToDevice);

      int lwork = 0;
      cusolverDnZgeev_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, L, d_A, L, d_W, d_V, L, NULL,
                                 1, &lwork);
      hipDoubleComplex *d_work;
      hipMalloc((void **)&d_work, sizeof(hipDoubleComplex) * lwork);

      int *devInfo;
      hipMalloc((void **)&devInfo, sizeof(int));

      cusolverDnZgeev(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_OP_N, L, d_A, L, d_W, d_V, L,
                      NULL, 1, d_work, lwork, devInfo);

      hipMemcpy(e->Mem, d_W, sizeof(hipDoubleComplex) * L, hipMemcpyDeviceToHost);
      if (v->dtype != Type.Void) {
        hipMemcpy(v->Mem, d_V, sizeof(hipDoubleComplex) * L * L, hipMemcpyDeviceToHost);
      }

      hipFree(d_A);
      hipFree(d_W);
      hipFree(d_V);
      hipFree(d_work);
      hipFree(devInfo);
      hipsolverDnDestroy(cusolverH);
      hipStreamDestroy(stream);
    }

    void cuEig_internal_cf(const boost::intrusive_ptr<Storage_base> &in,
                           boost::intrusive_ptr<Storage_base> &e,
                           boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverHandle_t cusolverH = NULL;
      hipStream_t stream = NULL;
      hipsolverDnCreate(&cusolverH);
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
      hipsolverSetStream(cusolverH, stream);

      hipFloatComplex *d_A, *d_W, *d_V;
      hipMalloc((void **)&d_A, sizeof(hipFloatComplex) * L * L);
      hipMalloc((void **)&d_W, sizeof(hipFloatComplex) * L);
      hipMemcpy(d_A, in->Mem, sizeof(hipFloatComplex) * L * L, hipMemcpyHostToDevice);

      int lwork = 0;
      cusolverDnCgeev_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, L, d_A, L, d_W, d_V, L, NULL,
                                 1, &lwork);
      hipFloatComplex *d_work;
      hipMalloc((void **)&d_work, sizeof(hipFloatComplex) * lwork);

      int *devInfo;
      hipMalloc((void **)&devInfo, sizeof(int));

      cusolverDnCgeev(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_OP_N, L, d_A, L, d_W, d_V, L,
                      NULL, 1, d_work, lwork, devInfo);

      hipMemcpy(e->Mem, d_W, sizeof(hipFloatComplex) * L, hipMemcpyDeviceToHost);
      if (v->dtype != Type.Void) {
        hipMemcpy(v->Mem, d_V, sizeof(hipFloatComplex) * L * L, hipMemcpyDeviceToHost);
      }

      hipFree(d_A);
      hipFree(d_W);
      hipFree(d_V);
      hipFree(d_work);
      hipFree(devInfo);
      hipsolverDnDestroy(cusolverH);
      hipStreamDestroy(stream);
    }

    void cuEig_internal_d(const boost::intrusive_ptr<Storage_base> &in,
                          boost::intrusive_ptr<Storage_base> &e,
                          boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverHandle_t cusolverH = NULL;
      hipStream_t stream = NULL;
      hipsolverDnCreate(&cusolverH);
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
      hipsolverSetStream(cusolverH, stream);

      double *d_A, *d_W, *d_V;
      hipMalloc((void **)&d_A, sizeof(double) * L * L);
      hipMalloc((void **)&d_W, sizeof(double) * L);
      hipMemcpy(d_A, in->Mem, sizeof(double) * L * L, hipMemcpyHostToDevice);

      int lwork = 0;
      cusolverDnDgeev_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, L, d_A, L, d_W, d_V, L, NULL,
                                 1, &lwork);
      double *d_work;
      hipMalloc((void **)&d_work, sizeof(double) * lwork);

      int *devInfo;
      hipMalloc((void **)&devInfo, sizeof(int));

      cusolverDnDgeev(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_OP_N, L, d_A, L, d_W, d_V, L,
                      NULL, 1, d_work, lwork, devInfo);

      hipMemcpy(e->Mem, d_W, sizeof(double) * L, hipMemcpyDeviceToHost);
      if (v->dtype != Type.Void) {
        hipMemcpy(v->Mem, d_V, sizeof(double) * L * L, hipMemcpyDeviceToHost);
      }

      hipFree(d_A);
      hipFree(d_W);
      hipFree(d_V);
      hipFree(d_work);
      hipFree(devInfo);
      hipsolverDnDestroy(cusolverH);
      hipStreamDestroy(stream);
    }

    void cuEig_internal_f(const boost::intrusive_ptr<Storage_base> &in,
                          boost::intrusive_ptr<Storage_base> &e,
                          boost::intrusive_ptr<Storage_base> &v, const cytnx_int64 &L) {
      hipsolverHandle_t cusolverH = NULL;
      hipStream_t stream = NULL;
      hipsolverDnCreate(&cusolverH);
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
      hipsolverSetStream(cusolverH, stream);

      float *d_A, *d_W, *d_V;
      hipMalloc((void **)&d_A, sizeof(float) * L * L);
      hipMalloc((void **)&d_W, sizeof(float) * L);
      hipMemcpy(d_A, in->Mem, sizeof(float) * L * L, hipMemcpyHostToDevice);

      int lwork = 0;
      cusolverDnSgeev_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, L, d_A, L, d_W, d_V, L, NULL,
                                 1, &lwork);
      float *d_work;
      hipMalloc((void **)&d_work, sizeof(float) * lwork);

      int *devInfo;
      hipMalloc((void **)&devInfo, sizeof(int));

      cusolverDnSgeev(cusolverH, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_OP_N, L, d_A, L, d_W, d_V, L,
                      NULL, 1, d_work, lwork, devInfo);

      hipMemcpy(e->Mem, d_W, sizeof(float) * L, hipMemcpyDeviceToHost);
      if (v->dtype != Type.Void) {
        hipMemcpy(v->Mem, d_V, sizeof(float) * L * L, hipMemcpyDeviceToHost);
      }

      hipFree(d_A);
      hipFree(d_W);
      hipFree(d_V);
      hipFree(d_work);
      hipFree(devInfo);
      hipsolverDnDestroy(cusolverH);
      hipStreamDestroy(stream);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
