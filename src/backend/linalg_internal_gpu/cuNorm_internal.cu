#include "cuNorm_internal.hpp"
#include "../utils_internal_interface.hpp"
#include "utils/utils.hpp"
#include "cytnx_error.hpp"
#include "backend/lapack_wrapper.hpp"

#ifdef UNI_OMP
  #include <omp.h>
#endif

namespace cytnx {

  namespace linalg_internal {

    /// cuNorm
    void cuNorm_internal_cd(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));

      checkCudaErrors(
        hipblasDznrm2(cublasH, Rin->len, (hipDoubleComplex *)Rin->Mem, 1, (double *)out));

      hipblasDestroy(cublasH);
    }
    void cuNorm_internal_cf(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));

      checkCudaErrors(hipblasScnrm2(cublasH, Rin->len, (hipComplex *)Rin->Mem, 1, (float *)out));

      hipblasDestroy(cublasH);
    }
    void cuNorm_internal_d(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));

      checkCudaErrors(hipblasDnrm2(cublasH, Rin->len, (double *)Rin->Mem, 1, (double *)out));

      hipblasDestroy(cublasH);
    }
    void cuNorm_internal_f(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      checkCudaErrors(hipblasSnrm2(cublasH, Rin->len, (float *)Rin->Mem, 1, (float *)out));
      hipblasDestroy(cublasH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
