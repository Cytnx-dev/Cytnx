#include "cuNorm_internal.hpp"
#include "backend/utils_internal_interface.hpp"
#include "utils/utils.hpp"
#include "cytnx_error.hpp"
#include "backend/lapack_wrapper.hpp"

namespace cytnx {

  namespace linalg_internal {

    /// cuNorm
    void cuNorm_internal_cd(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));

      checkCudaErrors(
        hipblasDznrm2(cublasH, Rin->size(), (hipDoubleComplex *)Rin->data(), 1, (double *)out));

      hipblasDestroy(cublasH);
    }
    void cuNorm_internal_cf(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));

      checkCudaErrors(
        hipblasScnrm2(cublasH, Rin->size(), (hipComplex *)Rin->data(), 1, (float *)out));

      hipblasDestroy(cublasH);
    }
    void cuNorm_internal_d(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));

      checkCudaErrors(hipblasDnrm2(cublasH, Rin->size(), (double *)Rin->data(), 1, (double *)out));

      hipblasDestroy(cublasH);
    }
    void cuNorm_internal_f(void *out, const boost::intrusive_ptr<Storage_base> &Rin) {
      hipblasHandle_t cublasH = NULL;
      checkCudaErrors(hipblasCreate(&cublasH));
      checkCudaErrors(hipblasSnrm2(cublasH, Rin->size(), (float *)Rin->data(), 1, (float *)out));
      hipblasDestroy(cublasH);
    }

  }  // namespace linalg_internal
}  // namespace cytnx
